#include "hip/hip_runtime.h"
__global__ void RF_with_IF(float *F0, float *F1, float *F2, float *F3, float *F4, float *F5, int *P, const int N)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int Class[3]; 
	Class[0] = 0;
	Class[1] = 0;
	Class[2] = 0;
	if (i < N) {
		if (F[0] <= 48.5) {
			Class[0]++;
		} else {
			if (F[4] <= 1.7) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[2] <= 3.35) {
			if (F[0] <= 99.5) {
				if (F[4] <= 0.6) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			} else {
				Class[2]++;
			}
		} else {
			if (F[3] <= 3.75) {
				Class[0]++;
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 100.5) {
			if (F[4] <= 0.8) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[1] <= 5.45) {
			if (F[3] <= 2.6) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			if (F[3] <= 4.95) {
				if (F[3] <= 2.55) {
					Class[0]++;
				} else {
					if (F[0] <= 112.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[3] <= 4.95) {
				if (F[0] <= 114.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 100.0) {
			if (F[4] <= 0.7) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.5) {
			if (F[3] <= 2.6) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[4] <= 1.7) {
					Class[1]++;
				} else {
					if (F[1] <= 6.05) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			} else {
				if (F[4] <= 1.7) {
					if (F[3] <= 5.35) {
						if (F[0] <= 109.0) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[0] <= 101.0) {
			if (F[1] <= 5.45) {
				if (F[4] <= 0.7) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			} else {
				if (F[0] <= 44.0) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.5) {
			if (F[0] <= 50.5) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[4] <= 1.55) {
				if (F[3] <= 5.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[0] <= 94.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[4] <= 1.65) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 50.0) {
			Class[0]++;
		} else {
			if (F[4] <= 1.65) {
				if (F[4] <= 1.45) {
					Class[1]++;
				} else {
					if (F[2] <= 2.75) {
						Class[1]++;
					} else {
						if (F[0] <= 95.5) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					}
				}
			} else {
				if (F[0] <= 86.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.6) {
			Class[0]++;
		} else {
			if (F[0] <= 102.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[4] <= 1.65) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[1] <= 6.05) {
					if (F[1] <= 5.9) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[1] <= 5.45) {
			if (F[0] <= 54.0) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			if (F[3] <= 4.95) {
				if (F[2] <= 3.55) {
					if (F[1] <= 6.0) {
						Class[1]++;
					} else {
						if (F[4] <= 1.7) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					}
				} else {
					Class[0]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 100.0) {
			if (F[3] <= 2.45) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 50.5) {
			Class[0]++;
		} else {
			if (F[0] <= 98.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 4.85) {
			if (F[4] <= 0.8) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[3] <= 4.85) {
			if (F[4] <= 0.8) {
				Class[0]++;
			} else {
				if (F[1] <= 4.95) {
					if (F[4] <= 1.35) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[1]++;
				}
			}
		} else {
			if (F[0] <= 87.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 52.0) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 4.85) {
			if (F[3] <= 2.6) {
				Class[0]++;
			} else {
				if (F[0] <= 103.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			if (F[3] <= 4.95) {
				if (F[0] <= 98.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[1] <= 6.15) {
					if (F[2] <= 2.75) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[0] <= 52.0) {
			Class[0]++;
		} else {
			if (F[3] <= 4.75) {
				Class[1]++;
			} else {
				if (F[0] <= 92.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[0] <= 50.0) {
			Class[0]++;
		} else {
			if (F[3] <= 4.75) {
				Class[1]++;
			} else {
				if (F[0] <= 93.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[0] <= 52.0) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[0] <= 113.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[2] <= 2.55) {
					if (F[4] <= 1.65) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[0] <= 99.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.6) {
			Class[0]++;
		} else {
			if (F[0] <= 99.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 4.85) {
			if (F[3] <= 2.6) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[4] <= 1.7) {
				if (F[0] <= 117.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 100.5) {
			if (F[4] <= 0.8) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[1] <= 6.15) {
			if (F[4] <= 0.8) {
				Class[0]++;
			} else {
				if (F[0] <= 101.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			if (F[3] <= 5.0) {
				if (F[0] <= 98.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[0] <= 102.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 50.0) {
			Class[0]++;
		} else {
			if (F[3] <= 4.95) {
				if (F[0] <= 103.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[4] <= 1.7) {
					if (F[0] <= 107.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[3] <= 5.0) {
				Class[1]++;
			} else {
				if (F[0] <= 93.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.35) {
			Class[0]++;
		} else {
			if (F[0] <= 101.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[4] <= 1.65) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 52.0) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 52.5) {
			Class[0]++;
		} else {
			if (F[3] <= 4.75) {
				if (F[0] <= 103.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[2] <= 2.75) {
					if (F[4] <= 1.7) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[4] <= 1.65) {
				if (F[3] <= 4.95) {
					Class[1]++;
				} else {
					if (F[1] <= 6.05) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			} else {
				if (F[0] <= 86.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 4.85) {
			if (F[0] <= 50.0) {
				Class[0]++;
			} else {
				if (F[3] <= 4.75) {
					Class[1]++;
				} else {
					if (F[1] <= 6.05) {
						Class[1]++;
					} else {
						if (F[1] <= 6.5) {
							Class[2]++;
						} else {
							Class[1]++;
						}
					}
				}
			}
		} else {
			if (F[0] <= 87.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 52.0) {
			Class[0]++;
		} else {
			if (F[4] <= 1.55) {
				if (F[0] <= 116.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[1] <= 6.05) {
					if (F[2] <= 2.6) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.6) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[3] <= 5.0) {
				if (F[1] <= 4.95) {
					if (F[0] <= 82.5) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					if (F[3] <= 4.75) {
						Class[1]++;
					} else {
						if (F[1] <= 6.05) {
							Class[1]++;
						} else {
							if (F[3] <= 4.85) {
								Class[2]++;
							} else {
								Class[1]++;
							}
						}
					}
				}
			} else {
				Class[2]++;
			}
		}

		if (F[1] <= 5.55) {
			if (F[0] <= 52.0) {
				Class[0]++;
			} else {
				if (F[4] <= 1.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			if (F[3] <= 4.85) {
				if (F[3] <= 2.65) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			} else {
				if (F[0] <= 92.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 1.65) {
			if (F[0] <= 51.0) {
				Class[0]++;
			} else {
				if (F[4] <= 1.55) {
					Class[1]++;
				} else {
					if (F[3] <= 5.25) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			}
		} else {
			if (F[0] <= 86.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[2] <= 2.55) {
					if (F[0] <= 98.5) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[1]++;
				}
			} else {
				if (F[1] <= 6.15) {
					if (F[1] <= 5.9) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 1.65) {
			if (F[1] <= 5.45) {
				if (F[4] <= 0.8) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			} else {
				if (F[3] <= 2.6) {
					Class[0]++;
				} else {
					if (F[0] <= 115.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			}
		} else {
			if (F[0] <= 86.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 52.5) {
			Class[0]++;
		} else {
			if (F[0] <= 100.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 50.0) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 101.0) {
			if (F[3] <= 2.45) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[4] <= 0.7) {
			Class[0]++;
		} else {
			if (F[3] <= 4.75) {
				if (F[2] <= 2.55) {
					if (F[0] <= 85.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[1]++;
				}
			} else {
				if (F[0] <= 92.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[1] <= 5.55) {
			if (F[4] <= 0.8) {
				Class[0]++;
			} else {
				if (F[3] <= 4.25) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			if (F[0] <= 103.5) {
				if (F[4] <= 0.7) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.6) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[0] <= 103.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 100.5) {
			if (F[4] <= 0.8) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 47.5) {
			Class[0]++;
		} else {
			if (F[0] <= 102.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[1] <= 6.35) {
				if (F[4] <= 1.7) {
					if (F[3] <= 5.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					if (F[3] <= 4.85) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			} else {
				if (F[0] <= 91.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[3] <= 5.0) {
				if (F[4] <= 1.65) {
					Class[1]++;
				} else {
					if (F[0] <= 89.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			} else {
				if (F[4] <= 1.7) {
					if (F[4] <= 1.55) {
						Class[2]++;
					} else {
						if (F[2] <= 2.85) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.7) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[0] <= 113.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 52.5) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				if (F[0] <= 103.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[4] <= 1.7) {
					if (F[0] <= 107.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.35) {
			Class[0]++;
		} else {
			if (F[4] <= 1.7) {
				if (F[0] <= 115.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[0] <= 87.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[0] <= 100.5) {
			if (F[0] <= 50.5) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[3] <= 4.85) {
			if (F[3] <= 2.45) {
				Class[0]++;
			} else {
				if (F[4] <= 1.7) {
					Class[1]++;
				} else {
					if (F[2] <= 3.0) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				}
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.5) {
			if (F[4] <= 0.7) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 52.0) {
			Class[0]++;
		} else {
			if (F[1] <= 6.35) {
				if (F[4] <= 1.65) {
					if (F[0] <= 117.5) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.7) {
			Class[0]++;
		} else {
			if (F[3] <= 4.75) {
				Class[1]++;
			} else {
				if (F[1] <= 6.0) {
					if (F[2] <= 3.1) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				} else {
					if (F[3] <= 4.95) {
						if (F[4] <= 1.65) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					} else {
						Class[2]++;
					}
				}
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 52.0) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[1] <= 5.75) {
			if (F[0] <= 49.5) {
				Class[0]++;
			} else {
				if (F[0] <= 103.5) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			if (F[3] <= 4.85) {
				if (F[3] <= 4.75) {
					Class[1]++;
				} else {
					if (F[1] <= 6.5) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				}
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.7) {
			Class[0]++;
		} else {
			if (F[0] <= 101.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 4.95) {
			if (F[0] <= 52.0) {
				Class[0]++;
			} else {
				if (F[0] <= 103.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.5) {
			if (F[0] <= 50.5) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.5) {
			if (F[3] <= 2.35) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[2] <= 2.95) {
			if (F[0] <= 103.5) {
				if (F[3] <= 2.2) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			} else {
				Class[2]++;
			}
		} else {
			if (F[3] <= 3.2) {
				Class[0]++;
			} else {
				if (F[3] <= 4.85) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[1] <= 6.25) {
				if (F[4] <= 1.7) {
					Class[1]++;
				} else {
					if (F[2] <= 3.1) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				}
			} else {
				if (F[4] <= 1.7) {
					if (F[0] <= 103.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[0] <= 100.5) {
			if (F[0] <= 50.0) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.0) {
			if (F[4] <= 0.7) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[4] <= 1.65) {
			if (F[0] <= 50.0) {
				Class[0]++;
			} else {
				if (F[1] <= 7.1) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.5) {
			if (F[0] <= 50.0) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[1] <= 5.55) {
			if (F[4] <= 0.7) {
				Class[0]++;
			} else {
				if (F[3] <= 4.45) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		} else {
			if (F[3] <= 4.95) {
				if (F[4] <= 1.7) {
					if (F[3] <= 2.5) {
						Class[0]++;
					} else {
						Class[1]++;
					}
				} else {
					Class[2]++;
				}
			} else {
				Class[2]++;
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[4] <= 1.65) {
				if (F[3] <= 5.35) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[4] <= 1.85) {
					if (F[2] <= 3.15) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[3] <= 5.0) {
				if (F[1] <= 4.95) {
					Class[2]++;
				} else {
					if (F[0] <= 111.5) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			} else {
				if (F[4] <= 1.7) {
					if (F[3] <= 5.35) {
						if (F[0] <= 109.0) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[3] <= 4.85) {
				Class[1]++;
			} else {
				if (F[4] <= 1.7) {
					if (F[3] <= 5.35) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[3] <= 2.6) {
			Class[0]++;
		} else {
			if (F[4] <= 1.7) {
				if (F[3] <= 5.35) {
					if (F[0] <= 117.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			} else {
				if (F[1] <= 6.0) {
					if (F[3] <= 4.95) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[1] <= 6.05) {
				if (F[4] <= 2.1) {
					if (F[0] <= 103.5) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				} else {
					Class[2]++;
				}
			} else {
				if (F[3] <= 4.8) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[4] <= 1.65) {
				if (F[4] <= 1.35) {
					Class[1]++;
				} else {
					if (F[1] <= 7.0) {
						if (F[0] <= 113.0) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					} else {
						Class[2]++;
					}
				}
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.45) {
			Class[0]++;
		} else {
			if (F[4] <= 1.65) {
				if (F[3] <= 5.0) {
					Class[1]++;
				} else {
					if (F[0] <= 107.0) {
						Class[1]++;
					} else {
						Class[2]++;
					}
				}
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.5) {
			Class[0]++;
		} else {
			if (F[0] <= 103.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[1] <= 5.75) {
			if (F[3] <= 2.35) {
				Class[0]++;
			} else {
				if (F[3] <= 4.45) {
					Class[1]++;
				} else {
					if (F[2] <= 2.65) {
						Class[2]++;
					} else {
						Class[1]++;
					}
				}
			}
		} else {
			if (F[0] <= 100.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 2.6) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[3] <= 4.75) {
			if (F[3] <= 2.45) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			if (F[1] <= 6.05) {
				if (F[4] <= 1.85) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			} else {
				if (F[0] <= 87.0) {
					Class[1]++;
				} else {
					Class[2]++;
				}
			}
		}

		if (F[0] <= 100.0) {
			if (F[0] <= 50.0) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 102.0) {
			if (F[3] <= 2.5) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.0) {
			if (F[3] <= 2.6) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[0] <= 100.5) {
			if (F[3] <= 2.45) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[0] <= 100.5) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}

		if (F[0] <= 100.5) {
			if (F[3] <= 2.45) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			Class[2]++;
		}

		if (F[4] <= 0.8) {
			Class[0]++;
		} else {
			if (F[0] <= 100.0) {
				Class[1]++;
			} else {
				Class[2]++;
			}
		}
		int p0 = (Class[0] > Class[1])?0:1;
		int Q0 = (Class[0] > Class[1])?Class[0]:Class[1];
		int p1 = (Class[2] > Q0)?2:p0;
		int Q1 = (Class[2] > Q0)?Class[2]:Q0;
		P[i] = p1;	
	}
}