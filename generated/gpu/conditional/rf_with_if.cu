#include "hip/hip_runtime.h"
#include <sys/time.h>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}
#define N_ELEM 4999999


#include <hip/hip_runtime.h>
#include <stdio.h>


void initialData(float *ip, int size){

  for (int i = 0; i < size; i++){
      ip[i] = (float)( rand() & 15 );
  }
  return;
}
FILE *inFile;
FILE *outFile;

void readInFile(float **ip) {
    FILE *file = fopen("assets/datasets/SUSY.csv","r");
    if (file == NULL) {
        printf("[CUDA]: Failed to open the file.\n");
        exit(1);
    }

    char line[1000];
    
    int row = 0;
    fgets(line, sizeof(line), file);
    while (fgets(line, sizeof(line), file)) {
        char *token = strtok(line, ",");
        
        int column = 0;
        while (token != NULL) {
            ip[column][row] = atof(token);
            token = strtok(NULL, ",");
            column++;
        }

        row++;
        if(row > N_ELEM) break;
    }

    fclose(file);
}
void writeOutFile(int value){
    outFile = fopen("out_rf_with_if.csv","a");
    fprintf(outFile, "%d\n", value);
    fclose(outFile);
} 

void registerTime(float value){
    outFile = fopen("results/results.csv","a");
    fprintf(outFile, ",%.2f", value);
    fclose(outFile);
}__global__ void RF_with_IF(float *F0, float *F1, float *F2, float *F3, float *F4, float *F5, float *F6, float *F7, float *F8, float *F9, float *F10, float *F11, float *F12, float *F13, float *F14, float *F15, float *F16, float *F17, float *F18, int *P, const int N)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int Class[2]; 
	Class[0] = 0;
	Class[1] = 0;
	if (i < N) {
		if (F16[i]<= 1.734) {
			if (F7[i]<= 1.192) {
				Class[1]++;
			} else {
				Class[0]++;
			}
		} else {
			if (F10[i]<= -0.27) {
				Class[0]++;
			} else {
				Class[0]++;
			}
		}

		if (F7[i]<= 1.218) {
			if (F15[i]<= 0.723) {
				Class[1]++;
			} else {
				Class[1]++;
			}
		} else {
			if (F7[i]<= 1.571) {
				Class[0]++;
			} else {
				Class[0]++;
			}
		}
		int p0 = (Class[0] > Class[1])?0:1;
		int Q0 = (Class[0] > Class[1])?Class[0]:Class[1];
		P[i] = p0;
	}
}int main(int argc, char ** argv) {
     
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 4999999;
    printf("[CUDA]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float  * h_0,  * h_1,  * h_2,  * h_3,  * h_4,  * h_5,  * h_6,  * h_7,  * h_8,  * h_9,  * h_10,  * h_11,  * h_12,  * h_13,  * h_14,  * h_15,  * h_16,  * h_17,  * h_18;
    int * h_P, * hostRef;
    	h_0 = (float * ) malloc(nBytes);
	h_1 = (float * ) malloc(nBytes);
	h_2 = (float * ) malloc(nBytes);
	h_3 = (float * ) malloc(nBytes);
	h_4 = (float * ) malloc(nBytes);
	h_5 = (float * ) malloc(nBytes);
	h_6 = (float * ) malloc(nBytes);
	h_7 = (float * ) malloc(nBytes);
	h_8 = (float * ) malloc(nBytes);
	h_9 = (float * ) malloc(nBytes);
	h_10 = (float * ) malloc(nBytes);
	h_11 = (float * ) malloc(nBytes);
	h_12 = (float * ) malloc(nBytes);
	h_13 = (float * ) malloc(nBytes);
	h_14 = (float * ) malloc(nBytes);
	h_15 = (float * ) malloc(nBytes);
	h_16 = (float * ) malloc(nBytes);
	h_17 = (float * ) malloc(nBytes);
	h_18 = (float * ) malloc(nBytes);
    hostRef = (int * ) malloc(nBytes);
    h_P = (int * ) malloc(nBytes);

    printf("[CUDA]: Start Reading Dataset\n");
	float* h[19] = { h_0, h_1, h_2, h_3, h_4, h_5, h_6, h_7, h_8, h_9, h_10, h_11, h_12, h_13, h_14, h_15, h_16, h_17, h_18};
	readInFile(h);
	printf("[CUDA]: Finished Reading Dataset\n");


    memset(hostRef, 0, nBytes);
    memset(h_P, 0, nBytes);

    // malloc device global memory
    float * d_0, * d_1, * d_2, * d_3, * d_4, * d_5, * d_6, * d_7, * d_8, * d_9, * d_10, * d_11, * d_12, * d_13, * d_14, * d_15, * d_16, * d_17, * d_18;
    int * d_P;
        CHECK(hipMalloc((float ** ) & d_0, nBytes));
    CHECK(hipMalloc((float ** ) & d_1, nBytes));
    CHECK(hipMalloc((float ** ) & d_2, nBytes));
    CHECK(hipMalloc((float ** ) & d_3, nBytes));
    CHECK(hipMalloc((float ** ) & d_4, nBytes));
    CHECK(hipMalloc((float ** ) & d_5, nBytes));
    CHECK(hipMalloc((float ** ) & d_6, nBytes));
    CHECK(hipMalloc((float ** ) & d_7, nBytes));
    CHECK(hipMalloc((float ** ) & d_8, nBytes));
    CHECK(hipMalloc((float ** ) & d_9, nBytes));
    CHECK(hipMalloc((float ** ) & d_10, nBytes));
    CHECK(hipMalloc((float ** ) & d_11, nBytes));
    CHECK(hipMalloc((float ** ) & d_12, nBytes));
    CHECK(hipMalloc((float ** ) & d_13, nBytes));
    CHECK(hipMalloc((float ** ) & d_14, nBytes));
    CHECK(hipMalloc((float ** ) & d_15, nBytes));
    CHECK(hipMalloc((float ** ) & d_16, nBytes));
    CHECK(hipMalloc((float ** ) & d_17, nBytes));
    CHECK(hipMalloc((float ** ) & d_18, nBytes));
    CHECK(hipMalloc((int ** ) & d_P, nBytes));


    // transfer data from host to device
        CHECK(hipMemcpy(d_0, h_0, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_1, h_1, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_2, h_2, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_3, h_3, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_4, h_4, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_5, h_5, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_6, h_6, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_7, h_7, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_8, h_8, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_9, h_9, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_10, h_10, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_11, h_11, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_12, h_12, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_13, h_13, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_14, h_14, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_15, h_15, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_16, h_16, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_17, h_17, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_18, h_18, nBytes, hipMemcpyHostToDevice));
    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_with_IF << < grid, block >>> (d_0, d_1, d_2, d_3, d_4, d_5, d_6, d_7, d_8, d_9, d_10, d_11, d_12, d_13, d_14, d_15, d_16, d_17, d_18, d_P, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("[CUDA]: RF with IF - execution time = %.6fms\n", elapsed_time);

    registerTime(elapsed_time);
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    hipDeviceSynchronize();
    for(int i = 0; i < nElem; i++){
        writeOutFile(h_P[i]);
    }

    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    printf("\n ");

    // free host memory
        hipFree(d_0);
    hipFree(d_1);
    hipFree(d_2);
    hipFree(d_3);
    hipFree(d_4);
    hipFree(d_5);
    hipFree(d_6);
    hipFree(d_7);
    hipFree(d_8);
    hipFree(d_9);
    hipFree(d_10);
    hipFree(d_11);
    hipFree(d_12);
    hipFree(d_13);
    hipFree(d_14);
    hipFree(d_15);
    hipFree(d_16);
    hipFree(d_17);
    hipFree(d_18);
    hipFree(d_P);
        free(h_0);
    free(h_1);
    free(h_2);
    free(h_3);
    free(h_4);
    free(h_5);
    free(h_6);
    free(h_7);
    free(h_8);
    free(h_9);
    free(h_10);
    free(h_11);
    free(h_12);
    free(h_13);
    free(h_14);
    free(h_15);
    free(h_16);
    free(h_17);
    free(h_18);
    free(hostRef);

    return (0);
 }