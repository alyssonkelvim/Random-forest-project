#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 50000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 64
#define _4TREE
#define EXP_NAME "_RF_CALC_4TREE_6LV_1INPUT_50000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt1_0_0 = (input[i] > (1+offset1));
int t1_0_0 = rt1_0_0 * (2 + (input[i] > (2+offset1)));
t1_0_0 += (1 - rt1_0_0) * (input[i] > (0+offset1));

int rt1_0_1 = (input[i] > (5+offset1));
int t1_0_1 = rt1_0_1 * (2 + (input[i] > (6+offset1)));
t1_0_1 += (1 - rt1_0_1) * (input[i] > (4+offset1));

int rt1_0_2 = (input[i] > (9+offset1));
int t1_0_2 = rt1_0_2 * (2 + (input[i] > (10+offset1)));
t1_0_2 += (1 - rt1_0_2) * (input[i] > (8+offset1));

int rt1_0_3 = (input[i] > (13+offset1));
int t1_0_3 = rt1_0_3 * (2 + (input[i] > (14+offset1)));
t1_0_3 += (1 - rt1_0_3) * (input[i] > (12+offset1));

int rt1_0_4 = (input[i] > (17+offset1));
int t1_0_4 = rt1_0_4 * (2 + (input[i] > (18+offset1)));
t1_0_4 += (1 - rt1_0_4) * (input[i] > (16+offset1));

int rt1_0_5 = (input[i] > (21+offset1));
int t1_0_5 = rt1_0_5 * (2 + (input[i] > (22+offset1)));
t1_0_5 += (1 - rt1_0_5) * (input[i] > (20+offset1));

int rt1_0_6 = (input[i] > (25+offset1));
int t1_0_6 = rt1_0_6 * (2 + (input[i] > (26+offset1)));
t1_0_6 += (1 - rt1_0_6) * (input[i] > (24+offset1));

int rt1_0_7 = (input[i] > (29+offset1));
int t1_0_7 = rt1_0_7 * (2 + (input[i] > (30+offset1)));
t1_0_7 += (1 - rt1_0_7) * (input[i] > (28+offset1));

int rt1_0_8 = (input[i] > (33+offset1));
int t1_0_8 = rt1_0_8 * (2 + (input[i] > (34+offset1)));
t1_0_8 += (1 - rt1_0_8) * (input[i] > (32+offset1));

int rt1_0_9 = (input[i] > (37+offset1));
int t1_0_9 = rt1_0_9 * (2 + (input[i] > (38+offset1)));
t1_0_9 += (1 - rt1_0_9) * (input[i] > (36+offset1));

int rt1_0_10 = (input[i] > (41+offset1));
int t1_0_10 = rt1_0_10 * (2 + (input[i] > (42+offset1)));
t1_0_10 += (1 - rt1_0_10) * (input[i] > (40+offset1));

int rt1_0_11 = (input[i] > (45+offset1));
int t1_0_11 = rt1_0_11 * (2 + (input[i] > (46+offset1)));
t1_0_11 += (1 - rt1_0_11) * (input[i] > (44+offset1));

int rt1_0_12 = (input[i] > (49+offset1));
int t1_0_12 = rt1_0_12 * (2 + (input[i] > (50+offset1)));
t1_0_12 += (1 - rt1_0_12) * (input[i] > (48+offset1));

int rt1_0_13 = (input[i] > (53+offset1));
int t1_0_13 = rt1_0_13 * (2 + (input[i] > (54+offset1)));
t1_0_13 += (1 - rt1_0_13) * (input[i] > (52+offset1));

int rt1_0_14 = (input[i] > (57+offset1));
int t1_0_14 = rt1_0_14 * (2 + (input[i] > (58+offset1)));
t1_0_14 += (1 - rt1_0_14) * (input[i] > (56+offset1));

int rt1_0_15 = (input[i] > (61+offset1));
int t1_0_15 = rt1_0_15 * (2 + (input[i] > (62+offset1)));
t1_0_15 += (1 - rt1_0_15) * (input[i] > (60+offset1));

int rt1_1_0 = (input[i] > (3+offset1));
int t1_1_0 = rt1_1_0 * (4 + t1_0_1);
t1_1_0 += (1 - rt1_1_0) * t1_0_0;

int rt1_1_1 = (input[i] > (11+offset1));
int t1_1_1 = rt1_1_1 * (4 + t1_0_3);
t1_1_1 += (1 - rt1_1_1) * t1_0_2;

int rt1_1_2 = (input[i] > (19+offset1));
int t1_1_2 = rt1_1_2 * (4 + t1_0_5);
t1_1_2 += (1 - rt1_1_2) * t1_0_4;

int rt1_1_3 = (input[i] > (27+offset1));
int t1_1_3 = rt1_1_3 * (4 + t1_0_7);
t1_1_3 += (1 - rt1_1_3) * t1_0_6;

int rt1_1_4 = (input[i] > (35+offset1));
int t1_1_4 = rt1_1_4 * (4 + t1_0_9);
t1_1_4 += (1 - rt1_1_4) * t1_0_8;

int rt1_1_5 = (input[i] > (43+offset1));
int t1_1_5 = rt1_1_5 * (4 + t1_0_11);
t1_1_5 += (1 - rt1_1_5) * t1_0_10;

int rt1_1_6 = (input[i] > (51+offset1));
int t1_1_6 = rt1_1_6 * (4 + t1_0_13);
t1_1_6 += (1 - rt1_1_6) * t1_0_12;

int rt1_1_7 = (input[i] > (59+offset1));
int t1_1_7 = rt1_1_7 * (4 + t1_0_15);
t1_1_7 += (1 - rt1_1_7) * t1_0_14;

int rt1_2_0 = (input[i] > (7+offset1));
int t1_2_0 = rt1_2_0 * (8 + t1_1_1);
t1_2_0 += (1 - rt1_2_0) * t1_1_0;

int rt1_2_1 = (input[i] > (23+offset1));
int t1_2_1 = rt1_2_1 * (8 + t1_1_3);
t1_2_1 += (1 - rt1_2_1) * t1_1_2;

int rt1_2_2 = (input[i] > (39+offset1));
int t1_2_2 = rt1_2_2 * (8 + t1_1_5);
t1_2_2 += (1 - rt1_2_2) * t1_1_4;

int rt1_2_3 = (input[i] > (55+offset1));
int t1_2_3 = rt1_2_3 * (8 + t1_1_7);
t1_2_3 += (1 - rt1_2_3) * t1_1_6;

int rt1_3_0 = (input[i] > (15+offset1));
int t1_3_0 = rt1_3_0 * (16 + t1_2_1);
t1_3_0 += (1 - rt1_3_0) * t1_2_0;

int rt1_3_1 = (input[i] > (47+offset1));
int t1_3_1 = rt1_3_1 * (16 + t1_2_3);
t1_3_1 += (1 - rt1_3_1) * t1_2_2;

int rt1_4_0 = (input[i] > (31+offset1));
int t1_4_0 = rt1_4_0 * (32 + t1_3_1);
t1_4_0 += (1 - rt1_4_0) * t1_3_0;

int rt2_0_0 = (input[i] > (1+offset2));
int t2_0_0 = rt2_0_0 * (2 + (input[i] > (2+offset2)));
t2_0_0 += (1 - rt2_0_0) * (input[i] > (0+offset2));

int rt2_0_1 = (input[i] > (5+offset2));
int t2_0_1 = rt2_0_1 * (2 + (input[i] > (6+offset2)));
t2_0_1 += (1 - rt2_0_1) * (input[i] > (4+offset2));

int rt2_0_2 = (input[i] > (9+offset2));
int t2_0_2 = rt2_0_2 * (2 + (input[i] > (10+offset2)));
t2_0_2 += (1 - rt2_0_2) * (input[i] > (8+offset2));

int rt2_0_3 = (input[i] > (13+offset2));
int t2_0_3 = rt2_0_3 * (2 + (input[i] > (14+offset2)));
t2_0_3 += (1 - rt2_0_3) * (input[i] > (12+offset2));

int rt2_0_4 = (input[i] > (17+offset2));
int t2_0_4 = rt2_0_4 * (2 + (input[i] > (18+offset2)));
t2_0_4 += (1 - rt2_0_4) * (input[i] > (16+offset2));

int rt2_0_5 = (input[i] > (21+offset2));
int t2_0_5 = rt2_0_5 * (2 + (input[i] > (22+offset2)));
t2_0_5 += (1 - rt2_0_5) * (input[i] > (20+offset2));

int rt2_0_6 = (input[i] > (25+offset2));
int t2_0_6 = rt2_0_6 * (2 + (input[i] > (26+offset2)));
t2_0_6 += (1 - rt2_0_6) * (input[i] > (24+offset2));

int rt2_0_7 = (input[i] > (29+offset2));
int t2_0_7 = rt2_0_7 * (2 + (input[i] > (30+offset2)));
t2_0_7 += (1 - rt2_0_7) * (input[i] > (28+offset2));

int rt2_0_8 = (input[i] > (33+offset2));
int t2_0_8 = rt2_0_8 * (2 + (input[i] > (34+offset2)));
t2_0_8 += (1 - rt2_0_8) * (input[i] > (32+offset2));

int rt2_0_9 = (input[i] > (37+offset2));
int t2_0_9 = rt2_0_9 * (2 + (input[i] > (38+offset2)));
t2_0_9 += (1 - rt2_0_9) * (input[i] > (36+offset2));

int rt2_0_10 = (input[i] > (41+offset2));
int t2_0_10 = rt2_0_10 * (2 + (input[i] > (42+offset2)));
t2_0_10 += (1 - rt2_0_10) * (input[i] > (40+offset2));

int rt2_0_11 = (input[i] > (45+offset2));
int t2_0_11 = rt2_0_11 * (2 + (input[i] > (46+offset2)));
t2_0_11 += (1 - rt2_0_11) * (input[i] > (44+offset2));

int rt2_0_12 = (input[i] > (49+offset2));
int t2_0_12 = rt2_0_12 * (2 + (input[i] > (50+offset2)));
t2_0_12 += (1 - rt2_0_12) * (input[i] > (48+offset2));

int rt2_0_13 = (input[i] > (53+offset2));
int t2_0_13 = rt2_0_13 * (2 + (input[i] > (54+offset2)));
t2_0_13 += (1 - rt2_0_13) * (input[i] > (52+offset2));

int rt2_0_14 = (input[i] > (57+offset2));
int t2_0_14 = rt2_0_14 * (2 + (input[i] > (58+offset2)));
t2_0_14 += (1 - rt2_0_14) * (input[i] > (56+offset2));

int rt2_0_15 = (input[i] > (61+offset2));
int t2_0_15 = rt2_0_15 * (2 + (input[i] > (62+offset2)));
t2_0_15 += (1 - rt2_0_15) * (input[i] > (60+offset2));

int rt2_1_0 = (input[i] > (3+offset2));
int t2_1_0 = rt2_1_0 * (4 + t2_0_1);
t2_1_0 += (1 - rt2_1_0) * t2_0_0;

int rt2_1_1 = (input[i] > (11+offset2));
int t2_1_1 = rt2_1_1 * (4 + t2_0_3);
t2_1_1 += (1 - rt2_1_1) * t2_0_2;

int rt2_1_2 = (input[i] > (19+offset2));
int t2_1_2 = rt2_1_2 * (4 + t2_0_5);
t2_1_2 += (1 - rt2_1_2) * t2_0_4;

int rt2_1_3 = (input[i] > (27+offset2));
int t2_1_3 = rt2_1_3 * (4 + t2_0_7);
t2_1_3 += (1 - rt2_1_3) * t2_0_6;

int rt2_1_4 = (input[i] > (35+offset2));
int t2_1_4 = rt2_1_4 * (4 + t2_0_9);
t2_1_4 += (1 - rt2_1_4) * t2_0_8;

int rt2_1_5 = (input[i] > (43+offset2));
int t2_1_5 = rt2_1_5 * (4 + t2_0_11);
t2_1_5 += (1 - rt2_1_5) * t2_0_10;

int rt2_1_6 = (input[i] > (51+offset2));
int t2_1_6 = rt2_1_6 * (4 + t2_0_13);
t2_1_6 += (1 - rt2_1_6) * t2_0_12;

int rt2_1_7 = (input[i] > (59+offset2));
int t2_1_7 = rt2_1_7 * (4 + t2_0_15);
t2_1_7 += (1 - rt2_1_7) * t2_0_14;

int rt2_2_0 = (input[i] > (7+offset2));
int t2_2_0 = rt2_2_0 * (8 + t2_1_1);
t2_2_0 += (1 - rt2_2_0) * t2_1_0;

int rt2_2_1 = (input[i] > (23+offset2));
int t2_2_1 = rt2_2_1 * (8 + t2_1_3);
t2_2_1 += (1 - rt2_2_1) * t2_1_2;

int rt2_2_2 = (input[i] > (39+offset2));
int t2_2_2 = rt2_2_2 * (8 + t2_1_5);
t2_2_2 += (1 - rt2_2_2) * t2_1_4;

int rt2_2_3 = (input[i] > (55+offset2));
int t2_2_3 = rt2_2_3 * (8 + t2_1_7);
t2_2_3 += (1 - rt2_2_3) * t2_1_6;

int rt2_3_0 = (input[i] > (15+offset2));
int t2_3_0 = rt2_3_0 * (16 + t2_2_1);
t2_3_0 += (1 - rt2_3_0) * t2_2_0;

int rt2_3_1 = (input[i] > (47+offset2));
int t2_3_1 = rt2_3_1 * (16 + t2_2_3);
t2_3_1 += (1 - rt2_3_1) * t2_2_2;

int rt2_4_0 = (input[i] > (31+offset2));
int t2_4_0 = rt2_4_0 * (32 + t2_3_1);
t2_4_0 += (1 - rt2_4_0) * t2_3_0;

int rt3_0_0 = (input[i] > (1+offset3));
int t3_0_0 = rt3_0_0 * (2 + (input[i] > (2+offset3)));
t3_0_0 += (1 - rt3_0_0) * (input[i] > (0+offset3));

int rt3_0_1 = (input[i] > (5+offset3));
int t3_0_1 = rt3_0_1 * (2 + (input[i] > (6+offset3)));
t3_0_1 += (1 - rt3_0_1) * (input[i] > (4+offset3));

int rt3_0_2 = (input[i] > (9+offset3));
int t3_0_2 = rt3_0_2 * (2 + (input[i] > (10+offset3)));
t3_0_2 += (1 - rt3_0_2) * (input[i] > (8+offset3));

int rt3_0_3 = (input[i] > (13+offset3));
int t3_0_3 = rt3_0_3 * (2 + (input[i] > (14+offset3)));
t3_0_3 += (1 - rt3_0_3) * (input[i] > (12+offset3));

int rt3_0_4 = (input[i] > (17+offset3));
int t3_0_4 = rt3_0_4 * (2 + (input[i] > (18+offset3)));
t3_0_4 += (1 - rt3_0_4) * (input[i] > (16+offset3));

int rt3_0_5 = (input[i] > (21+offset3));
int t3_0_5 = rt3_0_5 * (2 + (input[i] > (22+offset3)));
t3_0_5 += (1 - rt3_0_5) * (input[i] > (20+offset3));

int rt3_0_6 = (input[i] > (25+offset3));
int t3_0_6 = rt3_0_6 * (2 + (input[i] > (26+offset3)));
t3_0_6 += (1 - rt3_0_6) * (input[i] > (24+offset3));

int rt3_0_7 = (input[i] > (29+offset3));
int t3_0_7 = rt3_0_7 * (2 + (input[i] > (30+offset3)));
t3_0_7 += (1 - rt3_0_7) * (input[i] > (28+offset3));

int rt3_0_8 = (input[i] > (33+offset3));
int t3_0_8 = rt3_0_8 * (2 + (input[i] > (34+offset3)));
t3_0_8 += (1 - rt3_0_8) * (input[i] > (32+offset3));

int rt3_0_9 = (input[i] > (37+offset3));
int t3_0_9 = rt3_0_9 * (2 + (input[i] > (38+offset3)));
t3_0_9 += (1 - rt3_0_9) * (input[i] > (36+offset3));

int rt3_0_10 = (input[i] > (41+offset3));
int t3_0_10 = rt3_0_10 * (2 + (input[i] > (42+offset3)));
t3_0_10 += (1 - rt3_0_10) * (input[i] > (40+offset3));

int rt3_0_11 = (input[i] > (45+offset3));
int t3_0_11 = rt3_0_11 * (2 + (input[i] > (46+offset3)));
t3_0_11 += (1 - rt3_0_11) * (input[i] > (44+offset3));

int rt3_0_12 = (input[i] > (49+offset3));
int t3_0_12 = rt3_0_12 * (2 + (input[i] > (50+offset3)));
t3_0_12 += (1 - rt3_0_12) * (input[i] > (48+offset3));

int rt3_0_13 = (input[i] > (53+offset3));
int t3_0_13 = rt3_0_13 * (2 + (input[i] > (54+offset3)));
t3_0_13 += (1 - rt3_0_13) * (input[i] > (52+offset3));

int rt3_0_14 = (input[i] > (57+offset3));
int t3_0_14 = rt3_0_14 * (2 + (input[i] > (58+offset3)));
t3_0_14 += (1 - rt3_0_14) * (input[i] > (56+offset3));

int rt3_0_15 = (input[i] > (61+offset3));
int t3_0_15 = rt3_0_15 * (2 + (input[i] > (62+offset3)));
t3_0_15 += (1 - rt3_0_15) * (input[i] > (60+offset3));

int rt3_1_0 = (input[i] > (3+offset3));
int t3_1_0 = rt3_1_0 * (4 + t3_0_1);
t3_1_0 += (1 - rt3_1_0) * t3_0_0;

int rt3_1_1 = (input[i] > (11+offset3));
int t3_1_1 = rt3_1_1 * (4 + t3_0_3);
t3_1_1 += (1 - rt3_1_1) * t3_0_2;

int rt3_1_2 = (input[i] > (19+offset3));
int t3_1_2 = rt3_1_2 * (4 + t3_0_5);
t3_1_2 += (1 - rt3_1_2) * t3_0_4;

int rt3_1_3 = (input[i] > (27+offset3));
int t3_1_3 = rt3_1_3 * (4 + t3_0_7);
t3_1_3 += (1 - rt3_1_3) * t3_0_6;

int rt3_1_4 = (input[i] > (35+offset3));
int t3_1_4 = rt3_1_4 * (4 + t3_0_9);
t3_1_4 += (1 - rt3_1_4) * t3_0_8;

int rt3_1_5 = (input[i] > (43+offset3));
int t3_1_5 = rt3_1_5 * (4 + t3_0_11);
t3_1_5 += (1 - rt3_1_5) * t3_0_10;

int rt3_1_6 = (input[i] > (51+offset3));
int t3_1_6 = rt3_1_6 * (4 + t3_0_13);
t3_1_6 += (1 - rt3_1_6) * t3_0_12;

int rt3_1_7 = (input[i] > (59+offset3));
int t3_1_7 = rt3_1_7 * (4 + t3_0_15);
t3_1_7 += (1 - rt3_1_7) * t3_0_14;

int rt3_2_0 = (input[i] > (7+offset3));
int t3_2_0 = rt3_2_0 * (8 + t3_1_1);
t3_2_0 += (1 - rt3_2_0) * t3_1_0;

int rt3_2_1 = (input[i] > (23+offset3));
int t3_2_1 = rt3_2_1 * (8 + t3_1_3);
t3_2_1 += (1 - rt3_2_1) * t3_1_2;

int rt3_2_2 = (input[i] > (39+offset3));
int t3_2_2 = rt3_2_2 * (8 + t3_1_5);
t3_2_2 += (1 - rt3_2_2) * t3_1_4;

int rt3_2_3 = (input[i] > (55+offset3));
int t3_2_3 = rt3_2_3 * (8 + t3_1_7);
t3_2_3 += (1 - rt3_2_3) * t3_1_6;

int rt3_3_0 = (input[i] > (15+offset3));
int t3_3_0 = rt3_3_0 * (16 + t3_2_1);
t3_3_0 += (1 - rt3_3_0) * t3_2_0;

int rt3_3_1 = (input[i] > (47+offset3));
int t3_3_1 = rt3_3_1 * (16 + t3_2_3);
t3_3_1 += (1 - rt3_3_1) * t3_2_2;

int rt3_4_0 = (input[i] > (31+offset3));
int t3_4_0 = rt3_4_0 * (32 + t3_3_1);
t3_4_0 += (1 - rt3_4_0) * t3_3_0;

vR[i] = t0_4_0+ (t0_4_0 == t1_4_0)+ (t0_4_0 == t2_4_0)+ (t0_4_0 == t3_4_0);
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
