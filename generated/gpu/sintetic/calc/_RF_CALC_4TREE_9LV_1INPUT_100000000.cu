#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 512
#define _4TREE
#define EXP_NAME "_RF_CALC_4TREE_9LV_1INPUT_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_0_16 = (input[i] > (65+offset0));
int t0_0_16 = rt0_0_16 * (2 + (input[i] > (66+offset0)));
t0_0_16 += (1 - rt0_0_16) * (input[i] > (64+offset0));

int rt0_0_17 = (input[i] > (69+offset0));
int t0_0_17 = rt0_0_17 * (2 + (input[i] > (70+offset0)));
t0_0_17 += (1 - rt0_0_17) * (input[i] > (68+offset0));

int rt0_0_18 = (input[i] > (73+offset0));
int t0_0_18 = rt0_0_18 * (2 + (input[i] > (74+offset0)));
t0_0_18 += (1 - rt0_0_18) * (input[i] > (72+offset0));

int rt0_0_19 = (input[i] > (77+offset0));
int t0_0_19 = rt0_0_19 * (2 + (input[i] > (78+offset0)));
t0_0_19 += (1 - rt0_0_19) * (input[i] > (76+offset0));

int rt0_0_20 = (input[i] > (81+offset0));
int t0_0_20 = rt0_0_20 * (2 + (input[i] > (82+offset0)));
t0_0_20 += (1 - rt0_0_20) * (input[i] > (80+offset0));

int rt0_0_21 = (input[i] > (85+offset0));
int t0_0_21 = rt0_0_21 * (2 + (input[i] > (86+offset0)));
t0_0_21 += (1 - rt0_0_21) * (input[i] > (84+offset0));

int rt0_0_22 = (input[i] > (89+offset0));
int t0_0_22 = rt0_0_22 * (2 + (input[i] > (90+offset0)));
t0_0_22 += (1 - rt0_0_22) * (input[i] > (88+offset0));

int rt0_0_23 = (input[i] > (93+offset0));
int t0_0_23 = rt0_0_23 * (2 + (input[i] > (94+offset0)));
t0_0_23 += (1 - rt0_0_23) * (input[i] > (92+offset0));

int rt0_0_24 = (input[i] > (97+offset0));
int t0_0_24 = rt0_0_24 * (2 + (input[i] > (98+offset0)));
t0_0_24 += (1 - rt0_0_24) * (input[i] > (96+offset0));

int rt0_0_25 = (input[i] > (101+offset0));
int t0_0_25 = rt0_0_25 * (2 + (input[i] > (102+offset0)));
t0_0_25 += (1 - rt0_0_25) * (input[i] > (100+offset0));

int rt0_0_26 = (input[i] > (105+offset0));
int t0_0_26 = rt0_0_26 * (2 + (input[i] > (106+offset0)));
t0_0_26 += (1 - rt0_0_26) * (input[i] > (104+offset0));

int rt0_0_27 = (input[i] > (109+offset0));
int t0_0_27 = rt0_0_27 * (2 + (input[i] > (110+offset0)));
t0_0_27 += (1 - rt0_0_27) * (input[i] > (108+offset0));

int rt0_0_28 = (input[i] > (113+offset0));
int t0_0_28 = rt0_0_28 * (2 + (input[i] > (114+offset0)));
t0_0_28 += (1 - rt0_0_28) * (input[i] > (112+offset0));

int rt0_0_29 = (input[i] > (117+offset0));
int t0_0_29 = rt0_0_29 * (2 + (input[i] > (118+offset0)));
t0_0_29 += (1 - rt0_0_29) * (input[i] > (116+offset0));

int rt0_0_30 = (input[i] > (121+offset0));
int t0_0_30 = rt0_0_30 * (2 + (input[i] > (122+offset0)));
t0_0_30 += (1 - rt0_0_30) * (input[i] > (120+offset0));

int rt0_0_31 = (input[i] > (125+offset0));
int t0_0_31 = rt0_0_31 * (2 + (input[i] > (126+offset0)));
t0_0_31 += (1 - rt0_0_31) * (input[i] > (124+offset0));

int rt0_0_32 = (input[i] > (129+offset0));
int t0_0_32 = rt0_0_32 * (2 + (input[i] > (130+offset0)));
t0_0_32 += (1 - rt0_0_32) * (input[i] > (128+offset0));

int rt0_0_33 = (input[i] > (133+offset0));
int t0_0_33 = rt0_0_33 * (2 + (input[i] > (134+offset0)));
t0_0_33 += (1 - rt0_0_33) * (input[i] > (132+offset0));

int rt0_0_34 = (input[i] > (137+offset0));
int t0_0_34 = rt0_0_34 * (2 + (input[i] > (138+offset0)));
t0_0_34 += (1 - rt0_0_34) * (input[i] > (136+offset0));

int rt0_0_35 = (input[i] > (141+offset0));
int t0_0_35 = rt0_0_35 * (2 + (input[i] > (142+offset0)));
t0_0_35 += (1 - rt0_0_35) * (input[i] > (140+offset0));

int rt0_0_36 = (input[i] > (145+offset0));
int t0_0_36 = rt0_0_36 * (2 + (input[i] > (146+offset0)));
t0_0_36 += (1 - rt0_0_36) * (input[i] > (144+offset0));

int rt0_0_37 = (input[i] > (149+offset0));
int t0_0_37 = rt0_0_37 * (2 + (input[i] > (150+offset0)));
t0_0_37 += (1 - rt0_0_37) * (input[i] > (148+offset0));

int rt0_0_38 = (input[i] > (153+offset0));
int t0_0_38 = rt0_0_38 * (2 + (input[i] > (154+offset0)));
t0_0_38 += (1 - rt0_0_38) * (input[i] > (152+offset0));

int rt0_0_39 = (input[i] > (157+offset0));
int t0_0_39 = rt0_0_39 * (2 + (input[i] > (158+offset0)));
t0_0_39 += (1 - rt0_0_39) * (input[i] > (156+offset0));

int rt0_0_40 = (input[i] > (161+offset0));
int t0_0_40 = rt0_0_40 * (2 + (input[i] > (162+offset0)));
t0_0_40 += (1 - rt0_0_40) * (input[i] > (160+offset0));

int rt0_0_41 = (input[i] > (165+offset0));
int t0_0_41 = rt0_0_41 * (2 + (input[i] > (166+offset0)));
t0_0_41 += (1 - rt0_0_41) * (input[i] > (164+offset0));

int rt0_0_42 = (input[i] > (169+offset0));
int t0_0_42 = rt0_0_42 * (2 + (input[i] > (170+offset0)));
t0_0_42 += (1 - rt0_0_42) * (input[i] > (168+offset0));

int rt0_0_43 = (input[i] > (173+offset0));
int t0_0_43 = rt0_0_43 * (2 + (input[i] > (174+offset0)));
t0_0_43 += (1 - rt0_0_43) * (input[i] > (172+offset0));

int rt0_0_44 = (input[i] > (177+offset0));
int t0_0_44 = rt0_0_44 * (2 + (input[i] > (178+offset0)));
t0_0_44 += (1 - rt0_0_44) * (input[i] > (176+offset0));

int rt0_0_45 = (input[i] > (181+offset0));
int t0_0_45 = rt0_0_45 * (2 + (input[i] > (182+offset0)));
t0_0_45 += (1 - rt0_0_45) * (input[i] > (180+offset0));

int rt0_0_46 = (input[i] > (185+offset0));
int t0_0_46 = rt0_0_46 * (2 + (input[i] > (186+offset0)));
t0_0_46 += (1 - rt0_0_46) * (input[i] > (184+offset0));

int rt0_0_47 = (input[i] > (189+offset0));
int t0_0_47 = rt0_0_47 * (2 + (input[i] > (190+offset0)));
t0_0_47 += (1 - rt0_0_47) * (input[i] > (188+offset0));

int rt0_0_48 = (input[i] > (193+offset0));
int t0_0_48 = rt0_0_48 * (2 + (input[i] > (194+offset0)));
t0_0_48 += (1 - rt0_0_48) * (input[i] > (192+offset0));

int rt0_0_49 = (input[i] > (197+offset0));
int t0_0_49 = rt0_0_49 * (2 + (input[i] > (198+offset0)));
t0_0_49 += (1 - rt0_0_49) * (input[i] > (196+offset0));

int rt0_0_50 = (input[i] > (201+offset0));
int t0_0_50 = rt0_0_50 * (2 + (input[i] > (202+offset0)));
t0_0_50 += (1 - rt0_0_50) * (input[i] > (200+offset0));

int rt0_0_51 = (input[i] > (205+offset0));
int t0_0_51 = rt0_0_51 * (2 + (input[i] > (206+offset0)));
t0_0_51 += (1 - rt0_0_51) * (input[i] > (204+offset0));

int rt0_0_52 = (input[i] > (209+offset0));
int t0_0_52 = rt0_0_52 * (2 + (input[i] > (210+offset0)));
t0_0_52 += (1 - rt0_0_52) * (input[i] > (208+offset0));

int rt0_0_53 = (input[i] > (213+offset0));
int t0_0_53 = rt0_0_53 * (2 + (input[i] > (214+offset0)));
t0_0_53 += (1 - rt0_0_53) * (input[i] > (212+offset0));

int rt0_0_54 = (input[i] > (217+offset0));
int t0_0_54 = rt0_0_54 * (2 + (input[i] > (218+offset0)));
t0_0_54 += (1 - rt0_0_54) * (input[i] > (216+offset0));

int rt0_0_55 = (input[i] > (221+offset0));
int t0_0_55 = rt0_0_55 * (2 + (input[i] > (222+offset0)));
t0_0_55 += (1 - rt0_0_55) * (input[i] > (220+offset0));

int rt0_0_56 = (input[i] > (225+offset0));
int t0_0_56 = rt0_0_56 * (2 + (input[i] > (226+offset0)));
t0_0_56 += (1 - rt0_0_56) * (input[i] > (224+offset0));

int rt0_0_57 = (input[i] > (229+offset0));
int t0_0_57 = rt0_0_57 * (2 + (input[i] > (230+offset0)));
t0_0_57 += (1 - rt0_0_57) * (input[i] > (228+offset0));

int rt0_0_58 = (input[i] > (233+offset0));
int t0_0_58 = rt0_0_58 * (2 + (input[i] > (234+offset0)));
t0_0_58 += (1 - rt0_0_58) * (input[i] > (232+offset0));

int rt0_0_59 = (input[i] > (237+offset0));
int t0_0_59 = rt0_0_59 * (2 + (input[i] > (238+offset0)));
t0_0_59 += (1 - rt0_0_59) * (input[i] > (236+offset0));

int rt0_0_60 = (input[i] > (241+offset0));
int t0_0_60 = rt0_0_60 * (2 + (input[i] > (242+offset0)));
t0_0_60 += (1 - rt0_0_60) * (input[i] > (240+offset0));

int rt0_0_61 = (input[i] > (245+offset0));
int t0_0_61 = rt0_0_61 * (2 + (input[i] > (246+offset0)));
t0_0_61 += (1 - rt0_0_61) * (input[i] > (244+offset0));

int rt0_0_62 = (input[i] > (249+offset0));
int t0_0_62 = rt0_0_62 * (2 + (input[i] > (250+offset0)));
t0_0_62 += (1 - rt0_0_62) * (input[i] > (248+offset0));

int rt0_0_63 = (input[i] > (253+offset0));
int t0_0_63 = rt0_0_63 * (2 + (input[i] > (254+offset0)));
t0_0_63 += (1 - rt0_0_63) * (input[i] > (252+offset0));

int rt0_0_64 = (input[i] > (257+offset0));
int t0_0_64 = rt0_0_64 * (2 + (input[i] > (258+offset0)));
t0_0_64 += (1 - rt0_0_64) * (input[i] > (256+offset0));

int rt0_0_65 = (input[i] > (261+offset0));
int t0_0_65 = rt0_0_65 * (2 + (input[i] > (262+offset0)));
t0_0_65 += (1 - rt0_0_65) * (input[i] > (260+offset0));

int rt0_0_66 = (input[i] > (265+offset0));
int t0_0_66 = rt0_0_66 * (2 + (input[i] > (266+offset0)));
t0_0_66 += (1 - rt0_0_66) * (input[i] > (264+offset0));

int rt0_0_67 = (input[i] > (269+offset0));
int t0_0_67 = rt0_0_67 * (2 + (input[i] > (270+offset0)));
t0_0_67 += (1 - rt0_0_67) * (input[i] > (268+offset0));

int rt0_0_68 = (input[i] > (273+offset0));
int t0_0_68 = rt0_0_68 * (2 + (input[i] > (274+offset0)));
t0_0_68 += (1 - rt0_0_68) * (input[i] > (272+offset0));

int rt0_0_69 = (input[i] > (277+offset0));
int t0_0_69 = rt0_0_69 * (2 + (input[i] > (278+offset0)));
t0_0_69 += (1 - rt0_0_69) * (input[i] > (276+offset0));

int rt0_0_70 = (input[i] > (281+offset0));
int t0_0_70 = rt0_0_70 * (2 + (input[i] > (282+offset0)));
t0_0_70 += (1 - rt0_0_70) * (input[i] > (280+offset0));

int rt0_0_71 = (input[i] > (285+offset0));
int t0_0_71 = rt0_0_71 * (2 + (input[i] > (286+offset0)));
t0_0_71 += (1 - rt0_0_71) * (input[i] > (284+offset0));

int rt0_0_72 = (input[i] > (289+offset0));
int t0_0_72 = rt0_0_72 * (2 + (input[i] > (290+offset0)));
t0_0_72 += (1 - rt0_0_72) * (input[i] > (288+offset0));

int rt0_0_73 = (input[i] > (293+offset0));
int t0_0_73 = rt0_0_73 * (2 + (input[i] > (294+offset0)));
t0_0_73 += (1 - rt0_0_73) * (input[i] > (292+offset0));

int rt0_0_74 = (input[i] > (297+offset0));
int t0_0_74 = rt0_0_74 * (2 + (input[i] > (298+offset0)));
t0_0_74 += (1 - rt0_0_74) * (input[i] > (296+offset0));

int rt0_0_75 = (input[i] > (301+offset0));
int t0_0_75 = rt0_0_75 * (2 + (input[i] > (302+offset0)));
t0_0_75 += (1 - rt0_0_75) * (input[i] > (300+offset0));

int rt0_0_76 = (input[i] > (305+offset0));
int t0_0_76 = rt0_0_76 * (2 + (input[i] > (306+offset0)));
t0_0_76 += (1 - rt0_0_76) * (input[i] > (304+offset0));

int rt0_0_77 = (input[i] > (309+offset0));
int t0_0_77 = rt0_0_77 * (2 + (input[i] > (310+offset0)));
t0_0_77 += (1 - rt0_0_77) * (input[i] > (308+offset0));

int rt0_0_78 = (input[i] > (313+offset0));
int t0_0_78 = rt0_0_78 * (2 + (input[i] > (314+offset0)));
t0_0_78 += (1 - rt0_0_78) * (input[i] > (312+offset0));

int rt0_0_79 = (input[i] > (317+offset0));
int t0_0_79 = rt0_0_79 * (2 + (input[i] > (318+offset0)));
t0_0_79 += (1 - rt0_0_79) * (input[i] > (316+offset0));

int rt0_0_80 = (input[i] > (321+offset0));
int t0_0_80 = rt0_0_80 * (2 + (input[i] > (322+offset0)));
t0_0_80 += (1 - rt0_0_80) * (input[i] > (320+offset0));

int rt0_0_81 = (input[i] > (325+offset0));
int t0_0_81 = rt0_0_81 * (2 + (input[i] > (326+offset0)));
t0_0_81 += (1 - rt0_0_81) * (input[i] > (324+offset0));

int rt0_0_82 = (input[i] > (329+offset0));
int t0_0_82 = rt0_0_82 * (2 + (input[i] > (330+offset0)));
t0_0_82 += (1 - rt0_0_82) * (input[i] > (328+offset0));

int rt0_0_83 = (input[i] > (333+offset0));
int t0_0_83 = rt0_0_83 * (2 + (input[i] > (334+offset0)));
t0_0_83 += (1 - rt0_0_83) * (input[i] > (332+offset0));

int rt0_0_84 = (input[i] > (337+offset0));
int t0_0_84 = rt0_0_84 * (2 + (input[i] > (338+offset0)));
t0_0_84 += (1 - rt0_0_84) * (input[i] > (336+offset0));

int rt0_0_85 = (input[i] > (341+offset0));
int t0_0_85 = rt0_0_85 * (2 + (input[i] > (342+offset0)));
t0_0_85 += (1 - rt0_0_85) * (input[i] > (340+offset0));

int rt0_0_86 = (input[i] > (345+offset0));
int t0_0_86 = rt0_0_86 * (2 + (input[i] > (346+offset0)));
t0_0_86 += (1 - rt0_0_86) * (input[i] > (344+offset0));

int rt0_0_87 = (input[i] > (349+offset0));
int t0_0_87 = rt0_0_87 * (2 + (input[i] > (350+offset0)));
t0_0_87 += (1 - rt0_0_87) * (input[i] > (348+offset0));

int rt0_0_88 = (input[i] > (353+offset0));
int t0_0_88 = rt0_0_88 * (2 + (input[i] > (354+offset0)));
t0_0_88 += (1 - rt0_0_88) * (input[i] > (352+offset0));

int rt0_0_89 = (input[i] > (357+offset0));
int t0_0_89 = rt0_0_89 * (2 + (input[i] > (358+offset0)));
t0_0_89 += (1 - rt0_0_89) * (input[i] > (356+offset0));

int rt0_0_90 = (input[i] > (361+offset0));
int t0_0_90 = rt0_0_90 * (2 + (input[i] > (362+offset0)));
t0_0_90 += (1 - rt0_0_90) * (input[i] > (360+offset0));

int rt0_0_91 = (input[i] > (365+offset0));
int t0_0_91 = rt0_0_91 * (2 + (input[i] > (366+offset0)));
t0_0_91 += (1 - rt0_0_91) * (input[i] > (364+offset0));

int rt0_0_92 = (input[i] > (369+offset0));
int t0_0_92 = rt0_0_92 * (2 + (input[i] > (370+offset0)));
t0_0_92 += (1 - rt0_0_92) * (input[i] > (368+offset0));

int rt0_0_93 = (input[i] > (373+offset0));
int t0_0_93 = rt0_0_93 * (2 + (input[i] > (374+offset0)));
t0_0_93 += (1 - rt0_0_93) * (input[i] > (372+offset0));

int rt0_0_94 = (input[i] > (377+offset0));
int t0_0_94 = rt0_0_94 * (2 + (input[i] > (378+offset0)));
t0_0_94 += (1 - rt0_0_94) * (input[i] > (376+offset0));

int rt0_0_95 = (input[i] > (381+offset0));
int t0_0_95 = rt0_0_95 * (2 + (input[i] > (382+offset0)));
t0_0_95 += (1 - rt0_0_95) * (input[i] > (380+offset0));

int rt0_0_96 = (input[i] > (385+offset0));
int t0_0_96 = rt0_0_96 * (2 + (input[i] > (386+offset0)));
t0_0_96 += (1 - rt0_0_96) * (input[i] > (384+offset0));

int rt0_0_97 = (input[i] > (389+offset0));
int t0_0_97 = rt0_0_97 * (2 + (input[i] > (390+offset0)));
t0_0_97 += (1 - rt0_0_97) * (input[i] > (388+offset0));

int rt0_0_98 = (input[i] > (393+offset0));
int t0_0_98 = rt0_0_98 * (2 + (input[i] > (394+offset0)));
t0_0_98 += (1 - rt0_0_98) * (input[i] > (392+offset0));

int rt0_0_99 = (input[i] > (397+offset0));
int t0_0_99 = rt0_0_99 * (2 + (input[i] > (398+offset0)));
t0_0_99 += (1 - rt0_0_99) * (input[i] > (396+offset0));

int rt0_0_100 = (input[i] > (401+offset0));
int t0_0_100 = rt0_0_100 * (2 + (input[i] > (402+offset0)));
t0_0_100 += (1 - rt0_0_100) * (input[i] > (400+offset0));

int rt0_0_101 = (input[i] > (405+offset0));
int t0_0_101 = rt0_0_101 * (2 + (input[i] > (406+offset0)));
t0_0_101 += (1 - rt0_0_101) * (input[i] > (404+offset0));

int rt0_0_102 = (input[i] > (409+offset0));
int t0_0_102 = rt0_0_102 * (2 + (input[i] > (410+offset0)));
t0_0_102 += (1 - rt0_0_102) * (input[i] > (408+offset0));

int rt0_0_103 = (input[i] > (413+offset0));
int t0_0_103 = rt0_0_103 * (2 + (input[i] > (414+offset0)));
t0_0_103 += (1 - rt0_0_103) * (input[i] > (412+offset0));

int rt0_0_104 = (input[i] > (417+offset0));
int t0_0_104 = rt0_0_104 * (2 + (input[i] > (418+offset0)));
t0_0_104 += (1 - rt0_0_104) * (input[i] > (416+offset0));

int rt0_0_105 = (input[i] > (421+offset0));
int t0_0_105 = rt0_0_105 * (2 + (input[i] > (422+offset0)));
t0_0_105 += (1 - rt0_0_105) * (input[i] > (420+offset0));

int rt0_0_106 = (input[i] > (425+offset0));
int t0_0_106 = rt0_0_106 * (2 + (input[i] > (426+offset0)));
t0_0_106 += (1 - rt0_0_106) * (input[i] > (424+offset0));

int rt0_0_107 = (input[i] > (429+offset0));
int t0_0_107 = rt0_0_107 * (2 + (input[i] > (430+offset0)));
t0_0_107 += (1 - rt0_0_107) * (input[i] > (428+offset0));

int rt0_0_108 = (input[i] > (433+offset0));
int t0_0_108 = rt0_0_108 * (2 + (input[i] > (434+offset0)));
t0_0_108 += (1 - rt0_0_108) * (input[i] > (432+offset0));

int rt0_0_109 = (input[i] > (437+offset0));
int t0_0_109 = rt0_0_109 * (2 + (input[i] > (438+offset0)));
t0_0_109 += (1 - rt0_0_109) * (input[i] > (436+offset0));

int rt0_0_110 = (input[i] > (441+offset0));
int t0_0_110 = rt0_0_110 * (2 + (input[i] > (442+offset0)));
t0_0_110 += (1 - rt0_0_110) * (input[i] > (440+offset0));

int rt0_0_111 = (input[i] > (445+offset0));
int t0_0_111 = rt0_0_111 * (2 + (input[i] > (446+offset0)));
t0_0_111 += (1 - rt0_0_111) * (input[i] > (444+offset0));

int rt0_0_112 = (input[i] > (449+offset0));
int t0_0_112 = rt0_0_112 * (2 + (input[i] > (450+offset0)));
t0_0_112 += (1 - rt0_0_112) * (input[i] > (448+offset0));

int rt0_0_113 = (input[i] > (453+offset0));
int t0_0_113 = rt0_0_113 * (2 + (input[i] > (454+offset0)));
t0_0_113 += (1 - rt0_0_113) * (input[i] > (452+offset0));

int rt0_0_114 = (input[i] > (457+offset0));
int t0_0_114 = rt0_0_114 * (2 + (input[i] > (458+offset0)));
t0_0_114 += (1 - rt0_0_114) * (input[i] > (456+offset0));

int rt0_0_115 = (input[i] > (461+offset0));
int t0_0_115 = rt0_0_115 * (2 + (input[i] > (462+offset0)));
t0_0_115 += (1 - rt0_0_115) * (input[i] > (460+offset0));

int rt0_0_116 = (input[i] > (465+offset0));
int t0_0_116 = rt0_0_116 * (2 + (input[i] > (466+offset0)));
t0_0_116 += (1 - rt0_0_116) * (input[i] > (464+offset0));

int rt0_0_117 = (input[i] > (469+offset0));
int t0_0_117 = rt0_0_117 * (2 + (input[i] > (470+offset0)));
t0_0_117 += (1 - rt0_0_117) * (input[i] > (468+offset0));

int rt0_0_118 = (input[i] > (473+offset0));
int t0_0_118 = rt0_0_118 * (2 + (input[i] > (474+offset0)));
t0_0_118 += (1 - rt0_0_118) * (input[i] > (472+offset0));

int rt0_0_119 = (input[i] > (477+offset0));
int t0_0_119 = rt0_0_119 * (2 + (input[i] > (478+offset0)));
t0_0_119 += (1 - rt0_0_119) * (input[i] > (476+offset0));

int rt0_0_120 = (input[i] > (481+offset0));
int t0_0_120 = rt0_0_120 * (2 + (input[i] > (482+offset0)));
t0_0_120 += (1 - rt0_0_120) * (input[i] > (480+offset0));

int rt0_0_121 = (input[i] > (485+offset0));
int t0_0_121 = rt0_0_121 * (2 + (input[i] > (486+offset0)));
t0_0_121 += (1 - rt0_0_121) * (input[i] > (484+offset0));

int rt0_0_122 = (input[i] > (489+offset0));
int t0_0_122 = rt0_0_122 * (2 + (input[i] > (490+offset0)));
t0_0_122 += (1 - rt0_0_122) * (input[i] > (488+offset0));

int rt0_0_123 = (input[i] > (493+offset0));
int t0_0_123 = rt0_0_123 * (2 + (input[i] > (494+offset0)));
t0_0_123 += (1 - rt0_0_123) * (input[i] > (492+offset0));

int rt0_0_124 = (input[i] > (497+offset0));
int t0_0_124 = rt0_0_124 * (2 + (input[i] > (498+offset0)));
t0_0_124 += (1 - rt0_0_124) * (input[i] > (496+offset0));

int rt0_0_125 = (input[i] > (501+offset0));
int t0_0_125 = rt0_0_125 * (2 + (input[i] > (502+offset0)));
t0_0_125 += (1 - rt0_0_125) * (input[i] > (500+offset0));

int rt0_0_126 = (input[i] > (505+offset0));
int t0_0_126 = rt0_0_126 * (2 + (input[i] > (506+offset0)));
t0_0_126 += (1 - rt0_0_126) * (input[i] > (504+offset0));

int rt0_0_127 = (input[i] > (509+offset0));
int t0_0_127 = rt0_0_127 * (2 + (input[i] > (510+offset0)));
t0_0_127 += (1 - rt0_0_127) * (input[i] > (508+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_1_8 = (input[i] > (67+offset0));
int t0_1_8 = rt0_1_8 * (4 + t0_0_17);
t0_1_8 += (1 - rt0_1_8) * t0_0_16;

int rt0_1_9 = (input[i] > (75+offset0));
int t0_1_9 = rt0_1_9 * (4 + t0_0_19);
t0_1_9 += (1 - rt0_1_9) * t0_0_18;

int rt0_1_10 = (input[i] > (83+offset0));
int t0_1_10 = rt0_1_10 * (4 + t0_0_21);
t0_1_10 += (1 - rt0_1_10) * t0_0_20;

int rt0_1_11 = (input[i] > (91+offset0));
int t0_1_11 = rt0_1_11 * (4 + t0_0_23);
t0_1_11 += (1 - rt0_1_11) * t0_0_22;

int rt0_1_12 = (input[i] > (99+offset0));
int t0_1_12 = rt0_1_12 * (4 + t0_0_25);
t0_1_12 += (1 - rt0_1_12) * t0_0_24;

int rt0_1_13 = (input[i] > (107+offset0));
int t0_1_13 = rt0_1_13 * (4 + t0_0_27);
t0_1_13 += (1 - rt0_1_13) * t0_0_26;

int rt0_1_14 = (input[i] > (115+offset0));
int t0_1_14 = rt0_1_14 * (4 + t0_0_29);
t0_1_14 += (1 - rt0_1_14) * t0_0_28;

int rt0_1_15 = (input[i] > (123+offset0));
int t0_1_15 = rt0_1_15 * (4 + t0_0_31);
t0_1_15 += (1 - rt0_1_15) * t0_0_30;

int rt0_1_16 = (input[i] > (131+offset0));
int t0_1_16 = rt0_1_16 * (4 + t0_0_33);
t0_1_16 += (1 - rt0_1_16) * t0_0_32;

int rt0_1_17 = (input[i] > (139+offset0));
int t0_1_17 = rt0_1_17 * (4 + t0_0_35);
t0_1_17 += (1 - rt0_1_17) * t0_0_34;

int rt0_1_18 = (input[i] > (147+offset0));
int t0_1_18 = rt0_1_18 * (4 + t0_0_37);
t0_1_18 += (1 - rt0_1_18) * t0_0_36;

int rt0_1_19 = (input[i] > (155+offset0));
int t0_1_19 = rt0_1_19 * (4 + t0_0_39);
t0_1_19 += (1 - rt0_1_19) * t0_0_38;

int rt0_1_20 = (input[i] > (163+offset0));
int t0_1_20 = rt0_1_20 * (4 + t0_0_41);
t0_1_20 += (1 - rt0_1_20) * t0_0_40;

int rt0_1_21 = (input[i] > (171+offset0));
int t0_1_21 = rt0_1_21 * (4 + t0_0_43);
t0_1_21 += (1 - rt0_1_21) * t0_0_42;

int rt0_1_22 = (input[i] > (179+offset0));
int t0_1_22 = rt0_1_22 * (4 + t0_0_45);
t0_1_22 += (1 - rt0_1_22) * t0_0_44;

int rt0_1_23 = (input[i] > (187+offset0));
int t0_1_23 = rt0_1_23 * (4 + t0_0_47);
t0_1_23 += (1 - rt0_1_23) * t0_0_46;

int rt0_1_24 = (input[i] > (195+offset0));
int t0_1_24 = rt0_1_24 * (4 + t0_0_49);
t0_1_24 += (1 - rt0_1_24) * t0_0_48;

int rt0_1_25 = (input[i] > (203+offset0));
int t0_1_25 = rt0_1_25 * (4 + t0_0_51);
t0_1_25 += (1 - rt0_1_25) * t0_0_50;

int rt0_1_26 = (input[i] > (211+offset0));
int t0_1_26 = rt0_1_26 * (4 + t0_0_53);
t0_1_26 += (1 - rt0_1_26) * t0_0_52;

int rt0_1_27 = (input[i] > (219+offset0));
int t0_1_27 = rt0_1_27 * (4 + t0_0_55);
t0_1_27 += (1 - rt0_1_27) * t0_0_54;

int rt0_1_28 = (input[i] > (227+offset0));
int t0_1_28 = rt0_1_28 * (4 + t0_0_57);
t0_1_28 += (1 - rt0_1_28) * t0_0_56;

int rt0_1_29 = (input[i] > (235+offset0));
int t0_1_29 = rt0_1_29 * (4 + t0_0_59);
t0_1_29 += (1 - rt0_1_29) * t0_0_58;

int rt0_1_30 = (input[i] > (243+offset0));
int t0_1_30 = rt0_1_30 * (4 + t0_0_61);
t0_1_30 += (1 - rt0_1_30) * t0_0_60;

int rt0_1_31 = (input[i] > (251+offset0));
int t0_1_31 = rt0_1_31 * (4 + t0_0_63);
t0_1_31 += (1 - rt0_1_31) * t0_0_62;

int rt0_1_32 = (input[i] > (259+offset0));
int t0_1_32 = rt0_1_32 * (4 + t0_0_65);
t0_1_32 += (1 - rt0_1_32) * t0_0_64;

int rt0_1_33 = (input[i] > (267+offset0));
int t0_1_33 = rt0_1_33 * (4 + t0_0_67);
t0_1_33 += (1 - rt0_1_33) * t0_0_66;

int rt0_1_34 = (input[i] > (275+offset0));
int t0_1_34 = rt0_1_34 * (4 + t0_0_69);
t0_1_34 += (1 - rt0_1_34) * t0_0_68;

int rt0_1_35 = (input[i] > (283+offset0));
int t0_1_35 = rt0_1_35 * (4 + t0_0_71);
t0_1_35 += (1 - rt0_1_35) * t0_0_70;

int rt0_1_36 = (input[i] > (291+offset0));
int t0_1_36 = rt0_1_36 * (4 + t0_0_73);
t0_1_36 += (1 - rt0_1_36) * t0_0_72;

int rt0_1_37 = (input[i] > (299+offset0));
int t0_1_37 = rt0_1_37 * (4 + t0_0_75);
t0_1_37 += (1 - rt0_1_37) * t0_0_74;

int rt0_1_38 = (input[i] > (307+offset0));
int t0_1_38 = rt0_1_38 * (4 + t0_0_77);
t0_1_38 += (1 - rt0_1_38) * t0_0_76;

int rt0_1_39 = (input[i] > (315+offset0));
int t0_1_39 = rt0_1_39 * (4 + t0_0_79);
t0_1_39 += (1 - rt0_1_39) * t0_0_78;

int rt0_1_40 = (input[i] > (323+offset0));
int t0_1_40 = rt0_1_40 * (4 + t0_0_81);
t0_1_40 += (1 - rt0_1_40) * t0_0_80;

int rt0_1_41 = (input[i] > (331+offset0));
int t0_1_41 = rt0_1_41 * (4 + t0_0_83);
t0_1_41 += (1 - rt0_1_41) * t0_0_82;

int rt0_1_42 = (input[i] > (339+offset0));
int t0_1_42 = rt0_1_42 * (4 + t0_0_85);
t0_1_42 += (1 - rt0_1_42) * t0_0_84;

int rt0_1_43 = (input[i] > (347+offset0));
int t0_1_43 = rt0_1_43 * (4 + t0_0_87);
t0_1_43 += (1 - rt0_1_43) * t0_0_86;

int rt0_1_44 = (input[i] > (355+offset0));
int t0_1_44 = rt0_1_44 * (4 + t0_0_89);
t0_1_44 += (1 - rt0_1_44) * t0_0_88;

int rt0_1_45 = (input[i] > (363+offset0));
int t0_1_45 = rt0_1_45 * (4 + t0_0_91);
t0_1_45 += (1 - rt0_1_45) * t0_0_90;

int rt0_1_46 = (input[i] > (371+offset0));
int t0_1_46 = rt0_1_46 * (4 + t0_0_93);
t0_1_46 += (1 - rt0_1_46) * t0_0_92;

int rt0_1_47 = (input[i] > (379+offset0));
int t0_1_47 = rt0_1_47 * (4 + t0_0_95);
t0_1_47 += (1 - rt0_1_47) * t0_0_94;

int rt0_1_48 = (input[i] > (387+offset0));
int t0_1_48 = rt0_1_48 * (4 + t0_0_97);
t0_1_48 += (1 - rt0_1_48) * t0_0_96;

int rt0_1_49 = (input[i] > (395+offset0));
int t0_1_49 = rt0_1_49 * (4 + t0_0_99);
t0_1_49 += (1 - rt0_1_49) * t0_0_98;

int rt0_1_50 = (input[i] > (403+offset0));
int t0_1_50 = rt0_1_50 * (4 + t0_0_101);
t0_1_50 += (1 - rt0_1_50) * t0_0_100;

int rt0_1_51 = (input[i] > (411+offset0));
int t0_1_51 = rt0_1_51 * (4 + t0_0_103);
t0_1_51 += (1 - rt0_1_51) * t0_0_102;

int rt0_1_52 = (input[i] > (419+offset0));
int t0_1_52 = rt0_1_52 * (4 + t0_0_105);
t0_1_52 += (1 - rt0_1_52) * t0_0_104;

int rt0_1_53 = (input[i] > (427+offset0));
int t0_1_53 = rt0_1_53 * (4 + t0_0_107);
t0_1_53 += (1 - rt0_1_53) * t0_0_106;

int rt0_1_54 = (input[i] > (435+offset0));
int t0_1_54 = rt0_1_54 * (4 + t0_0_109);
t0_1_54 += (1 - rt0_1_54) * t0_0_108;

int rt0_1_55 = (input[i] > (443+offset0));
int t0_1_55 = rt0_1_55 * (4 + t0_0_111);
t0_1_55 += (1 - rt0_1_55) * t0_0_110;

int rt0_1_56 = (input[i] > (451+offset0));
int t0_1_56 = rt0_1_56 * (4 + t0_0_113);
t0_1_56 += (1 - rt0_1_56) * t0_0_112;

int rt0_1_57 = (input[i] > (459+offset0));
int t0_1_57 = rt0_1_57 * (4 + t0_0_115);
t0_1_57 += (1 - rt0_1_57) * t0_0_114;

int rt0_1_58 = (input[i] > (467+offset0));
int t0_1_58 = rt0_1_58 * (4 + t0_0_117);
t0_1_58 += (1 - rt0_1_58) * t0_0_116;

int rt0_1_59 = (input[i] > (475+offset0));
int t0_1_59 = rt0_1_59 * (4 + t0_0_119);
t0_1_59 += (1 - rt0_1_59) * t0_0_118;

int rt0_1_60 = (input[i] > (483+offset0));
int t0_1_60 = rt0_1_60 * (4 + t0_0_121);
t0_1_60 += (1 - rt0_1_60) * t0_0_120;

int rt0_1_61 = (input[i] > (491+offset0));
int t0_1_61 = rt0_1_61 * (4 + t0_0_123);
t0_1_61 += (1 - rt0_1_61) * t0_0_122;

int rt0_1_62 = (input[i] > (499+offset0));
int t0_1_62 = rt0_1_62 * (4 + t0_0_125);
t0_1_62 += (1 - rt0_1_62) * t0_0_124;

int rt0_1_63 = (input[i] > (507+offset0));
int t0_1_63 = rt0_1_63 * (4 + t0_0_127);
t0_1_63 += (1 - rt0_1_63) * t0_0_126;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_2_4 = (input[i] > (71+offset0));
int t0_2_4 = rt0_2_4 * (8 + t0_1_9);
t0_2_4 += (1 - rt0_2_4) * t0_1_8;

int rt0_2_5 = (input[i] > (87+offset0));
int t0_2_5 = rt0_2_5 * (8 + t0_1_11);
t0_2_5 += (1 - rt0_2_5) * t0_1_10;

int rt0_2_6 = (input[i] > (103+offset0));
int t0_2_6 = rt0_2_6 * (8 + t0_1_13);
t0_2_6 += (1 - rt0_2_6) * t0_1_12;

int rt0_2_7 = (input[i] > (119+offset0));
int t0_2_7 = rt0_2_7 * (8 + t0_1_15);
t0_2_7 += (1 - rt0_2_7) * t0_1_14;

int rt0_2_8 = (input[i] > (135+offset0));
int t0_2_8 = rt0_2_8 * (8 + t0_1_17);
t0_2_8 += (1 - rt0_2_8) * t0_1_16;

int rt0_2_9 = (input[i] > (151+offset0));
int t0_2_9 = rt0_2_9 * (8 + t0_1_19);
t0_2_9 += (1 - rt0_2_9) * t0_1_18;

int rt0_2_10 = (input[i] > (167+offset0));
int t0_2_10 = rt0_2_10 * (8 + t0_1_21);
t0_2_10 += (1 - rt0_2_10) * t0_1_20;

int rt0_2_11 = (input[i] > (183+offset0));
int t0_2_11 = rt0_2_11 * (8 + t0_1_23);
t0_2_11 += (1 - rt0_2_11) * t0_1_22;

int rt0_2_12 = (input[i] > (199+offset0));
int t0_2_12 = rt0_2_12 * (8 + t0_1_25);
t0_2_12 += (1 - rt0_2_12) * t0_1_24;

int rt0_2_13 = (input[i] > (215+offset0));
int t0_2_13 = rt0_2_13 * (8 + t0_1_27);
t0_2_13 += (1 - rt0_2_13) * t0_1_26;

int rt0_2_14 = (input[i] > (231+offset0));
int t0_2_14 = rt0_2_14 * (8 + t0_1_29);
t0_2_14 += (1 - rt0_2_14) * t0_1_28;

int rt0_2_15 = (input[i] > (247+offset0));
int t0_2_15 = rt0_2_15 * (8 + t0_1_31);
t0_2_15 += (1 - rt0_2_15) * t0_1_30;

int rt0_2_16 = (input[i] > (263+offset0));
int t0_2_16 = rt0_2_16 * (8 + t0_1_33);
t0_2_16 += (1 - rt0_2_16) * t0_1_32;

int rt0_2_17 = (input[i] > (279+offset0));
int t0_2_17 = rt0_2_17 * (8 + t0_1_35);
t0_2_17 += (1 - rt0_2_17) * t0_1_34;

int rt0_2_18 = (input[i] > (295+offset0));
int t0_2_18 = rt0_2_18 * (8 + t0_1_37);
t0_2_18 += (1 - rt0_2_18) * t0_1_36;

int rt0_2_19 = (input[i] > (311+offset0));
int t0_2_19 = rt0_2_19 * (8 + t0_1_39);
t0_2_19 += (1 - rt0_2_19) * t0_1_38;

int rt0_2_20 = (input[i] > (327+offset0));
int t0_2_20 = rt0_2_20 * (8 + t0_1_41);
t0_2_20 += (1 - rt0_2_20) * t0_1_40;

int rt0_2_21 = (input[i] > (343+offset0));
int t0_2_21 = rt0_2_21 * (8 + t0_1_43);
t0_2_21 += (1 - rt0_2_21) * t0_1_42;

int rt0_2_22 = (input[i] > (359+offset0));
int t0_2_22 = rt0_2_22 * (8 + t0_1_45);
t0_2_22 += (1 - rt0_2_22) * t0_1_44;

int rt0_2_23 = (input[i] > (375+offset0));
int t0_2_23 = rt0_2_23 * (8 + t0_1_47);
t0_2_23 += (1 - rt0_2_23) * t0_1_46;

int rt0_2_24 = (input[i] > (391+offset0));
int t0_2_24 = rt0_2_24 * (8 + t0_1_49);
t0_2_24 += (1 - rt0_2_24) * t0_1_48;

int rt0_2_25 = (input[i] > (407+offset0));
int t0_2_25 = rt0_2_25 * (8 + t0_1_51);
t0_2_25 += (1 - rt0_2_25) * t0_1_50;

int rt0_2_26 = (input[i] > (423+offset0));
int t0_2_26 = rt0_2_26 * (8 + t0_1_53);
t0_2_26 += (1 - rt0_2_26) * t0_1_52;

int rt0_2_27 = (input[i] > (439+offset0));
int t0_2_27 = rt0_2_27 * (8 + t0_1_55);
t0_2_27 += (1 - rt0_2_27) * t0_1_54;

int rt0_2_28 = (input[i] > (455+offset0));
int t0_2_28 = rt0_2_28 * (8 + t0_1_57);
t0_2_28 += (1 - rt0_2_28) * t0_1_56;

int rt0_2_29 = (input[i] > (471+offset0));
int t0_2_29 = rt0_2_29 * (8 + t0_1_59);
t0_2_29 += (1 - rt0_2_29) * t0_1_58;

int rt0_2_30 = (input[i] > (487+offset0));
int t0_2_30 = rt0_2_30 * (8 + t0_1_61);
t0_2_30 += (1 - rt0_2_30) * t0_1_60;

int rt0_2_31 = (input[i] > (503+offset0));
int t0_2_31 = rt0_2_31 * (8 + t0_1_63);
t0_2_31 += (1 - rt0_2_31) * t0_1_62;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_3_2 = (input[i] > (79+offset0));
int t0_3_2 = rt0_3_2 * (16 + t0_2_5);
t0_3_2 += (1 - rt0_3_2) * t0_2_4;

int rt0_3_3 = (input[i] > (111+offset0));
int t0_3_3 = rt0_3_3 * (16 + t0_2_7);
t0_3_3 += (1 - rt0_3_3) * t0_2_6;

int rt0_3_4 = (input[i] > (143+offset0));
int t0_3_4 = rt0_3_4 * (16 + t0_2_9);
t0_3_4 += (1 - rt0_3_4) * t0_2_8;

int rt0_3_5 = (input[i] > (175+offset0));
int t0_3_5 = rt0_3_5 * (16 + t0_2_11);
t0_3_5 += (1 - rt0_3_5) * t0_2_10;

int rt0_3_6 = (input[i] > (207+offset0));
int t0_3_6 = rt0_3_6 * (16 + t0_2_13);
t0_3_6 += (1 - rt0_3_6) * t0_2_12;

int rt0_3_7 = (input[i] > (239+offset0));
int t0_3_7 = rt0_3_7 * (16 + t0_2_15);
t0_3_7 += (1 - rt0_3_7) * t0_2_14;

int rt0_3_8 = (input[i] > (271+offset0));
int t0_3_8 = rt0_3_8 * (16 + t0_2_17);
t0_3_8 += (1 - rt0_3_8) * t0_2_16;

int rt0_3_9 = (input[i] > (303+offset0));
int t0_3_9 = rt0_3_9 * (16 + t0_2_19);
t0_3_9 += (1 - rt0_3_9) * t0_2_18;

int rt0_3_10 = (input[i] > (335+offset0));
int t0_3_10 = rt0_3_10 * (16 + t0_2_21);
t0_3_10 += (1 - rt0_3_10) * t0_2_20;

int rt0_3_11 = (input[i] > (367+offset0));
int t0_3_11 = rt0_3_11 * (16 + t0_2_23);
t0_3_11 += (1 - rt0_3_11) * t0_2_22;

int rt0_3_12 = (input[i] > (399+offset0));
int t0_3_12 = rt0_3_12 * (16 + t0_2_25);
t0_3_12 += (1 - rt0_3_12) * t0_2_24;

int rt0_3_13 = (input[i] > (431+offset0));
int t0_3_13 = rt0_3_13 * (16 + t0_2_27);
t0_3_13 += (1 - rt0_3_13) * t0_2_26;

int rt0_3_14 = (input[i] > (463+offset0));
int t0_3_14 = rt0_3_14 * (16 + t0_2_29);
t0_3_14 += (1 - rt0_3_14) * t0_2_28;

int rt0_3_15 = (input[i] > (495+offset0));
int t0_3_15 = rt0_3_15 * (16 + t0_2_31);
t0_3_15 += (1 - rt0_3_15) * t0_2_30;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt0_4_1 = (input[i] > (95+offset0));
int t0_4_1 = rt0_4_1 * (32 + t0_3_3);
t0_4_1 += (1 - rt0_4_1) * t0_3_2;

int rt0_4_2 = (input[i] > (159+offset0));
int t0_4_2 = rt0_4_2 * (32 + t0_3_5);
t0_4_2 += (1 - rt0_4_2) * t0_3_4;

int rt0_4_3 = (input[i] > (223+offset0));
int t0_4_3 = rt0_4_3 * (32 + t0_3_7);
t0_4_3 += (1 - rt0_4_3) * t0_3_6;

int rt0_4_4 = (input[i] > (287+offset0));
int t0_4_4 = rt0_4_4 * (32 + t0_3_9);
t0_4_4 += (1 - rt0_4_4) * t0_3_8;

int rt0_4_5 = (input[i] > (351+offset0));
int t0_4_5 = rt0_4_5 * (32 + t0_3_11);
t0_4_5 += (1 - rt0_4_5) * t0_3_10;

int rt0_4_6 = (input[i] > (415+offset0));
int t0_4_6 = rt0_4_6 * (32 + t0_3_13);
t0_4_6 += (1 - rt0_4_6) * t0_3_12;

int rt0_4_7 = (input[i] > (479+offset0));
int t0_4_7 = rt0_4_7 * (32 + t0_3_15);
t0_4_7 += (1 - rt0_4_7) * t0_3_14;

int rt0_5_0 = (input[i] > (63+offset0));
int t0_5_0 = rt0_5_0 * (64 + t0_4_1);
t0_5_0 += (1 - rt0_5_0) * t0_4_0;

int rt0_5_1 = (input[i] > (191+offset0));
int t0_5_1 = rt0_5_1 * (64 + t0_4_3);
t0_5_1 += (1 - rt0_5_1) * t0_4_2;

int rt0_5_2 = (input[i] > (319+offset0));
int t0_5_2 = rt0_5_2 * (64 + t0_4_5);
t0_5_2 += (1 - rt0_5_2) * t0_4_4;

int rt0_5_3 = (input[i] > (447+offset0));
int t0_5_3 = rt0_5_3 * (64 + t0_4_7);
t0_5_3 += (1 - rt0_5_3) * t0_4_6;

int rt0_6_0 = (input[i] > (127+offset0));
int t0_6_0 = rt0_6_0 * (128 + t0_5_1);
t0_6_0 += (1 - rt0_6_0) * t0_5_0;

int rt0_6_1 = (input[i] > (383+offset0));
int t0_6_1 = rt0_6_1 * (128 + t0_5_3);
t0_6_1 += (1 - rt0_6_1) * t0_5_2;

int rt0_7_0 = (input[i] > (255+offset0));
int t0_7_0 = rt0_7_0 * (256 + t0_6_1);
t0_7_0 += (1 - rt0_7_0) * t0_6_0;

int rt1_0_0 = (input[i] > (1+offset1));
int t1_0_0 = rt1_0_0 * (2 + (input[i] > (2+offset1)));
t1_0_0 += (1 - rt1_0_0) * (input[i] > (0+offset1));

int rt1_0_1 = (input[i] > (5+offset1));
int t1_0_1 = rt1_0_1 * (2 + (input[i] > (6+offset1)));
t1_0_1 += (1 - rt1_0_1) * (input[i] > (4+offset1));

int rt1_0_2 = (input[i] > (9+offset1));
int t1_0_2 = rt1_0_2 * (2 + (input[i] > (10+offset1)));
t1_0_2 += (1 - rt1_0_2) * (input[i] > (8+offset1));

int rt1_0_3 = (input[i] > (13+offset1));
int t1_0_3 = rt1_0_3 * (2 + (input[i] > (14+offset1)));
t1_0_3 += (1 - rt1_0_3) * (input[i] > (12+offset1));

int rt1_0_4 = (input[i] > (17+offset1));
int t1_0_4 = rt1_0_4 * (2 + (input[i] > (18+offset1)));
t1_0_4 += (1 - rt1_0_4) * (input[i] > (16+offset1));

int rt1_0_5 = (input[i] > (21+offset1));
int t1_0_5 = rt1_0_5 * (2 + (input[i] > (22+offset1)));
t1_0_5 += (1 - rt1_0_5) * (input[i] > (20+offset1));

int rt1_0_6 = (input[i] > (25+offset1));
int t1_0_6 = rt1_0_6 * (2 + (input[i] > (26+offset1)));
t1_0_6 += (1 - rt1_0_6) * (input[i] > (24+offset1));

int rt1_0_7 = (input[i] > (29+offset1));
int t1_0_7 = rt1_0_7 * (2 + (input[i] > (30+offset1)));
t1_0_7 += (1 - rt1_0_7) * (input[i] > (28+offset1));

int rt1_0_8 = (input[i] > (33+offset1));
int t1_0_8 = rt1_0_8 * (2 + (input[i] > (34+offset1)));
t1_0_8 += (1 - rt1_0_8) * (input[i] > (32+offset1));

int rt1_0_9 = (input[i] > (37+offset1));
int t1_0_9 = rt1_0_9 * (2 + (input[i] > (38+offset1)));
t1_0_9 += (1 - rt1_0_9) * (input[i] > (36+offset1));

int rt1_0_10 = (input[i] > (41+offset1));
int t1_0_10 = rt1_0_10 * (2 + (input[i] > (42+offset1)));
t1_0_10 += (1 - rt1_0_10) * (input[i] > (40+offset1));

int rt1_0_11 = (input[i] > (45+offset1));
int t1_0_11 = rt1_0_11 * (2 + (input[i] > (46+offset1)));
t1_0_11 += (1 - rt1_0_11) * (input[i] > (44+offset1));

int rt1_0_12 = (input[i] > (49+offset1));
int t1_0_12 = rt1_0_12 * (2 + (input[i] > (50+offset1)));
t1_0_12 += (1 - rt1_0_12) * (input[i] > (48+offset1));

int rt1_0_13 = (input[i] > (53+offset1));
int t1_0_13 = rt1_0_13 * (2 + (input[i] > (54+offset1)));
t1_0_13 += (1 - rt1_0_13) * (input[i] > (52+offset1));

int rt1_0_14 = (input[i] > (57+offset1));
int t1_0_14 = rt1_0_14 * (2 + (input[i] > (58+offset1)));
t1_0_14 += (1 - rt1_0_14) * (input[i] > (56+offset1));

int rt1_0_15 = (input[i] > (61+offset1));
int t1_0_15 = rt1_0_15 * (2 + (input[i] > (62+offset1)));
t1_0_15 += (1 - rt1_0_15) * (input[i] > (60+offset1));

int rt1_0_16 = (input[i] > (65+offset1));
int t1_0_16 = rt1_0_16 * (2 + (input[i] > (66+offset1)));
t1_0_16 += (1 - rt1_0_16) * (input[i] > (64+offset1));

int rt1_0_17 = (input[i] > (69+offset1));
int t1_0_17 = rt1_0_17 * (2 + (input[i] > (70+offset1)));
t1_0_17 += (1 - rt1_0_17) * (input[i] > (68+offset1));

int rt1_0_18 = (input[i] > (73+offset1));
int t1_0_18 = rt1_0_18 * (2 + (input[i] > (74+offset1)));
t1_0_18 += (1 - rt1_0_18) * (input[i] > (72+offset1));

int rt1_0_19 = (input[i] > (77+offset1));
int t1_0_19 = rt1_0_19 * (2 + (input[i] > (78+offset1)));
t1_0_19 += (1 - rt1_0_19) * (input[i] > (76+offset1));

int rt1_0_20 = (input[i] > (81+offset1));
int t1_0_20 = rt1_0_20 * (2 + (input[i] > (82+offset1)));
t1_0_20 += (1 - rt1_0_20) * (input[i] > (80+offset1));

int rt1_0_21 = (input[i] > (85+offset1));
int t1_0_21 = rt1_0_21 * (2 + (input[i] > (86+offset1)));
t1_0_21 += (1 - rt1_0_21) * (input[i] > (84+offset1));

int rt1_0_22 = (input[i] > (89+offset1));
int t1_0_22 = rt1_0_22 * (2 + (input[i] > (90+offset1)));
t1_0_22 += (1 - rt1_0_22) * (input[i] > (88+offset1));

int rt1_0_23 = (input[i] > (93+offset1));
int t1_0_23 = rt1_0_23 * (2 + (input[i] > (94+offset1)));
t1_0_23 += (1 - rt1_0_23) * (input[i] > (92+offset1));

int rt1_0_24 = (input[i] > (97+offset1));
int t1_0_24 = rt1_0_24 * (2 + (input[i] > (98+offset1)));
t1_0_24 += (1 - rt1_0_24) * (input[i] > (96+offset1));

int rt1_0_25 = (input[i] > (101+offset1));
int t1_0_25 = rt1_0_25 * (2 + (input[i] > (102+offset1)));
t1_0_25 += (1 - rt1_0_25) * (input[i] > (100+offset1));

int rt1_0_26 = (input[i] > (105+offset1));
int t1_0_26 = rt1_0_26 * (2 + (input[i] > (106+offset1)));
t1_0_26 += (1 - rt1_0_26) * (input[i] > (104+offset1));

int rt1_0_27 = (input[i] > (109+offset1));
int t1_0_27 = rt1_0_27 * (2 + (input[i] > (110+offset1)));
t1_0_27 += (1 - rt1_0_27) * (input[i] > (108+offset1));

int rt1_0_28 = (input[i] > (113+offset1));
int t1_0_28 = rt1_0_28 * (2 + (input[i] > (114+offset1)));
t1_0_28 += (1 - rt1_0_28) * (input[i] > (112+offset1));

int rt1_0_29 = (input[i] > (117+offset1));
int t1_0_29 = rt1_0_29 * (2 + (input[i] > (118+offset1)));
t1_0_29 += (1 - rt1_0_29) * (input[i] > (116+offset1));

int rt1_0_30 = (input[i] > (121+offset1));
int t1_0_30 = rt1_0_30 * (2 + (input[i] > (122+offset1)));
t1_0_30 += (1 - rt1_0_30) * (input[i] > (120+offset1));

int rt1_0_31 = (input[i] > (125+offset1));
int t1_0_31 = rt1_0_31 * (2 + (input[i] > (126+offset1)));
t1_0_31 += (1 - rt1_0_31) * (input[i] > (124+offset1));

int rt1_0_32 = (input[i] > (129+offset1));
int t1_0_32 = rt1_0_32 * (2 + (input[i] > (130+offset1)));
t1_0_32 += (1 - rt1_0_32) * (input[i] > (128+offset1));

int rt1_0_33 = (input[i] > (133+offset1));
int t1_0_33 = rt1_0_33 * (2 + (input[i] > (134+offset1)));
t1_0_33 += (1 - rt1_0_33) * (input[i] > (132+offset1));

int rt1_0_34 = (input[i] > (137+offset1));
int t1_0_34 = rt1_0_34 * (2 + (input[i] > (138+offset1)));
t1_0_34 += (1 - rt1_0_34) * (input[i] > (136+offset1));

int rt1_0_35 = (input[i] > (141+offset1));
int t1_0_35 = rt1_0_35 * (2 + (input[i] > (142+offset1)));
t1_0_35 += (1 - rt1_0_35) * (input[i] > (140+offset1));

int rt1_0_36 = (input[i] > (145+offset1));
int t1_0_36 = rt1_0_36 * (2 + (input[i] > (146+offset1)));
t1_0_36 += (1 - rt1_0_36) * (input[i] > (144+offset1));

int rt1_0_37 = (input[i] > (149+offset1));
int t1_0_37 = rt1_0_37 * (2 + (input[i] > (150+offset1)));
t1_0_37 += (1 - rt1_0_37) * (input[i] > (148+offset1));

int rt1_0_38 = (input[i] > (153+offset1));
int t1_0_38 = rt1_0_38 * (2 + (input[i] > (154+offset1)));
t1_0_38 += (1 - rt1_0_38) * (input[i] > (152+offset1));

int rt1_0_39 = (input[i] > (157+offset1));
int t1_0_39 = rt1_0_39 * (2 + (input[i] > (158+offset1)));
t1_0_39 += (1 - rt1_0_39) * (input[i] > (156+offset1));

int rt1_0_40 = (input[i] > (161+offset1));
int t1_0_40 = rt1_0_40 * (2 + (input[i] > (162+offset1)));
t1_0_40 += (1 - rt1_0_40) * (input[i] > (160+offset1));

int rt1_0_41 = (input[i] > (165+offset1));
int t1_0_41 = rt1_0_41 * (2 + (input[i] > (166+offset1)));
t1_0_41 += (1 - rt1_0_41) * (input[i] > (164+offset1));

int rt1_0_42 = (input[i] > (169+offset1));
int t1_0_42 = rt1_0_42 * (2 + (input[i] > (170+offset1)));
t1_0_42 += (1 - rt1_0_42) * (input[i] > (168+offset1));

int rt1_0_43 = (input[i] > (173+offset1));
int t1_0_43 = rt1_0_43 * (2 + (input[i] > (174+offset1)));
t1_0_43 += (1 - rt1_0_43) * (input[i] > (172+offset1));

int rt1_0_44 = (input[i] > (177+offset1));
int t1_0_44 = rt1_0_44 * (2 + (input[i] > (178+offset1)));
t1_0_44 += (1 - rt1_0_44) * (input[i] > (176+offset1));

int rt1_0_45 = (input[i] > (181+offset1));
int t1_0_45 = rt1_0_45 * (2 + (input[i] > (182+offset1)));
t1_0_45 += (1 - rt1_0_45) * (input[i] > (180+offset1));

int rt1_0_46 = (input[i] > (185+offset1));
int t1_0_46 = rt1_0_46 * (2 + (input[i] > (186+offset1)));
t1_0_46 += (1 - rt1_0_46) * (input[i] > (184+offset1));

int rt1_0_47 = (input[i] > (189+offset1));
int t1_0_47 = rt1_0_47 * (2 + (input[i] > (190+offset1)));
t1_0_47 += (1 - rt1_0_47) * (input[i] > (188+offset1));

int rt1_0_48 = (input[i] > (193+offset1));
int t1_0_48 = rt1_0_48 * (2 + (input[i] > (194+offset1)));
t1_0_48 += (1 - rt1_0_48) * (input[i] > (192+offset1));

int rt1_0_49 = (input[i] > (197+offset1));
int t1_0_49 = rt1_0_49 * (2 + (input[i] > (198+offset1)));
t1_0_49 += (1 - rt1_0_49) * (input[i] > (196+offset1));

int rt1_0_50 = (input[i] > (201+offset1));
int t1_0_50 = rt1_0_50 * (2 + (input[i] > (202+offset1)));
t1_0_50 += (1 - rt1_0_50) * (input[i] > (200+offset1));

int rt1_0_51 = (input[i] > (205+offset1));
int t1_0_51 = rt1_0_51 * (2 + (input[i] > (206+offset1)));
t1_0_51 += (1 - rt1_0_51) * (input[i] > (204+offset1));

int rt1_0_52 = (input[i] > (209+offset1));
int t1_0_52 = rt1_0_52 * (2 + (input[i] > (210+offset1)));
t1_0_52 += (1 - rt1_0_52) * (input[i] > (208+offset1));

int rt1_0_53 = (input[i] > (213+offset1));
int t1_0_53 = rt1_0_53 * (2 + (input[i] > (214+offset1)));
t1_0_53 += (1 - rt1_0_53) * (input[i] > (212+offset1));

int rt1_0_54 = (input[i] > (217+offset1));
int t1_0_54 = rt1_0_54 * (2 + (input[i] > (218+offset1)));
t1_0_54 += (1 - rt1_0_54) * (input[i] > (216+offset1));

int rt1_0_55 = (input[i] > (221+offset1));
int t1_0_55 = rt1_0_55 * (2 + (input[i] > (222+offset1)));
t1_0_55 += (1 - rt1_0_55) * (input[i] > (220+offset1));

int rt1_0_56 = (input[i] > (225+offset1));
int t1_0_56 = rt1_0_56 * (2 + (input[i] > (226+offset1)));
t1_0_56 += (1 - rt1_0_56) * (input[i] > (224+offset1));

int rt1_0_57 = (input[i] > (229+offset1));
int t1_0_57 = rt1_0_57 * (2 + (input[i] > (230+offset1)));
t1_0_57 += (1 - rt1_0_57) * (input[i] > (228+offset1));

int rt1_0_58 = (input[i] > (233+offset1));
int t1_0_58 = rt1_0_58 * (2 + (input[i] > (234+offset1)));
t1_0_58 += (1 - rt1_0_58) * (input[i] > (232+offset1));

int rt1_0_59 = (input[i] > (237+offset1));
int t1_0_59 = rt1_0_59 * (2 + (input[i] > (238+offset1)));
t1_0_59 += (1 - rt1_0_59) * (input[i] > (236+offset1));

int rt1_0_60 = (input[i] > (241+offset1));
int t1_0_60 = rt1_0_60 * (2 + (input[i] > (242+offset1)));
t1_0_60 += (1 - rt1_0_60) * (input[i] > (240+offset1));

int rt1_0_61 = (input[i] > (245+offset1));
int t1_0_61 = rt1_0_61 * (2 + (input[i] > (246+offset1)));
t1_0_61 += (1 - rt1_0_61) * (input[i] > (244+offset1));

int rt1_0_62 = (input[i] > (249+offset1));
int t1_0_62 = rt1_0_62 * (2 + (input[i] > (250+offset1)));
t1_0_62 += (1 - rt1_0_62) * (input[i] > (248+offset1));

int rt1_0_63 = (input[i] > (253+offset1));
int t1_0_63 = rt1_0_63 * (2 + (input[i] > (254+offset1)));
t1_0_63 += (1 - rt1_0_63) * (input[i] > (252+offset1));

int rt1_0_64 = (input[i] > (257+offset1));
int t1_0_64 = rt1_0_64 * (2 + (input[i] > (258+offset1)));
t1_0_64 += (1 - rt1_0_64) * (input[i] > (256+offset1));

int rt1_0_65 = (input[i] > (261+offset1));
int t1_0_65 = rt1_0_65 * (2 + (input[i] > (262+offset1)));
t1_0_65 += (1 - rt1_0_65) * (input[i] > (260+offset1));

int rt1_0_66 = (input[i] > (265+offset1));
int t1_0_66 = rt1_0_66 * (2 + (input[i] > (266+offset1)));
t1_0_66 += (1 - rt1_0_66) * (input[i] > (264+offset1));

int rt1_0_67 = (input[i] > (269+offset1));
int t1_0_67 = rt1_0_67 * (2 + (input[i] > (270+offset1)));
t1_0_67 += (1 - rt1_0_67) * (input[i] > (268+offset1));

int rt1_0_68 = (input[i] > (273+offset1));
int t1_0_68 = rt1_0_68 * (2 + (input[i] > (274+offset1)));
t1_0_68 += (1 - rt1_0_68) * (input[i] > (272+offset1));

int rt1_0_69 = (input[i] > (277+offset1));
int t1_0_69 = rt1_0_69 * (2 + (input[i] > (278+offset1)));
t1_0_69 += (1 - rt1_0_69) * (input[i] > (276+offset1));

int rt1_0_70 = (input[i] > (281+offset1));
int t1_0_70 = rt1_0_70 * (2 + (input[i] > (282+offset1)));
t1_0_70 += (1 - rt1_0_70) * (input[i] > (280+offset1));

int rt1_0_71 = (input[i] > (285+offset1));
int t1_0_71 = rt1_0_71 * (2 + (input[i] > (286+offset1)));
t1_0_71 += (1 - rt1_0_71) * (input[i] > (284+offset1));

int rt1_0_72 = (input[i] > (289+offset1));
int t1_0_72 = rt1_0_72 * (2 + (input[i] > (290+offset1)));
t1_0_72 += (1 - rt1_0_72) * (input[i] > (288+offset1));

int rt1_0_73 = (input[i] > (293+offset1));
int t1_0_73 = rt1_0_73 * (2 + (input[i] > (294+offset1)));
t1_0_73 += (1 - rt1_0_73) * (input[i] > (292+offset1));

int rt1_0_74 = (input[i] > (297+offset1));
int t1_0_74 = rt1_0_74 * (2 + (input[i] > (298+offset1)));
t1_0_74 += (1 - rt1_0_74) * (input[i] > (296+offset1));

int rt1_0_75 = (input[i] > (301+offset1));
int t1_0_75 = rt1_0_75 * (2 + (input[i] > (302+offset1)));
t1_0_75 += (1 - rt1_0_75) * (input[i] > (300+offset1));

int rt1_0_76 = (input[i] > (305+offset1));
int t1_0_76 = rt1_0_76 * (2 + (input[i] > (306+offset1)));
t1_0_76 += (1 - rt1_0_76) * (input[i] > (304+offset1));

int rt1_0_77 = (input[i] > (309+offset1));
int t1_0_77 = rt1_0_77 * (2 + (input[i] > (310+offset1)));
t1_0_77 += (1 - rt1_0_77) * (input[i] > (308+offset1));

int rt1_0_78 = (input[i] > (313+offset1));
int t1_0_78 = rt1_0_78 * (2 + (input[i] > (314+offset1)));
t1_0_78 += (1 - rt1_0_78) * (input[i] > (312+offset1));

int rt1_0_79 = (input[i] > (317+offset1));
int t1_0_79 = rt1_0_79 * (2 + (input[i] > (318+offset1)));
t1_0_79 += (1 - rt1_0_79) * (input[i] > (316+offset1));

int rt1_0_80 = (input[i] > (321+offset1));
int t1_0_80 = rt1_0_80 * (2 + (input[i] > (322+offset1)));
t1_0_80 += (1 - rt1_0_80) * (input[i] > (320+offset1));

int rt1_0_81 = (input[i] > (325+offset1));
int t1_0_81 = rt1_0_81 * (2 + (input[i] > (326+offset1)));
t1_0_81 += (1 - rt1_0_81) * (input[i] > (324+offset1));

int rt1_0_82 = (input[i] > (329+offset1));
int t1_0_82 = rt1_0_82 * (2 + (input[i] > (330+offset1)));
t1_0_82 += (1 - rt1_0_82) * (input[i] > (328+offset1));

int rt1_0_83 = (input[i] > (333+offset1));
int t1_0_83 = rt1_0_83 * (2 + (input[i] > (334+offset1)));
t1_0_83 += (1 - rt1_0_83) * (input[i] > (332+offset1));

int rt1_0_84 = (input[i] > (337+offset1));
int t1_0_84 = rt1_0_84 * (2 + (input[i] > (338+offset1)));
t1_0_84 += (1 - rt1_0_84) * (input[i] > (336+offset1));

int rt1_0_85 = (input[i] > (341+offset1));
int t1_0_85 = rt1_0_85 * (2 + (input[i] > (342+offset1)));
t1_0_85 += (1 - rt1_0_85) * (input[i] > (340+offset1));

int rt1_0_86 = (input[i] > (345+offset1));
int t1_0_86 = rt1_0_86 * (2 + (input[i] > (346+offset1)));
t1_0_86 += (1 - rt1_0_86) * (input[i] > (344+offset1));

int rt1_0_87 = (input[i] > (349+offset1));
int t1_0_87 = rt1_0_87 * (2 + (input[i] > (350+offset1)));
t1_0_87 += (1 - rt1_0_87) * (input[i] > (348+offset1));

int rt1_0_88 = (input[i] > (353+offset1));
int t1_0_88 = rt1_0_88 * (2 + (input[i] > (354+offset1)));
t1_0_88 += (1 - rt1_0_88) * (input[i] > (352+offset1));

int rt1_0_89 = (input[i] > (357+offset1));
int t1_0_89 = rt1_0_89 * (2 + (input[i] > (358+offset1)));
t1_0_89 += (1 - rt1_0_89) * (input[i] > (356+offset1));

int rt1_0_90 = (input[i] > (361+offset1));
int t1_0_90 = rt1_0_90 * (2 + (input[i] > (362+offset1)));
t1_0_90 += (1 - rt1_0_90) * (input[i] > (360+offset1));

int rt1_0_91 = (input[i] > (365+offset1));
int t1_0_91 = rt1_0_91 * (2 + (input[i] > (366+offset1)));
t1_0_91 += (1 - rt1_0_91) * (input[i] > (364+offset1));

int rt1_0_92 = (input[i] > (369+offset1));
int t1_0_92 = rt1_0_92 * (2 + (input[i] > (370+offset1)));
t1_0_92 += (1 - rt1_0_92) * (input[i] > (368+offset1));

int rt1_0_93 = (input[i] > (373+offset1));
int t1_0_93 = rt1_0_93 * (2 + (input[i] > (374+offset1)));
t1_0_93 += (1 - rt1_0_93) * (input[i] > (372+offset1));

int rt1_0_94 = (input[i] > (377+offset1));
int t1_0_94 = rt1_0_94 * (2 + (input[i] > (378+offset1)));
t1_0_94 += (1 - rt1_0_94) * (input[i] > (376+offset1));

int rt1_0_95 = (input[i] > (381+offset1));
int t1_0_95 = rt1_0_95 * (2 + (input[i] > (382+offset1)));
t1_0_95 += (1 - rt1_0_95) * (input[i] > (380+offset1));

int rt1_0_96 = (input[i] > (385+offset1));
int t1_0_96 = rt1_0_96 * (2 + (input[i] > (386+offset1)));
t1_0_96 += (1 - rt1_0_96) * (input[i] > (384+offset1));

int rt1_0_97 = (input[i] > (389+offset1));
int t1_0_97 = rt1_0_97 * (2 + (input[i] > (390+offset1)));
t1_0_97 += (1 - rt1_0_97) * (input[i] > (388+offset1));

int rt1_0_98 = (input[i] > (393+offset1));
int t1_0_98 = rt1_0_98 * (2 + (input[i] > (394+offset1)));
t1_0_98 += (1 - rt1_0_98) * (input[i] > (392+offset1));

int rt1_0_99 = (input[i] > (397+offset1));
int t1_0_99 = rt1_0_99 * (2 + (input[i] > (398+offset1)));
t1_0_99 += (1 - rt1_0_99) * (input[i] > (396+offset1));

int rt1_0_100 = (input[i] > (401+offset1));
int t1_0_100 = rt1_0_100 * (2 + (input[i] > (402+offset1)));
t1_0_100 += (1 - rt1_0_100) * (input[i] > (400+offset1));

int rt1_0_101 = (input[i] > (405+offset1));
int t1_0_101 = rt1_0_101 * (2 + (input[i] > (406+offset1)));
t1_0_101 += (1 - rt1_0_101) * (input[i] > (404+offset1));

int rt1_0_102 = (input[i] > (409+offset1));
int t1_0_102 = rt1_0_102 * (2 + (input[i] > (410+offset1)));
t1_0_102 += (1 - rt1_0_102) * (input[i] > (408+offset1));

int rt1_0_103 = (input[i] > (413+offset1));
int t1_0_103 = rt1_0_103 * (2 + (input[i] > (414+offset1)));
t1_0_103 += (1 - rt1_0_103) * (input[i] > (412+offset1));

int rt1_0_104 = (input[i] > (417+offset1));
int t1_0_104 = rt1_0_104 * (2 + (input[i] > (418+offset1)));
t1_0_104 += (1 - rt1_0_104) * (input[i] > (416+offset1));

int rt1_0_105 = (input[i] > (421+offset1));
int t1_0_105 = rt1_0_105 * (2 + (input[i] > (422+offset1)));
t1_0_105 += (1 - rt1_0_105) * (input[i] > (420+offset1));

int rt1_0_106 = (input[i] > (425+offset1));
int t1_0_106 = rt1_0_106 * (2 + (input[i] > (426+offset1)));
t1_0_106 += (1 - rt1_0_106) * (input[i] > (424+offset1));

int rt1_0_107 = (input[i] > (429+offset1));
int t1_0_107 = rt1_0_107 * (2 + (input[i] > (430+offset1)));
t1_0_107 += (1 - rt1_0_107) * (input[i] > (428+offset1));

int rt1_0_108 = (input[i] > (433+offset1));
int t1_0_108 = rt1_0_108 * (2 + (input[i] > (434+offset1)));
t1_0_108 += (1 - rt1_0_108) * (input[i] > (432+offset1));

int rt1_0_109 = (input[i] > (437+offset1));
int t1_0_109 = rt1_0_109 * (2 + (input[i] > (438+offset1)));
t1_0_109 += (1 - rt1_0_109) * (input[i] > (436+offset1));

int rt1_0_110 = (input[i] > (441+offset1));
int t1_0_110 = rt1_0_110 * (2 + (input[i] > (442+offset1)));
t1_0_110 += (1 - rt1_0_110) * (input[i] > (440+offset1));

int rt1_0_111 = (input[i] > (445+offset1));
int t1_0_111 = rt1_0_111 * (2 + (input[i] > (446+offset1)));
t1_0_111 += (1 - rt1_0_111) * (input[i] > (444+offset1));

int rt1_0_112 = (input[i] > (449+offset1));
int t1_0_112 = rt1_0_112 * (2 + (input[i] > (450+offset1)));
t1_0_112 += (1 - rt1_0_112) * (input[i] > (448+offset1));

int rt1_0_113 = (input[i] > (453+offset1));
int t1_0_113 = rt1_0_113 * (2 + (input[i] > (454+offset1)));
t1_0_113 += (1 - rt1_0_113) * (input[i] > (452+offset1));

int rt1_0_114 = (input[i] > (457+offset1));
int t1_0_114 = rt1_0_114 * (2 + (input[i] > (458+offset1)));
t1_0_114 += (1 - rt1_0_114) * (input[i] > (456+offset1));

int rt1_0_115 = (input[i] > (461+offset1));
int t1_0_115 = rt1_0_115 * (2 + (input[i] > (462+offset1)));
t1_0_115 += (1 - rt1_0_115) * (input[i] > (460+offset1));

int rt1_0_116 = (input[i] > (465+offset1));
int t1_0_116 = rt1_0_116 * (2 + (input[i] > (466+offset1)));
t1_0_116 += (1 - rt1_0_116) * (input[i] > (464+offset1));

int rt1_0_117 = (input[i] > (469+offset1));
int t1_0_117 = rt1_0_117 * (2 + (input[i] > (470+offset1)));
t1_0_117 += (1 - rt1_0_117) * (input[i] > (468+offset1));

int rt1_0_118 = (input[i] > (473+offset1));
int t1_0_118 = rt1_0_118 * (2 + (input[i] > (474+offset1)));
t1_0_118 += (1 - rt1_0_118) * (input[i] > (472+offset1));

int rt1_0_119 = (input[i] > (477+offset1));
int t1_0_119 = rt1_0_119 * (2 + (input[i] > (478+offset1)));
t1_0_119 += (1 - rt1_0_119) * (input[i] > (476+offset1));

int rt1_0_120 = (input[i] > (481+offset1));
int t1_0_120 = rt1_0_120 * (2 + (input[i] > (482+offset1)));
t1_0_120 += (1 - rt1_0_120) * (input[i] > (480+offset1));

int rt1_0_121 = (input[i] > (485+offset1));
int t1_0_121 = rt1_0_121 * (2 + (input[i] > (486+offset1)));
t1_0_121 += (1 - rt1_0_121) * (input[i] > (484+offset1));

int rt1_0_122 = (input[i] > (489+offset1));
int t1_0_122 = rt1_0_122 * (2 + (input[i] > (490+offset1)));
t1_0_122 += (1 - rt1_0_122) * (input[i] > (488+offset1));

int rt1_0_123 = (input[i] > (493+offset1));
int t1_0_123 = rt1_0_123 * (2 + (input[i] > (494+offset1)));
t1_0_123 += (1 - rt1_0_123) * (input[i] > (492+offset1));

int rt1_0_124 = (input[i] > (497+offset1));
int t1_0_124 = rt1_0_124 * (2 + (input[i] > (498+offset1)));
t1_0_124 += (1 - rt1_0_124) * (input[i] > (496+offset1));

int rt1_0_125 = (input[i] > (501+offset1));
int t1_0_125 = rt1_0_125 * (2 + (input[i] > (502+offset1)));
t1_0_125 += (1 - rt1_0_125) * (input[i] > (500+offset1));

int rt1_0_126 = (input[i] > (505+offset1));
int t1_0_126 = rt1_0_126 * (2 + (input[i] > (506+offset1)));
t1_0_126 += (1 - rt1_0_126) * (input[i] > (504+offset1));

int rt1_0_127 = (input[i] > (509+offset1));
int t1_0_127 = rt1_0_127 * (2 + (input[i] > (510+offset1)));
t1_0_127 += (1 - rt1_0_127) * (input[i] > (508+offset1));

int rt1_1_0 = (input[i] > (3+offset1));
int t1_1_0 = rt1_1_0 * (4 + t1_0_1);
t1_1_0 += (1 - rt1_1_0) * t1_0_0;

int rt1_1_1 = (input[i] > (11+offset1));
int t1_1_1 = rt1_1_1 * (4 + t1_0_3);
t1_1_1 += (1 - rt1_1_1) * t1_0_2;

int rt1_1_2 = (input[i] > (19+offset1));
int t1_1_2 = rt1_1_2 * (4 + t1_0_5);
t1_1_2 += (1 - rt1_1_2) * t1_0_4;

int rt1_1_3 = (input[i] > (27+offset1));
int t1_1_3 = rt1_1_3 * (4 + t1_0_7);
t1_1_3 += (1 - rt1_1_3) * t1_0_6;

int rt1_1_4 = (input[i] > (35+offset1));
int t1_1_4 = rt1_1_4 * (4 + t1_0_9);
t1_1_4 += (1 - rt1_1_4) * t1_0_8;

int rt1_1_5 = (input[i] > (43+offset1));
int t1_1_5 = rt1_1_5 * (4 + t1_0_11);
t1_1_5 += (1 - rt1_1_5) * t1_0_10;

int rt1_1_6 = (input[i] > (51+offset1));
int t1_1_6 = rt1_1_6 * (4 + t1_0_13);
t1_1_6 += (1 - rt1_1_6) * t1_0_12;

int rt1_1_7 = (input[i] > (59+offset1));
int t1_1_7 = rt1_1_7 * (4 + t1_0_15);
t1_1_7 += (1 - rt1_1_7) * t1_0_14;

int rt1_1_8 = (input[i] > (67+offset1));
int t1_1_8 = rt1_1_8 * (4 + t1_0_17);
t1_1_8 += (1 - rt1_1_8) * t1_0_16;

int rt1_1_9 = (input[i] > (75+offset1));
int t1_1_9 = rt1_1_9 * (4 + t1_0_19);
t1_1_9 += (1 - rt1_1_9) * t1_0_18;

int rt1_1_10 = (input[i] > (83+offset1));
int t1_1_10 = rt1_1_10 * (4 + t1_0_21);
t1_1_10 += (1 - rt1_1_10) * t1_0_20;

int rt1_1_11 = (input[i] > (91+offset1));
int t1_1_11 = rt1_1_11 * (4 + t1_0_23);
t1_1_11 += (1 - rt1_1_11) * t1_0_22;

int rt1_1_12 = (input[i] > (99+offset1));
int t1_1_12 = rt1_1_12 * (4 + t1_0_25);
t1_1_12 += (1 - rt1_1_12) * t1_0_24;

int rt1_1_13 = (input[i] > (107+offset1));
int t1_1_13 = rt1_1_13 * (4 + t1_0_27);
t1_1_13 += (1 - rt1_1_13) * t1_0_26;

int rt1_1_14 = (input[i] > (115+offset1));
int t1_1_14 = rt1_1_14 * (4 + t1_0_29);
t1_1_14 += (1 - rt1_1_14) * t1_0_28;

int rt1_1_15 = (input[i] > (123+offset1));
int t1_1_15 = rt1_1_15 * (4 + t1_0_31);
t1_1_15 += (1 - rt1_1_15) * t1_0_30;

int rt1_1_16 = (input[i] > (131+offset1));
int t1_1_16 = rt1_1_16 * (4 + t1_0_33);
t1_1_16 += (1 - rt1_1_16) * t1_0_32;

int rt1_1_17 = (input[i] > (139+offset1));
int t1_1_17 = rt1_1_17 * (4 + t1_0_35);
t1_1_17 += (1 - rt1_1_17) * t1_0_34;

int rt1_1_18 = (input[i] > (147+offset1));
int t1_1_18 = rt1_1_18 * (4 + t1_0_37);
t1_1_18 += (1 - rt1_1_18) * t1_0_36;

int rt1_1_19 = (input[i] > (155+offset1));
int t1_1_19 = rt1_1_19 * (4 + t1_0_39);
t1_1_19 += (1 - rt1_1_19) * t1_0_38;

int rt1_1_20 = (input[i] > (163+offset1));
int t1_1_20 = rt1_1_20 * (4 + t1_0_41);
t1_1_20 += (1 - rt1_1_20) * t1_0_40;

int rt1_1_21 = (input[i] > (171+offset1));
int t1_1_21 = rt1_1_21 * (4 + t1_0_43);
t1_1_21 += (1 - rt1_1_21) * t1_0_42;

int rt1_1_22 = (input[i] > (179+offset1));
int t1_1_22 = rt1_1_22 * (4 + t1_0_45);
t1_1_22 += (1 - rt1_1_22) * t1_0_44;

int rt1_1_23 = (input[i] > (187+offset1));
int t1_1_23 = rt1_1_23 * (4 + t1_0_47);
t1_1_23 += (1 - rt1_1_23) * t1_0_46;

int rt1_1_24 = (input[i] > (195+offset1));
int t1_1_24 = rt1_1_24 * (4 + t1_0_49);
t1_1_24 += (1 - rt1_1_24) * t1_0_48;

int rt1_1_25 = (input[i] > (203+offset1));
int t1_1_25 = rt1_1_25 * (4 + t1_0_51);
t1_1_25 += (1 - rt1_1_25) * t1_0_50;

int rt1_1_26 = (input[i] > (211+offset1));
int t1_1_26 = rt1_1_26 * (4 + t1_0_53);
t1_1_26 += (1 - rt1_1_26) * t1_0_52;

int rt1_1_27 = (input[i] > (219+offset1));
int t1_1_27 = rt1_1_27 * (4 + t1_0_55);
t1_1_27 += (1 - rt1_1_27) * t1_0_54;

int rt1_1_28 = (input[i] > (227+offset1));
int t1_1_28 = rt1_1_28 * (4 + t1_0_57);
t1_1_28 += (1 - rt1_1_28) * t1_0_56;

int rt1_1_29 = (input[i] > (235+offset1));
int t1_1_29 = rt1_1_29 * (4 + t1_0_59);
t1_1_29 += (1 - rt1_1_29) * t1_0_58;

int rt1_1_30 = (input[i] > (243+offset1));
int t1_1_30 = rt1_1_30 * (4 + t1_0_61);
t1_1_30 += (1 - rt1_1_30) * t1_0_60;

int rt1_1_31 = (input[i] > (251+offset1));
int t1_1_31 = rt1_1_31 * (4 + t1_0_63);
t1_1_31 += (1 - rt1_1_31) * t1_0_62;

int rt1_1_32 = (input[i] > (259+offset1));
int t1_1_32 = rt1_1_32 * (4 + t1_0_65);
t1_1_32 += (1 - rt1_1_32) * t1_0_64;

int rt1_1_33 = (input[i] > (267+offset1));
int t1_1_33 = rt1_1_33 * (4 + t1_0_67);
t1_1_33 += (1 - rt1_1_33) * t1_0_66;

int rt1_1_34 = (input[i] > (275+offset1));
int t1_1_34 = rt1_1_34 * (4 + t1_0_69);
t1_1_34 += (1 - rt1_1_34) * t1_0_68;

int rt1_1_35 = (input[i] > (283+offset1));
int t1_1_35 = rt1_1_35 * (4 + t1_0_71);
t1_1_35 += (1 - rt1_1_35) * t1_0_70;

int rt1_1_36 = (input[i] > (291+offset1));
int t1_1_36 = rt1_1_36 * (4 + t1_0_73);
t1_1_36 += (1 - rt1_1_36) * t1_0_72;

int rt1_1_37 = (input[i] > (299+offset1));
int t1_1_37 = rt1_1_37 * (4 + t1_0_75);
t1_1_37 += (1 - rt1_1_37) * t1_0_74;

int rt1_1_38 = (input[i] > (307+offset1));
int t1_1_38 = rt1_1_38 * (4 + t1_0_77);
t1_1_38 += (1 - rt1_1_38) * t1_0_76;

int rt1_1_39 = (input[i] > (315+offset1));
int t1_1_39 = rt1_1_39 * (4 + t1_0_79);
t1_1_39 += (1 - rt1_1_39) * t1_0_78;

int rt1_1_40 = (input[i] > (323+offset1));
int t1_1_40 = rt1_1_40 * (4 + t1_0_81);
t1_1_40 += (1 - rt1_1_40) * t1_0_80;

int rt1_1_41 = (input[i] > (331+offset1));
int t1_1_41 = rt1_1_41 * (4 + t1_0_83);
t1_1_41 += (1 - rt1_1_41) * t1_0_82;

int rt1_1_42 = (input[i] > (339+offset1));
int t1_1_42 = rt1_1_42 * (4 + t1_0_85);
t1_1_42 += (1 - rt1_1_42) * t1_0_84;

int rt1_1_43 = (input[i] > (347+offset1));
int t1_1_43 = rt1_1_43 * (4 + t1_0_87);
t1_1_43 += (1 - rt1_1_43) * t1_0_86;

int rt1_1_44 = (input[i] > (355+offset1));
int t1_1_44 = rt1_1_44 * (4 + t1_0_89);
t1_1_44 += (1 - rt1_1_44) * t1_0_88;

int rt1_1_45 = (input[i] > (363+offset1));
int t1_1_45 = rt1_1_45 * (4 + t1_0_91);
t1_1_45 += (1 - rt1_1_45) * t1_0_90;

int rt1_1_46 = (input[i] > (371+offset1));
int t1_1_46 = rt1_1_46 * (4 + t1_0_93);
t1_1_46 += (1 - rt1_1_46) * t1_0_92;

int rt1_1_47 = (input[i] > (379+offset1));
int t1_1_47 = rt1_1_47 * (4 + t1_0_95);
t1_1_47 += (1 - rt1_1_47) * t1_0_94;

int rt1_1_48 = (input[i] > (387+offset1));
int t1_1_48 = rt1_1_48 * (4 + t1_0_97);
t1_1_48 += (1 - rt1_1_48) * t1_0_96;

int rt1_1_49 = (input[i] > (395+offset1));
int t1_1_49 = rt1_1_49 * (4 + t1_0_99);
t1_1_49 += (1 - rt1_1_49) * t1_0_98;

int rt1_1_50 = (input[i] > (403+offset1));
int t1_1_50 = rt1_1_50 * (4 + t1_0_101);
t1_1_50 += (1 - rt1_1_50) * t1_0_100;

int rt1_1_51 = (input[i] > (411+offset1));
int t1_1_51 = rt1_1_51 * (4 + t1_0_103);
t1_1_51 += (1 - rt1_1_51) * t1_0_102;

int rt1_1_52 = (input[i] > (419+offset1));
int t1_1_52 = rt1_1_52 * (4 + t1_0_105);
t1_1_52 += (1 - rt1_1_52) * t1_0_104;

int rt1_1_53 = (input[i] > (427+offset1));
int t1_1_53 = rt1_1_53 * (4 + t1_0_107);
t1_1_53 += (1 - rt1_1_53) * t1_0_106;

int rt1_1_54 = (input[i] > (435+offset1));
int t1_1_54 = rt1_1_54 * (4 + t1_0_109);
t1_1_54 += (1 - rt1_1_54) * t1_0_108;

int rt1_1_55 = (input[i] > (443+offset1));
int t1_1_55 = rt1_1_55 * (4 + t1_0_111);
t1_1_55 += (1 - rt1_1_55) * t1_0_110;

int rt1_1_56 = (input[i] > (451+offset1));
int t1_1_56 = rt1_1_56 * (4 + t1_0_113);
t1_1_56 += (1 - rt1_1_56) * t1_0_112;

int rt1_1_57 = (input[i] > (459+offset1));
int t1_1_57 = rt1_1_57 * (4 + t1_0_115);
t1_1_57 += (1 - rt1_1_57) * t1_0_114;

int rt1_1_58 = (input[i] > (467+offset1));
int t1_1_58 = rt1_1_58 * (4 + t1_0_117);
t1_1_58 += (1 - rt1_1_58) * t1_0_116;

int rt1_1_59 = (input[i] > (475+offset1));
int t1_1_59 = rt1_1_59 * (4 + t1_0_119);
t1_1_59 += (1 - rt1_1_59) * t1_0_118;

int rt1_1_60 = (input[i] > (483+offset1));
int t1_1_60 = rt1_1_60 * (4 + t1_0_121);
t1_1_60 += (1 - rt1_1_60) * t1_0_120;

int rt1_1_61 = (input[i] > (491+offset1));
int t1_1_61 = rt1_1_61 * (4 + t1_0_123);
t1_1_61 += (1 - rt1_1_61) * t1_0_122;

int rt1_1_62 = (input[i] > (499+offset1));
int t1_1_62 = rt1_1_62 * (4 + t1_0_125);
t1_1_62 += (1 - rt1_1_62) * t1_0_124;

int rt1_1_63 = (input[i] > (507+offset1));
int t1_1_63 = rt1_1_63 * (4 + t1_0_127);
t1_1_63 += (1 - rt1_1_63) * t1_0_126;

int rt1_2_0 = (input[i] > (7+offset1));
int t1_2_0 = rt1_2_0 * (8 + t1_1_1);
t1_2_0 += (1 - rt1_2_0) * t1_1_0;

int rt1_2_1 = (input[i] > (23+offset1));
int t1_2_1 = rt1_2_1 * (8 + t1_1_3);
t1_2_1 += (1 - rt1_2_1) * t1_1_2;

int rt1_2_2 = (input[i] > (39+offset1));
int t1_2_2 = rt1_2_2 * (8 + t1_1_5);
t1_2_2 += (1 - rt1_2_2) * t1_1_4;

int rt1_2_3 = (input[i] > (55+offset1));
int t1_2_3 = rt1_2_3 * (8 + t1_1_7);
t1_2_3 += (1 - rt1_2_3) * t1_1_6;

int rt1_2_4 = (input[i] > (71+offset1));
int t1_2_4 = rt1_2_4 * (8 + t1_1_9);
t1_2_4 += (1 - rt1_2_4) * t1_1_8;

int rt1_2_5 = (input[i] > (87+offset1));
int t1_2_5 = rt1_2_5 * (8 + t1_1_11);
t1_2_5 += (1 - rt1_2_5) * t1_1_10;

int rt1_2_6 = (input[i] > (103+offset1));
int t1_2_6 = rt1_2_6 * (8 + t1_1_13);
t1_2_6 += (1 - rt1_2_6) * t1_1_12;

int rt1_2_7 = (input[i] > (119+offset1));
int t1_2_7 = rt1_2_7 * (8 + t1_1_15);
t1_2_7 += (1 - rt1_2_7) * t1_1_14;

int rt1_2_8 = (input[i] > (135+offset1));
int t1_2_8 = rt1_2_8 * (8 + t1_1_17);
t1_2_8 += (1 - rt1_2_8) * t1_1_16;

int rt1_2_9 = (input[i] > (151+offset1));
int t1_2_9 = rt1_2_9 * (8 + t1_1_19);
t1_2_9 += (1 - rt1_2_9) * t1_1_18;

int rt1_2_10 = (input[i] > (167+offset1));
int t1_2_10 = rt1_2_10 * (8 + t1_1_21);
t1_2_10 += (1 - rt1_2_10) * t1_1_20;

int rt1_2_11 = (input[i] > (183+offset1));
int t1_2_11 = rt1_2_11 * (8 + t1_1_23);
t1_2_11 += (1 - rt1_2_11) * t1_1_22;

int rt1_2_12 = (input[i] > (199+offset1));
int t1_2_12 = rt1_2_12 * (8 + t1_1_25);
t1_2_12 += (1 - rt1_2_12) * t1_1_24;

int rt1_2_13 = (input[i] > (215+offset1));
int t1_2_13 = rt1_2_13 * (8 + t1_1_27);
t1_2_13 += (1 - rt1_2_13) * t1_1_26;

int rt1_2_14 = (input[i] > (231+offset1));
int t1_2_14 = rt1_2_14 * (8 + t1_1_29);
t1_2_14 += (1 - rt1_2_14) * t1_1_28;

int rt1_2_15 = (input[i] > (247+offset1));
int t1_2_15 = rt1_2_15 * (8 + t1_1_31);
t1_2_15 += (1 - rt1_2_15) * t1_1_30;

int rt1_2_16 = (input[i] > (263+offset1));
int t1_2_16 = rt1_2_16 * (8 + t1_1_33);
t1_2_16 += (1 - rt1_2_16) * t1_1_32;

int rt1_2_17 = (input[i] > (279+offset1));
int t1_2_17 = rt1_2_17 * (8 + t1_1_35);
t1_2_17 += (1 - rt1_2_17) * t1_1_34;

int rt1_2_18 = (input[i] > (295+offset1));
int t1_2_18 = rt1_2_18 * (8 + t1_1_37);
t1_2_18 += (1 - rt1_2_18) * t1_1_36;

int rt1_2_19 = (input[i] > (311+offset1));
int t1_2_19 = rt1_2_19 * (8 + t1_1_39);
t1_2_19 += (1 - rt1_2_19) * t1_1_38;

int rt1_2_20 = (input[i] > (327+offset1));
int t1_2_20 = rt1_2_20 * (8 + t1_1_41);
t1_2_20 += (1 - rt1_2_20) * t1_1_40;

int rt1_2_21 = (input[i] > (343+offset1));
int t1_2_21 = rt1_2_21 * (8 + t1_1_43);
t1_2_21 += (1 - rt1_2_21) * t1_1_42;

int rt1_2_22 = (input[i] > (359+offset1));
int t1_2_22 = rt1_2_22 * (8 + t1_1_45);
t1_2_22 += (1 - rt1_2_22) * t1_1_44;

int rt1_2_23 = (input[i] > (375+offset1));
int t1_2_23 = rt1_2_23 * (8 + t1_1_47);
t1_2_23 += (1 - rt1_2_23) * t1_1_46;

int rt1_2_24 = (input[i] > (391+offset1));
int t1_2_24 = rt1_2_24 * (8 + t1_1_49);
t1_2_24 += (1 - rt1_2_24) * t1_1_48;

int rt1_2_25 = (input[i] > (407+offset1));
int t1_2_25 = rt1_2_25 * (8 + t1_1_51);
t1_2_25 += (1 - rt1_2_25) * t1_1_50;

int rt1_2_26 = (input[i] > (423+offset1));
int t1_2_26 = rt1_2_26 * (8 + t1_1_53);
t1_2_26 += (1 - rt1_2_26) * t1_1_52;

int rt1_2_27 = (input[i] > (439+offset1));
int t1_2_27 = rt1_2_27 * (8 + t1_1_55);
t1_2_27 += (1 - rt1_2_27) * t1_1_54;

int rt1_2_28 = (input[i] > (455+offset1));
int t1_2_28 = rt1_2_28 * (8 + t1_1_57);
t1_2_28 += (1 - rt1_2_28) * t1_1_56;

int rt1_2_29 = (input[i] > (471+offset1));
int t1_2_29 = rt1_2_29 * (8 + t1_1_59);
t1_2_29 += (1 - rt1_2_29) * t1_1_58;

int rt1_2_30 = (input[i] > (487+offset1));
int t1_2_30 = rt1_2_30 * (8 + t1_1_61);
t1_2_30 += (1 - rt1_2_30) * t1_1_60;

int rt1_2_31 = (input[i] > (503+offset1));
int t1_2_31 = rt1_2_31 * (8 + t1_1_63);
t1_2_31 += (1 - rt1_2_31) * t1_1_62;

int rt1_3_0 = (input[i] > (15+offset1));
int t1_3_0 = rt1_3_0 * (16 + t1_2_1);
t1_3_0 += (1 - rt1_3_0) * t1_2_0;

int rt1_3_1 = (input[i] > (47+offset1));
int t1_3_1 = rt1_3_1 * (16 + t1_2_3);
t1_3_1 += (1 - rt1_3_1) * t1_2_2;

int rt1_3_2 = (input[i] > (79+offset1));
int t1_3_2 = rt1_3_2 * (16 + t1_2_5);
t1_3_2 += (1 - rt1_3_2) * t1_2_4;

int rt1_3_3 = (input[i] > (111+offset1));
int t1_3_3 = rt1_3_3 * (16 + t1_2_7);
t1_3_3 += (1 - rt1_3_3) * t1_2_6;

int rt1_3_4 = (input[i] > (143+offset1));
int t1_3_4 = rt1_3_4 * (16 + t1_2_9);
t1_3_4 += (1 - rt1_3_4) * t1_2_8;

int rt1_3_5 = (input[i] > (175+offset1));
int t1_3_5 = rt1_3_5 * (16 + t1_2_11);
t1_3_5 += (1 - rt1_3_5) * t1_2_10;

int rt1_3_6 = (input[i] > (207+offset1));
int t1_3_6 = rt1_3_6 * (16 + t1_2_13);
t1_3_6 += (1 - rt1_3_6) * t1_2_12;

int rt1_3_7 = (input[i] > (239+offset1));
int t1_3_7 = rt1_3_7 * (16 + t1_2_15);
t1_3_7 += (1 - rt1_3_7) * t1_2_14;

int rt1_3_8 = (input[i] > (271+offset1));
int t1_3_8 = rt1_3_8 * (16 + t1_2_17);
t1_3_8 += (1 - rt1_3_8) * t1_2_16;

int rt1_3_9 = (input[i] > (303+offset1));
int t1_3_9 = rt1_3_9 * (16 + t1_2_19);
t1_3_9 += (1 - rt1_3_9) * t1_2_18;

int rt1_3_10 = (input[i] > (335+offset1));
int t1_3_10 = rt1_3_10 * (16 + t1_2_21);
t1_3_10 += (1 - rt1_3_10) * t1_2_20;

int rt1_3_11 = (input[i] > (367+offset1));
int t1_3_11 = rt1_3_11 * (16 + t1_2_23);
t1_3_11 += (1 - rt1_3_11) * t1_2_22;

int rt1_3_12 = (input[i] > (399+offset1));
int t1_3_12 = rt1_3_12 * (16 + t1_2_25);
t1_3_12 += (1 - rt1_3_12) * t1_2_24;

int rt1_3_13 = (input[i] > (431+offset1));
int t1_3_13 = rt1_3_13 * (16 + t1_2_27);
t1_3_13 += (1 - rt1_3_13) * t1_2_26;

int rt1_3_14 = (input[i] > (463+offset1));
int t1_3_14 = rt1_3_14 * (16 + t1_2_29);
t1_3_14 += (1 - rt1_3_14) * t1_2_28;

int rt1_3_15 = (input[i] > (495+offset1));
int t1_3_15 = rt1_3_15 * (16 + t1_2_31);
t1_3_15 += (1 - rt1_3_15) * t1_2_30;

int rt1_4_0 = (input[i] > (31+offset1));
int t1_4_0 = rt1_4_0 * (32 + t1_3_1);
t1_4_0 += (1 - rt1_4_0) * t1_3_0;

int rt1_4_1 = (input[i] > (95+offset1));
int t1_4_1 = rt1_4_1 * (32 + t1_3_3);
t1_4_1 += (1 - rt1_4_1) * t1_3_2;

int rt1_4_2 = (input[i] > (159+offset1));
int t1_4_2 = rt1_4_2 * (32 + t1_3_5);
t1_4_2 += (1 - rt1_4_2) * t1_3_4;

int rt1_4_3 = (input[i] > (223+offset1));
int t1_4_3 = rt1_4_3 * (32 + t1_3_7);
t1_4_3 += (1 - rt1_4_3) * t1_3_6;

int rt1_4_4 = (input[i] > (287+offset1));
int t1_4_4 = rt1_4_4 * (32 + t1_3_9);
t1_4_4 += (1 - rt1_4_4) * t1_3_8;

int rt1_4_5 = (input[i] > (351+offset1));
int t1_4_5 = rt1_4_5 * (32 + t1_3_11);
t1_4_5 += (1 - rt1_4_5) * t1_3_10;

int rt1_4_6 = (input[i] > (415+offset1));
int t1_4_6 = rt1_4_6 * (32 + t1_3_13);
t1_4_6 += (1 - rt1_4_6) * t1_3_12;

int rt1_4_7 = (input[i] > (479+offset1));
int t1_4_7 = rt1_4_7 * (32 + t1_3_15);
t1_4_7 += (1 - rt1_4_7) * t1_3_14;

int rt1_5_0 = (input[i] > (63+offset1));
int t1_5_0 = rt1_5_0 * (64 + t1_4_1);
t1_5_0 += (1 - rt1_5_0) * t1_4_0;

int rt1_5_1 = (input[i] > (191+offset1));
int t1_5_1 = rt1_5_1 * (64 + t1_4_3);
t1_5_1 += (1 - rt1_5_1) * t1_4_2;

int rt1_5_2 = (input[i] > (319+offset1));
int t1_5_2 = rt1_5_2 * (64 + t1_4_5);
t1_5_2 += (1 - rt1_5_2) * t1_4_4;

int rt1_5_3 = (input[i] > (447+offset1));
int t1_5_3 = rt1_5_3 * (64 + t1_4_7);
t1_5_3 += (1 - rt1_5_3) * t1_4_6;

int rt1_6_0 = (input[i] > (127+offset1));
int t1_6_0 = rt1_6_0 * (128 + t1_5_1);
t1_6_0 += (1 - rt1_6_0) * t1_5_0;

int rt1_6_1 = (input[i] > (383+offset1));
int t1_6_1 = rt1_6_1 * (128 + t1_5_3);
t1_6_1 += (1 - rt1_6_1) * t1_5_2;

int rt1_7_0 = (input[i] > (255+offset1));
int t1_7_0 = rt1_7_0 * (256 + t1_6_1);
t1_7_0 += (1 - rt1_7_0) * t1_6_0;

int rt2_0_0 = (input[i] > (1+offset2));
int t2_0_0 = rt2_0_0 * (2 + (input[i] > (2+offset2)));
t2_0_0 += (1 - rt2_0_0) * (input[i] > (0+offset2));

int rt2_0_1 = (input[i] > (5+offset2));
int t2_0_1 = rt2_0_1 * (2 + (input[i] > (6+offset2)));
t2_0_1 += (1 - rt2_0_1) * (input[i] > (4+offset2));

int rt2_0_2 = (input[i] > (9+offset2));
int t2_0_2 = rt2_0_2 * (2 + (input[i] > (10+offset2)));
t2_0_2 += (1 - rt2_0_2) * (input[i] > (8+offset2));

int rt2_0_3 = (input[i] > (13+offset2));
int t2_0_3 = rt2_0_3 * (2 + (input[i] > (14+offset2)));
t2_0_3 += (1 - rt2_0_3) * (input[i] > (12+offset2));

int rt2_0_4 = (input[i] > (17+offset2));
int t2_0_4 = rt2_0_4 * (2 + (input[i] > (18+offset2)));
t2_0_4 += (1 - rt2_0_4) * (input[i] > (16+offset2));

int rt2_0_5 = (input[i] > (21+offset2));
int t2_0_5 = rt2_0_5 * (2 + (input[i] > (22+offset2)));
t2_0_5 += (1 - rt2_0_5) * (input[i] > (20+offset2));

int rt2_0_6 = (input[i] > (25+offset2));
int t2_0_6 = rt2_0_6 * (2 + (input[i] > (26+offset2)));
t2_0_6 += (1 - rt2_0_6) * (input[i] > (24+offset2));

int rt2_0_7 = (input[i] > (29+offset2));
int t2_0_7 = rt2_0_7 * (2 + (input[i] > (30+offset2)));
t2_0_7 += (1 - rt2_0_7) * (input[i] > (28+offset2));

int rt2_0_8 = (input[i] > (33+offset2));
int t2_0_8 = rt2_0_8 * (2 + (input[i] > (34+offset2)));
t2_0_8 += (1 - rt2_0_8) * (input[i] > (32+offset2));

int rt2_0_9 = (input[i] > (37+offset2));
int t2_0_9 = rt2_0_9 * (2 + (input[i] > (38+offset2)));
t2_0_9 += (1 - rt2_0_9) * (input[i] > (36+offset2));

int rt2_0_10 = (input[i] > (41+offset2));
int t2_0_10 = rt2_0_10 * (2 + (input[i] > (42+offset2)));
t2_0_10 += (1 - rt2_0_10) * (input[i] > (40+offset2));

int rt2_0_11 = (input[i] > (45+offset2));
int t2_0_11 = rt2_0_11 * (2 + (input[i] > (46+offset2)));
t2_0_11 += (1 - rt2_0_11) * (input[i] > (44+offset2));

int rt2_0_12 = (input[i] > (49+offset2));
int t2_0_12 = rt2_0_12 * (2 + (input[i] > (50+offset2)));
t2_0_12 += (1 - rt2_0_12) * (input[i] > (48+offset2));

int rt2_0_13 = (input[i] > (53+offset2));
int t2_0_13 = rt2_0_13 * (2 + (input[i] > (54+offset2)));
t2_0_13 += (1 - rt2_0_13) * (input[i] > (52+offset2));

int rt2_0_14 = (input[i] > (57+offset2));
int t2_0_14 = rt2_0_14 * (2 + (input[i] > (58+offset2)));
t2_0_14 += (1 - rt2_0_14) * (input[i] > (56+offset2));

int rt2_0_15 = (input[i] > (61+offset2));
int t2_0_15 = rt2_0_15 * (2 + (input[i] > (62+offset2)));
t2_0_15 += (1 - rt2_0_15) * (input[i] > (60+offset2));

int rt2_0_16 = (input[i] > (65+offset2));
int t2_0_16 = rt2_0_16 * (2 + (input[i] > (66+offset2)));
t2_0_16 += (1 - rt2_0_16) * (input[i] > (64+offset2));

int rt2_0_17 = (input[i] > (69+offset2));
int t2_0_17 = rt2_0_17 * (2 + (input[i] > (70+offset2)));
t2_0_17 += (1 - rt2_0_17) * (input[i] > (68+offset2));

int rt2_0_18 = (input[i] > (73+offset2));
int t2_0_18 = rt2_0_18 * (2 + (input[i] > (74+offset2)));
t2_0_18 += (1 - rt2_0_18) * (input[i] > (72+offset2));

int rt2_0_19 = (input[i] > (77+offset2));
int t2_0_19 = rt2_0_19 * (2 + (input[i] > (78+offset2)));
t2_0_19 += (1 - rt2_0_19) * (input[i] > (76+offset2));

int rt2_0_20 = (input[i] > (81+offset2));
int t2_0_20 = rt2_0_20 * (2 + (input[i] > (82+offset2)));
t2_0_20 += (1 - rt2_0_20) * (input[i] > (80+offset2));

int rt2_0_21 = (input[i] > (85+offset2));
int t2_0_21 = rt2_0_21 * (2 + (input[i] > (86+offset2)));
t2_0_21 += (1 - rt2_0_21) * (input[i] > (84+offset2));

int rt2_0_22 = (input[i] > (89+offset2));
int t2_0_22 = rt2_0_22 * (2 + (input[i] > (90+offset2)));
t2_0_22 += (1 - rt2_0_22) * (input[i] > (88+offset2));

int rt2_0_23 = (input[i] > (93+offset2));
int t2_0_23 = rt2_0_23 * (2 + (input[i] > (94+offset2)));
t2_0_23 += (1 - rt2_0_23) * (input[i] > (92+offset2));

int rt2_0_24 = (input[i] > (97+offset2));
int t2_0_24 = rt2_0_24 * (2 + (input[i] > (98+offset2)));
t2_0_24 += (1 - rt2_0_24) * (input[i] > (96+offset2));

int rt2_0_25 = (input[i] > (101+offset2));
int t2_0_25 = rt2_0_25 * (2 + (input[i] > (102+offset2)));
t2_0_25 += (1 - rt2_0_25) * (input[i] > (100+offset2));

int rt2_0_26 = (input[i] > (105+offset2));
int t2_0_26 = rt2_0_26 * (2 + (input[i] > (106+offset2)));
t2_0_26 += (1 - rt2_0_26) * (input[i] > (104+offset2));

int rt2_0_27 = (input[i] > (109+offset2));
int t2_0_27 = rt2_0_27 * (2 + (input[i] > (110+offset2)));
t2_0_27 += (1 - rt2_0_27) * (input[i] > (108+offset2));

int rt2_0_28 = (input[i] > (113+offset2));
int t2_0_28 = rt2_0_28 * (2 + (input[i] > (114+offset2)));
t2_0_28 += (1 - rt2_0_28) * (input[i] > (112+offset2));

int rt2_0_29 = (input[i] > (117+offset2));
int t2_0_29 = rt2_0_29 * (2 + (input[i] > (118+offset2)));
t2_0_29 += (1 - rt2_0_29) * (input[i] > (116+offset2));

int rt2_0_30 = (input[i] > (121+offset2));
int t2_0_30 = rt2_0_30 * (2 + (input[i] > (122+offset2)));
t2_0_30 += (1 - rt2_0_30) * (input[i] > (120+offset2));

int rt2_0_31 = (input[i] > (125+offset2));
int t2_0_31 = rt2_0_31 * (2 + (input[i] > (126+offset2)));
t2_0_31 += (1 - rt2_0_31) * (input[i] > (124+offset2));

int rt2_0_32 = (input[i] > (129+offset2));
int t2_0_32 = rt2_0_32 * (2 + (input[i] > (130+offset2)));
t2_0_32 += (1 - rt2_0_32) * (input[i] > (128+offset2));

int rt2_0_33 = (input[i] > (133+offset2));
int t2_0_33 = rt2_0_33 * (2 + (input[i] > (134+offset2)));
t2_0_33 += (1 - rt2_0_33) * (input[i] > (132+offset2));

int rt2_0_34 = (input[i] > (137+offset2));
int t2_0_34 = rt2_0_34 * (2 + (input[i] > (138+offset2)));
t2_0_34 += (1 - rt2_0_34) * (input[i] > (136+offset2));

int rt2_0_35 = (input[i] > (141+offset2));
int t2_0_35 = rt2_0_35 * (2 + (input[i] > (142+offset2)));
t2_0_35 += (1 - rt2_0_35) * (input[i] > (140+offset2));

int rt2_0_36 = (input[i] > (145+offset2));
int t2_0_36 = rt2_0_36 * (2 + (input[i] > (146+offset2)));
t2_0_36 += (1 - rt2_0_36) * (input[i] > (144+offset2));

int rt2_0_37 = (input[i] > (149+offset2));
int t2_0_37 = rt2_0_37 * (2 + (input[i] > (150+offset2)));
t2_0_37 += (1 - rt2_0_37) * (input[i] > (148+offset2));

int rt2_0_38 = (input[i] > (153+offset2));
int t2_0_38 = rt2_0_38 * (2 + (input[i] > (154+offset2)));
t2_0_38 += (1 - rt2_0_38) * (input[i] > (152+offset2));

int rt2_0_39 = (input[i] > (157+offset2));
int t2_0_39 = rt2_0_39 * (2 + (input[i] > (158+offset2)));
t2_0_39 += (1 - rt2_0_39) * (input[i] > (156+offset2));

int rt2_0_40 = (input[i] > (161+offset2));
int t2_0_40 = rt2_0_40 * (2 + (input[i] > (162+offset2)));
t2_0_40 += (1 - rt2_0_40) * (input[i] > (160+offset2));

int rt2_0_41 = (input[i] > (165+offset2));
int t2_0_41 = rt2_0_41 * (2 + (input[i] > (166+offset2)));
t2_0_41 += (1 - rt2_0_41) * (input[i] > (164+offset2));

int rt2_0_42 = (input[i] > (169+offset2));
int t2_0_42 = rt2_0_42 * (2 + (input[i] > (170+offset2)));
t2_0_42 += (1 - rt2_0_42) * (input[i] > (168+offset2));

int rt2_0_43 = (input[i] > (173+offset2));
int t2_0_43 = rt2_0_43 * (2 + (input[i] > (174+offset2)));
t2_0_43 += (1 - rt2_0_43) * (input[i] > (172+offset2));

int rt2_0_44 = (input[i] > (177+offset2));
int t2_0_44 = rt2_0_44 * (2 + (input[i] > (178+offset2)));
t2_0_44 += (1 - rt2_0_44) * (input[i] > (176+offset2));

int rt2_0_45 = (input[i] > (181+offset2));
int t2_0_45 = rt2_0_45 * (2 + (input[i] > (182+offset2)));
t2_0_45 += (1 - rt2_0_45) * (input[i] > (180+offset2));

int rt2_0_46 = (input[i] > (185+offset2));
int t2_0_46 = rt2_0_46 * (2 + (input[i] > (186+offset2)));
t2_0_46 += (1 - rt2_0_46) * (input[i] > (184+offset2));

int rt2_0_47 = (input[i] > (189+offset2));
int t2_0_47 = rt2_0_47 * (2 + (input[i] > (190+offset2)));
t2_0_47 += (1 - rt2_0_47) * (input[i] > (188+offset2));

int rt2_0_48 = (input[i] > (193+offset2));
int t2_0_48 = rt2_0_48 * (2 + (input[i] > (194+offset2)));
t2_0_48 += (1 - rt2_0_48) * (input[i] > (192+offset2));

int rt2_0_49 = (input[i] > (197+offset2));
int t2_0_49 = rt2_0_49 * (2 + (input[i] > (198+offset2)));
t2_0_49 += (1 - rt2_0_49) * (input[i] > (196+offset2));

int rt2_0_50 = (input[i] > (201+offset2));
int t2_0_50 = rt2_0_50 * (2 + (input[i] > (202+offset2)));
t2_0_50 += (1 - rt2_0_50) * (input[i] > (200+offset2));

int rt2_0_51 = (input[i] > (205+offset2));
int t2_0_51 = rt2_0_51 * (2 + (input[i] > (206+offset2)));
t2_0_51 += (1 - rt2_0_51) * (input[i] > (204+offset2));

int rt2_0_52 = (input[i] > (209+offset2));
int t2_0_52 = rt2_0_52 * (2 + (input[i] > (210+offset2)));
t2_0_52 += (1 - rt2_0_52) * (input[i] > (208+offset2));

int rt2_0_53 = (input[i] > (213+offset2));
int t2_0_53 = rt2_0_53 * (2 + (input[i] > (214+offset2)));
t2_0_53 += (1 - rt2_0_53) * (input[i] > (212+offset2));

int rt2_0_54 = (input[i] > (217+offset2));
int t2_0_54 = rt2_0_54 * (2 + (input[i] > (218+offset2)));
t2_0_54 += (1 - rt2_0_54) * (input[i] > (216+offset2));

int rt2_0_55 = (input[i] > (221+offset2));
int t2_0_55 = rt2_0_55 * (2 + (input[i] > (222+offset2)));
t2_0_55 += (1 - rt2_0_55) * (input[i] > (220+offset2));

int rt2_0_56 = (input[i] > (225+offset2));
int t2_0_56 = rt2_0_56 * (2 + (input[i] > (226+offset2)));
t2_0_56 += (1 - rt2_0_56) * (input[i] > (224+offset2));

int rt2_0_57 = (input[i] > (229+offset2));
int t2_0_57 = rt2_0_57 * (2 + (input[i] > (230+offset2)));
t2_0_57 += (1 - rt2_0_57) * (input[i] > (228+offset2));

int rt2_0_58 = (input[i] > (233+offset2));
int t2_0_58 = rt2_0_58 * (2 + (input[i] > (234+offset2)));
t2_0_58 += (1 - rt2_0_58) * (input[i] > (232+offset2));

int rt2_0_59 = (input[i] > (237+offset2));
int t2_0_59 = rt2_0_59 * (2 + (input[i] > (238+offset2)));
t2_0_59 += (1 - rt2_0_59) * (input[i] > (236+offset2));

int rt2_0_60 = (input[i] > (241+offset2));
int t2_0_60 = rt2_0_60 * (2 + (input[i] > (242+offset2)));
t2_0_60 += (1 - rt2_0_60) * (input[i] > (240+offset2));

int rt2_0_61 = (input[i] > (245+offset2));
int t2_0_61 = rt2_0_61 * (2 + (input[i] > (246+offset2)));
t2_0_61 += (1 - rt2_0_61) * (input[i] > (244+offset2));

int rt2_0_62 = (input[i] > (249+offset2));
int t2_0_62 = rt2_0_62 * (2 + (input[i] > (250+offset2)));
t2_0_62 += (1 - rt2_0_62) * (input[i] > (248+offset2));

int rt2_0_63 = (input[i] > (253+offset2));
int t2_0_63 = rt2_0_63 * (2 + (input[i] > (254+offset2)));
t2_0_63 += (1 - rt2_0_63) * (input[i] > (252+offset2));

int rt2_0_64 = (input[i] > (257+offset2));
int t2_0_64 = rt2_0_64 * (2 + (input[i] > (258+offset2)));
t2_0_64 += (1 - rt2_0_64) * (input[i] > (256+offset2));

int rt2_0_65 = (input[i] > (261+offset2));
int t2_0_65 = rt2_0_65 * (2 + (input[i] > (262+offset2)));
t2_0_65 += (1 - rt2_0_65) * (input[i] > (260+offset2));

int rt2_0_66 = (input[i] > (265+offset2));
int t2_0_66 = rt2_0_66 * (2 + (input[i] > (266+offset2)));
t2_0_66 += (1 - rt2_0_66) * (input[i] > (264+offset2));

int rt2_0_67 = (input[i] > (269+offset2));
int t2_0_67 = rt2_0_67 * (2 + (input[i] > (270+offset2)));
t2_0_67 += (1 - rt2_0_67) * (input[i] > (268+offset2));

int rt2_0_68 = (input[i] > (273+offset2));
int t2_0_68 = rt2_0_68 * (2 + (input[i] > (274+offset2)));
t2_0_68 += (1 - rt2_0_68) * (input[i] > (272+offset2));

int rt2_0_69 = (input[i] > (277+offset2));
int t2_0_69 = rt2_0_69 * (2 + (input[i] > (278+offset2)));
t2_0_69 += (1 - rt2_0_69) * (input[i] > (276+offset2));

int rt2_0_70 = (input[i] > (281+offset2));
int t2_0_70 = rt2_0_70 * (2 + (input[i] > (282+offset2)));
t2_0_70 += (1 - rt2_0_70) * (input[i] > (280+offset2));

int rt2_0_71 = (input[i] > (285+offset2));
int t2_0_71 = rt2_0_71 * (2 + (input[i] > (286+offset2)));
t2_0_71 += (1 - rt2_0_71) * (input[i] > (284+offset2));

int rt2_0_72 = (input[i] > (289+offset2));
int t2_0_72 = rt2_0_72 * (2 + (input[i] > (290+offset2)));
t2_0_72 += (1 - rt2_0_72) * (input[i] > (288+offset2));

int rt2_0_73 = (input[i] > (293+offset2));
int t2_0_73 = rt2_0_73 * (2 + (input[i] > (294+offset2)));
t2_0_73 += (1 - rt2_0_73) * (input[i] > (292+offset2));

int rt2_0_74 = (input[i] > (297+offset2));
int t2_0_74 = rt2_0_74 * (2 + (input[i] > (298+offset2)));
t2_0_74 += (1 - rt2_0_74) * (input[i] > (296+offset2));

int rt2_0_75 = (input[i] > (301+offset2));
int t2_0_75 = rt2_0_75 * (2 + (input[i] > (302+offset2)));
t2_0_75 += (1 - rt2_0_75) * (input[i] > (300+offset2));

int rt2_0_76 = (input[i] > (305+offset2));
int t2_0_76 = rt2_0_76 * (2 + (input[i] > (306+offset2)));
t2_0_76 += (1 - rt2_0_76) * (input[i] > (304+offset2));

int rt2_0_77 = (input[i] > (309+offset2));
int t2_0_77 = rt2_0_77 * (2 + (input[i] > (310+offset2)));
t2_0_77 += (1 - rt2_0_77) * (input[i] > (308+offset2));

int rt2_0_78 = (input[i] > (313+offset2));
int t2_0_78 = rt2_0_78 * (2 + (input[i] > (314+offset2)));
t2_0_78 += (1 - rt2_0_78) * (input[i] > (312+offset2));

int rt2_0_79 = (input[i] > (317+offset2));
int t2_0_79 = rt2_0_79 * (2 + (input[i] > (318+offset2)));
t2_0_79 += (1 - rt2_0_79) * (input[i] > (316+offset2));

int rt2_0_80 = (input[i] > (321+offset2));
int t2_0_80 = rt2_0_80 * (2 + (input[i] > (322+offset2)));
t2_0_80 += (1 - rt2_0_80) * (input[i] > (320+offset2));

int rt2_0_81 = (input[i] > (325+offset2));
int t2_0_81 = rt2_0_81 * (2 + (input[i] > (326+offset2)));
t2_0_81 += (1 - rt2_0_81) * (input[i] > (324+offset2));

int rt2_0_82 = (input[i] > (329+offset2));
int t2_0_82 = rt2_0_82 * (2 + (input[i] > (330+offset2)));
t2_0_82 += (1 - rt2_0_82) * (input[i] > (328+offset2));

int rt2_0_83 = (input[i] > (333+offset2));
int t2_0_83 = rt2_0_83 * (2 + (input[i] > (334+offset2)));
t2_0_83 += (1 - rt2_0_83) * (input[i] > (332+offset2));

int rt2_0_84 = (input[i] > (337+offset2));
int t2_0_84 = rt2_0_84 * (2 + (input[i] > (338+offset2)));
t2_0_84 += (1 - rt2_0_84) * (input[i] > (336+offset2));

int rt2_0_85 = (input[i] > (341+offset2));
int t2_0_85 = rt2_0_85 * (2 + (input[i] > (342+offset2)));
t2_0_85 += (1 - rt2_0_85) * (input[i] > (340+offset2));

int rt2_0_86 = (input[i] > (345+offset2));
int t2_0_86 = rt2_0_86 * (2 + (input[i] > (346+offset2)));
t2_0_86 += (1 - rt2_0_86) * (input[i] > (344+offset2));

int rt2_0_87 = (input[i] > (349+offset2));
int t2_0_87 = rt2_0_87 * (2 + (input[i] > (350+offset2)));
t2_0_87 += (1 - rt2_0_87) * (input[i] > (348+offset2));

int rt2_0_88 = (input[i] > (353+offset2));
int t2_0_88 = rt2_0_88 * (2 + (input[i] > (354+offset2)));
t2_0_88 += (1 - rt2_0_88) * (input[i] > (352+offset2));

int rt2_0_89 = (input[i] > (357+offset2));
int t2_0_89 = rt2_0_89 * (2 + (input[i] > (358+offset2)));
t2_0_89 += (1 - rt2_0_89) * (input[i] > (356+offset2));

int rt2_0_90 = (input[i] > (361+offset2));
int t2_0_90 = rt2_0_90 * (2 + (input[i] > (362+offset2)));
t2_0_90 += (1 - rt2_0_90) * (input[i] > (360+offset2));

int rt2_0_91 = (input[i] > (365+offset2));
int t2_0_91 = rt2_0_91 * (2 + (input[i] > (366+offset2)));
t2_0_91 += (1 - rt2_0_91) * (input[i] > (364+offset2));

int rt2_0_92 = (input[i] > (369+offset2));
int t2_0_92 = rt2_0_92 * (2 + (input[i] > (370+offset2)));
t2_0_92 += (1 - rt2_0_92) * (input[i] > (368+offset2));

int rt2_0_93 = (input[i] > (373+offset2));
int t2_0_93 = rt2_0_93 * (2 + (input[i] > (374+offset2)));
t2_0_93 += (1 - rt2_0_93) * (input[i] > (372+offset2));

int rt2_0_94 = (input[i] > (377+offset2));
int t2_0_94 = rt2_0_94 * (2 + (input[i] > (378+offset2)));
t2_0_94 += (1 - rt2_0_94) * (input[i] > (376+offset2));

int rt2_0_95 = (input[i] > (381+offset2));
int t2_0_95 = rt2_0_95 * (2 + (input[i] > (382+offset2)));
t2_0_95 += (1 - rt2_0_95) * (input[i] > (380+offset2));

int rt2_0_96 = (input[i] > (385+offset2));
int t2_0_96 = rt2_0_96 * (2 + (input[i] > (386+offset2)));
t2_0_96 += (1 - rt2_0_96) * (input[i] > (384+offset2));

int rt2_0_97 = (input[i] > (389+offset2));
int t2_0_97 = rt2_0_97 * (2 + (input[i] > (390+offset2)));
t2_0_97 += (1 - rt2_0_97) * (input[i] > (388+offset2));

int rt2_0_98 = (input[i] > (393+offset2));
int t2_0_98 = rt2_0_98 * (2 + (input[i] > (394+offset2)));
t2_0_98 += (1 - rt2_0_98) * (input[i] > (392+offset2));

int rt2_0_99 = (input[i] > (397+offset2));
int t2_0_99 = rt2_0_99 * (2 + (input[i] > (398+offset2)));
t2_0_99 += (1 - rt2_0_99) * (input[i] > (396+offset2));

int rt2_0_100 = (input[i] > (401+offset2));
int t2_0_100 = rt2_0_100 * (2 + (input[i] > (402+offset2)));
t2_0_100 += (1 - rt2_0_100) * (input[i] > (400+offset2));

int rt2_0_101 = (input[i] > (405+offset2));
int t2_0_101 = rt2_0_101 * (2 + (input[i] > (406+offset2)));
t2_0_101 += (1 - rt2_0_101) * (input[i] > (404+offset2));

int rt2_0_102 = (input[i] > (409+offset2));
int t2_0_102 = rt2_0_102 * (2 + (input[i] > (410+offset2)));
t2_0_102 += (1 - rt2_0_102) * (input[i] > (408+offset2));

int rt2_0_103 = (input[i] > (413+offset2));
int t2_0_103 = rt2_0_103 * (2 + (input[i] > (414+offset2)));
t2_0_103 += (1 - rt2_0_103) * (input[i] > (412+offset2));

int rt2_0_104 = (input[i] > (417+offset2));
int t2_0_104 = rt2_0_104 * (2 + (input[i] > (418+offset2)));
t2_0_104 += (1 - rt2_0_104) * (input[i] > (416+offset2));

int rt2_0_105 = (input[i] > (421+offset2));
int t2_0_105 = rt2_0_105 * (2 + (input[i] > (422+offset2)));
t2_0_105 += (1 - rt2_0_105) * (input[i] > (420+offset2));

int rt2_0_106 = (input[i] > (425+offset2));
int t2_0_106 = rt2_0_106 * (2 + (input[i] > (426+offset2)));
t2_0_106 += (1 - rt2_0_106) * (input[i] > (424+offset2));

int rt2_0_107 = (input[i] > (429+offset2));
int t2_0_107 = rt2_0_107 * (2 + (input[i] > (430+offset2)));
t2_0_107 += (1 - rt2_0_107) * (input[i] > (428+offset2));

int rt2_0_108 = (input[i] > (433+offset2));
int t2_0_108 = rt2_0_108 * (2 + (input[i] > (434+offset2)));
t2_0_108 += (1 - rt2_0_108) * (input[i] > (432+offset2));

int rt2_0_109 = (input[i] > (437+offset2));
int t2_0_109 = rt2_0_109 * (2 + (input[i] > (438+offset2)));
t2_0_109 += (1 - rt2_0_109) * (input[i] > (436+offset2));

int rt2_0_110 = (input[i] > (441+offset2));
int t2_0_110 = rt2_0_110 * (2 + (input[i] > (442+offset2)));
t2_0_110 += (1 - rt2_0_110) * (input[i] > (440+offset2));

int rt2_0_111 = (input[i] > (445+offset2));
int t2_0_111 = rt2_0_111 * (2 + (input[i] > (446+offset2)));
t2_0_111 += (1 - rt2_0_111) * (input[i] > (444+offset2));

int rt2_0_112 = (input[i] > (449+offset2));
int t2_0_112 = rt2_0_112 * (2 + (input[i] > (450+offset2)));
t2_0_112 += (1 - rt2_0_112) * (input[i] > (448+offset2));

int rt2_0_113 = (input[i] > (453+offset2));
int t2_0_113 = rt2_0_113 * (2 + (input[i] > (454+offset2)));
t2_0_113 += (1 - rt2_0_113) * (input[i] > (452+offset2));

int rt2_0_114 = (input[i] > (457+offset2));
int t2_0_114 = rt2_0_114 * (2 + (input[i] > (458+offset2)));
t2_0_114 += (1 - rt2_0_114) * (input[i] > (456+offset2));

int rt2_0_115 = (input[i] > (461+offset2));
int t2_0_115 = rt2_0_115 * (2 + (input[i] > (462+offset2)));
t2_0_115 += (1 - rt2_0_115) * (input[i] > (460+offset2));

int rt2_0_116 = (input[i] > (465+offset2));
int t2_0_116 = rt2_0_116 * (2 + (input[i] > (466+offset2)));
t2_0_116 += (1 - rt2_0_116) * (input[i] > (464+offset2));

int rt2_0_117 = (input[i] > (469+offset2));
int t2_0_117 = rt2_0_117 * (2 + (input[i] > (470+offset2)));
t2_0_117 += (1 - rt2_0_117) * (input[i] > (468+offset2));

int rt2_0_118 = (input[i] > (473+offset2));
int t2_0_118 = rt2_0_118 * (2 + (input[i] > (474+offset2)));
t2_0_118 += (1 - rt2_0_118) * (input[i] > (472+offset2));

int rt2_0_119 = (input[i] > (477+offset2));
int t2_0_119 = rt2_0_119 * (2 + (input[i] > (478+offset2)));
t2_0_119 += (1 - rt2_0_119) * (input[i] > (476+offset2));

int rt2_0_120 = (input[i] > (481+offset2));
int t2_0_120 = rt2_0_120 * (2 + (input[i] > (482+offset2)));
t2_0_120 += (1 - rt2_0_120) * (input[i] > (480+offset2));

int rt2_0_121 = (input[i] > (485+offset2));
int t2_0_121 = rt2_0_121 * (2 + (input[i] > (486+offset2)));
t2_0_121 += (1 - rt2_0_121) * (input[i] > (484+offset2));

int rt2_0_122 = (input[i] > (489+offset2));
int t2_0_122 = rt2_0_122 * (2 + (input[i] > (490+offset2)));
t2_0_122 += (1 - rt2_0_122) * (input[i] > (488+offset2));

int rt2_0_123 = (input[i] > (493+offset2));
int t2_0_123 = rt2_0_123 * (2 + (input[i] > (494+offset2)));
t2_0_123 += (1 - rt2_0_123) * (input[i] > (492+offset2));

int rt2_0_124 = (input[i] > (497+offset2));
int t2_0_124 = rt2_0_124 * (2 + (input[i] > (498+offset2)));
t2_0_124 += (1 - rt2_0_124) * (input[i] > (496+offset2));

int rt2_0_125 = (input[i] > (501+offset2));
int t2_0_125 = rt2_0_125 * (2 + (input[i] > (502+offset2)));
t2_0_125 += (1 - rt2_0_125) * (input[i] > (500+offset2));

int rt2_0_126 = (input[i] > (505+offset2));
int t2_0_126 = rt2_0_126 * (2 + (input[i] > (506+offset2)));
t2_0_126 += (1 - rt2_0_126) * (input[i] > (504+offset2));

int rt2_0_127 = (input[i] > (509+offset2));
int t2_0_127 = rt2_0_127 * (2 + (input[i] > (510+offset2)));
t2_0_127 += (1 - rt2_0_127) * (input[i] > (508+offset2));

int rt2_1_0 = (input[i] > (3+offset2));
int t2_1_0 = rt2_1_0 * (4 + t2_0_1);
t2_1_0 += (1 - rt2_1_0) * t2_0_0;

int rt2_1_1 = (input[i] > (11+offset2));
int t2_1_1 = rt2_1_1 * (4 + t2_0_3);
t2_1_1 += (1 - rt2_1_1) * t2_0_2;

int rt2_1_2 = (input[i] > (19+offset2));
int t2_1_2 = rt2_1_2 * (4 + t2_0_5);
t2_1_2 += (1 - rt2_1_2) * t2_0_4;

int rt2_1_3 = (input[i] > (27+offset2));
int t2_1_3 = rt2_1_3 * (4 + t2_0_7);
t2_1_3 += (1 - rt2_1_3) * t2_0_6;

int rt2_1_4 = (input[i] > (35+offset2));
int t2_1_4 = rt2_1_4 * (4 + t2_0_9);
t2_1_4 += (1 - rt2_1_4) * t2_0_8;

int rt2_1_5 = (input[i] > (43+offset2));
int t2_1_5 = rt2_1_5 * (4 + t2_0_11);
t2_1_5 += (1 - rt2_1_5) * t2_0_10;

int rt2_1_6 = (input[i] > (51+offset2));
int t2_1_6 = rt2_1_6 * (4 + t2_0_13);
t2_1_6 += (1 - rt2_1_6) * t2_0_12;

int rt2_1_7 = (input[i] > (59+offset2));
int t2_1_7 = rt2_1_7 * (4 + t2_0_15);
t2_1_7 += (1 - rt2_1_7) * t2_0_14;

int rt2_1_8 = (input[i] > (67+offset2));
int t2_1_8 = rt2_1_8 * (4 + t2_0_17);
t2_1_8 += (1 - rt2_1_8) * t2_0_16;

int rt2_1_9 = (input[i] > (75+offset2));
int t2_1_9 = rt2_1_9 * (4 + t2_0_19);
t2_1_9 += (1 - rt2_1_9) * t2_0_18;

int rt2_1_10 = (input[i] > (83+offset2));
int t2_1_10 = rt2_1_10 * (4 + t2_0_21);
t2_1_10 += (1 - rt2_1_10) * t2_0_20;

int rt2_1_11 = (input[i] > (91+offset2));
int t2_1_11 = rt2_1_11 * (4 + t2_0_23);
t2_1_11 += (1 - rt2_1_11) * t2_0_22;

int rt2_1_12 = (input[i] > (99+offset2));
int t2_1_12 = rt2_1_12 * (4 + t2_0_25);
t2_1_12 += (1 - rt2_1_12) * t2_0_24;

int rt2_1_13 = (input[i] > (107+offset2));
int t2_1_13 = rt2_1_13 * (4 + t2_0_27);
t2_1_13 += (1 - rt2_1_13) * t2_0_26;

int rt2_1_14 = (input[i] > (115+offset2));
int t2_1_14 = rt2_1_14 * (4 + t2_0_29);
t2_1_14 += (1 - rt2_1_14) * t2_0_28;

int rt2_1_15 = (input[i] > (123+offset2));
int t2_1_15 = rt2_1_15 * (4 + t2_0_31);
t2_1_15 += (1 - rt2_1_15) * t2_0_30;

int rt2_1_16 = (input[i] > (131+offset2));
int t2_1_16 = rt2_1_16 * (4 + t2_0_33);
t2_1_16 += (1 - rt2_1_16) * t2_0_32;

int rt2_1_17 = (input[i] > (139+offset2));
int t2_1_17 = rt2_1_17 * (4 + t2_0_35);
t2_1_17 += (1 - rt2_1_17) * t2_0_34;

int rt2_1_18 = (input[i] > (147+offset2));
int t2_1_18 = rt2_1_18 * (4 + t2_0_37);
t2_1_18 += (1 - rt2_1_18) * t2_0_36;

int rt2_1_19 = (input[i] > (155+offset2));
int t2_1_19 = rt2_1_19 * (4 + t2_0_39);
t2_1_19 += (1 - rt2_1_19) * t2_0_38;

int rt2_1_20 = (input[i] > (163+offset2));
int t2_1_20 = rt2_1_20 * (4 + t2_0_41);
t2_1_20 += (1 - rt2_1_20) * t2_0_40;

int rt2_1_21 = (input[i] > (171+offset2));
int t2_1_21 = rt2_1_21 * (4 + t2_0_43);
t2_1_21 += (1 - rt2_1_21) * t2_0_42;

int rt2_1_22 = (input[i] > (179+offset2));
int t2_1_22 = rt2_1_22 * (4 + t2_0_45);
t2_1_22 += (1 - rt2_1_22) * t2_0_44;

int rt2_1_23 = (input[i] > (187+offset2));
int t2_1_23 = rt2_1_23 * (4 + t2_0_47);
t2_1_23 += (1 - rt2_1_23) * t2_0_46;

int rt2_1_24 = (input[i] > (195+offset2));
int t2_1_24 = rt2_1_24 * (4 + t2_0_49);
t2_1_24 += (1 - rt2_1_24) * t2_0_48;

int rt2_1_25 = (input[i] > (203+offset2));
int t2_1_25 = rt2_1_25 * (4 + t2_0_51);
t2_1_25 += (1 - rt2_1_25) * t2_0_50;

int rt2_1_26 = (input[i] > (211+offset2));
int t2_1_26 = rt2_1_26 * (4 + t2_0_53);
t2_1_26 += (1 - rt2_1_26) * t2_0_52;

int rt2_1_27 = (input[i] > (219+offset2));
int t2_1_27 = rt2_1_27 * (4 + t2_0_55);
t2_1_27 += (1 - rt2_1_27) * t2_0_54;

int rt2_1_28 = (input[i] > (227+offset2));
int t2_1_28 = rt2_1_28 * (4 + t2_0_57);
t2_1_28 += (1 - rt2_1_28) * t2_0_56;

int rt2_1_29 = (input[i] > (235+offset2));
int t2_1_29 = rt2_1_29 * (4 + t2_0_59);
t2_1_29 += (1 - rt2_1_29) * t2_0_58;

int rt2_1_30 = (input[i] > (243+offset2));
int t2_1_30 = rt2_1_30 * (4 + t2_0_61);
t2_1_30 += (1 - rt2_1_30) * t2_0_60;

int rt2_1_31 = (input[i] > (251+offset2));
int t2_1_31 = rt2_1_31 * (4 + t2_0_63);
t2_1_31 += (1 - rt2_1_31) * t2_0_62;

int rt2_1_32 = (input[i] > (259+offset2));
int t2_1_32 = rt2_1_32 * (4 + t2_0_65);
t2_1_32 += (1 - rt2_1_32) * t2_0_64;

int rt2_1_33 = (input[i] > (267+offset2));
int t2_1_33 = rt2_1_33 * (4 + t2_0_67);
t2_1_33 += (1 - rt2_1_33) * t2_0_66;

int rt2_1_34 = (input[i] > (275+offset2));
int t2_1_34 = rt2_1_34 * (4 + t2_0_69);
t2_1_34 += (1 - rt2_1_34) * t2_0_68;

int rt2_1_35 = (input[i] > (283+offset2));
int t2_1_35 = rt2_1_35 * (4 + t2_0_71);
t2_1_35 += (1 - rt2_1_35) * t2_0_70;

int rt2_1_36 = (input[i] > (291+offset2));
int t2_1_36 = rt2_1_36 * (4 + t2_0_73);
t2_1_36 += (1 - rt2_1_36) * t2_0_72;

int rt2_1_37 = (input[i] > (299+offset2));
int t2_1_37 = rt2_1_37 * (4 + t2_0_75);
t2_1_37 += (1 - rt2_1_37) * t2_0_74;

int rt2_1_38 = (input[i] > (307+offset2));
int t2_1_38 = rt2_1_38 * (4 + t2_0_77);
t2_1_38 += (1 - rt2_1_38) * t2_0_76;

int rt2_1_39 = (input[i] > (315+offset2));
int t2_1_39 = rt2_1_39 * (4 + t2_0_79);
t2_1_39 += (1 - rt2_1_39) * t2_0_78;

int rt2_1_40 = (input[i] > (323+offset2));
int t2_1_40 = rt2_1_40 * (4 + t2_0_81);
t2_1_40 += (1 - rt2_1_40) * t2_0_80;

int rt2_1_41 = (input[i] > (331+offset2));
int t2_1_41 = rt2_1_41 * (4 + t2_0_83);
t2_1_41 += (1 - rt2_1_41) * t2_0_82;

int rt2_1_42 = (input[i] > (339+offset2));
int t2_1_42 = rt2_1_42 * (4 + t2_0_85);
t2_1_42 += (1 - rt2_1_42) * t2_0_84;

int rt2_1_43 = (input[i] > (347+offset2));
int t2_1_43 = rt2_1_43 * (4 + t2_0_87);
t2_1_43 += (1 - rt2_1_43) * t2_0_86;

int rt2_1_44 = (input[i] > (355+offset2));
int t2_1_44 = rt2_1_44 * (4 + t2_0_89);
t2_1_44 += (1 - rt2_1_44) * t2_0_88;

int rt2_1_45 = (input[i] > (363+offset2));
int t2_1_45 = rt2_1_45 * (4 + t2_0_91);
t2_1_45 += (1 - rt2_1_45) * t2_0_90;

int rt2_1_46 = (input[i] > (371+offset2));
int t2_1_46 = rt2_1_46 * (4 + t2_0_93);
t2_1_46 += (1 - rt2_1_46) * t2_0_92;

int rt2_1_47 = (input[i] > (379+offset2));
int t2_1_47 = rt2_1_47 * (4 + t2_0_95);
t2_1_47 += (1 - rt2_1_47) * t2_0_94;

int rt2_1_48 = (input[i] > (387+offset2));
int t2_1_48 = rt2_1_48 * (4 + t2_0_97);
t2_1_48 += (1 - rt2_1_48) * t2_0_96;

int rt2_1_49 = (input[i] > (395+offset2));
int t2_1_49 = rt2_1_49 * (4 + t2_0_99);
t2_1_49 += (1 - rt2_1_49) * t2_0_98;

int rt2_1_50 = (input[i] > (403+offset2));
int t2_1_50 = rt2_1_50 * (4 + t2_0_101);
t2_1_50 += (1 - rt2_1_50) * t2_0_100;

int rt2_1_51 = (input[i] > (411+offset2));
int t2_1_51 = rt2_1_51 * (4 + t2_0_103);
t2_1_51 += (1 - rt2_1_51) * t2_0_102;

int rt2_1_52 = (input[i] > (419+offset2));
int t2_1_52 = rt2_1_52 * (4 + t2_0_105);
t2_1_52 += (1 - rt2_1_52) * t2_0_104;

int rt2_1_53 = (input[i] > (427+offset2));
int t2_1_53 = rt2_1_53 * (4 + t2_0_107);
t2_1_53 += (1 - rt2_1_53) * t2_0_106;

int rt2_1_54 = (input[i] > (435+offset2));
int t2_1_54 = rt2_1_54 * (4 + t2_0_109);
t2_1_54 += (1 - rt2_1_54) * t2_0_108;

int rt2_1_55 = (input[i] > (443+offset2));
int t2_1_55 = rt2_1_55 * (4 + t2_0_111);
t2_1_55 += (1 - rt2_1_55) * t2_0_110;

int rt2_1_56 = (input[i] > (451+offset2));
int t2_1_56 = rt2_1_56 * (4 + t2_0_113);
t2_1_56 += (1 - rt2_1_56) * t2_0_112;

int rt2_1_57 = (input[i] > (459+offset2));
int t2_1_57 = rt2_1_57 * (4 + t2_0_115);
t2_1_57 += (1 - rt2_1_57) * t2_0_114;

int rt2_1_58 = (input[i] > (467+offset2));
int t2_1_58 = rt2_1_58 * (4 + t2_0_117);
t2_1_58 += (1 - rt2_1_58) * t2_0_116;

int rt2_1_59 = (input[i] > (475+offset2));
int t2_1_59 = rt2_1_59 * (4 + t2_0_119);
t2_1_59 += (1 - rt2_1_59) * t2_0_118;

int rt2_1_60 = (input[i] > (483+offset2));
int t2_1_60 = rt2_1_60 * (4 + t2_0_121);
t2_1_60 += (1 - rt2_1_60) * t2_0_120;

int rt2_1_61 = (input[i] > (491+offset2));
int t2_1_61 = rt2_1_61 * (4 + t2_0_123);
t2_1_61 += (1 - rt2_1_61) * t2_0_122;

int rt2_1_62 = (input[i] > (499+offset2));
int t2_1_62 = rt2_1_62 * (4 + t2_0_125);
t2_1_62 += (1 - rt2_1_62) * t2_0_124;

int rt2_1_63 = (input[i] > (507+offset2));
int t2_1_63 = rt2_1_63 * (4 + t2_0_127);
t2_1_63 += (1 - rt2_1_63) * t2_0_126;

int rt2_2_0 = (input[i] > (7+offset2));
int t2_2_0 = rt2_2_0 * (8 + t2_1_1);
t2_2_0 += (1 - rt2_2_0) * t2_1_0;

int rt2_2_1 = (input[i] > (23+offset2));
int t2_2_1 = rt2_2_1 * (8 + t2_1_3);
t2_2_1 += (1 - rt2_2_1) * t2_1_2;

int rt2_2_2 = (input[i] > (39+offset2));
int t2_2_2 = rt2_2_2 * (8 + t2_1_5);
t2_2_2 += (1 - rt2_2_2) * t2_1_4;

int rt2_2_3 = (input[i] > (55+offset2));
int t2_2_3 = rt2_2_3 * (8 + t2_1_7);
t2_2_3 += (1 - rt2_2_3) * t2_1_6;

int rt2_2_4 = (input[i] > (71+offset2));
int t2_2_4 = rt2_2_4 * (8 + t2_1_9);
t2_2_4 += (1 - rt2_2_4) * t2_1_8;

int rt2_2_5 = (input[i] > (87+offset2));
int t2_2_5 = rt2_2_5 * (8 + t2_1_11);
t2_2_5 += (1 - rt2_2_5) * t2_1_10;

int rt2_2_6 = (input[i] > (103+offset2));
int t2_2_6 = rt2_2_6 * (8 + t2_1_13);
t2_2_6 += (1 - rt2_2_6) * t2_1_12;

int rt2_2_7 = (input[i] > (119+offset2));
int t2_2_7 = rt2_2_7 * (8 + t2_1_15);
t2_2_7 += (1 - rt2_2_7) * t2_1_14;

int rt2_2_8 = (input[i] > (135+offset2));
int t2_2_8 = rt2_2_8 * (8 + t2_1_17);
t2_2_8 += (1 - rt2_2_8) * t2_1_16;

int rt2_2_9 = (input[i] > (151+offset2));
int t2_2_9 = rt2_2_9 * (8 + t2_1_19);
t2_2_9 += (1 - rt2_2_9) * t2_1_18;

int rt2_2_10 = (input[i] > (167+offset2));
int t2_2_10 = rt2_2_10 * (8 + t2_1_21);
t2_2_10 += (1 - rt2_2_10) * t2_1_20;

int rt2_2_11 = (input[i] > (183+offset2));
int t2_2_11 = rt2_2_11 * (8 + t2_1_23);
t2_2_11 += (1 - rt2_2_11) * t2_1_22;

int rt2_2_12 = (input[i] > (199+offset2));
int t2_2_12 = rt2_2_12 * (8 + t2_1_25);
t2_2_12 += (1 - rt2_2_12) * t2_1_24;

int rt2_2_13 = (input[i] > (215+offset2));
int t2_2_13 = rt2_2_13 * (8 + t2_1_27);
t2_2_13 += (1 - rt2_2_13) * t2_1_26;

int rt2_2_14 = (input[i] > (231+offset2));
int t2_2_14 = rt2_2_14 * (8 + t2_1_29);
t2_2_14 += (1 - rt2_2_14) * t2_1_28;

int rt2_2_15 = (input[i] > (247+offset2));
int t2_2_15 = rt2_2_15 * (8 + t2_1_31);
t2_2_15 += (1 - rt2_2_15) * t2_1_30;

int rt2_2_16 = (input[i] > (263+offset2));
int t2_2_16 = rt2_2_16 * (8 + t2_1_33);
t2_2_16 += (1 - rt2_2_16) * t2_1_32;

int rt2_2_17 = (input[i] > (279+offset2));
int t2_2_17 = rt2_2_17 * (8 + t2_1_35);
t2_2_17 += (1 - rt2_2_17) * t2_1_34;

int rt2_2_18 = (input[i] > (295+offset2));
int t2_2_18 = rt2_2_18 * (8 + t2_1_37);
t2_2_18 += (1 - rt2_2_18) * t2_1_36;

int rt2_2_19 = (input[i] > (311+offset2));
int t2_2_19 = rt2_2_19 * (8 + t2_1_39);
t2_2_19 += (1 - rt2_2_19) * t2_1_38;

int rt2_2_20 = (input[i] > (327+offset2));
int t2_2_20 = rt2_2_20 * (8 + t2_1_41);
t2_2_20 += (1 - rt2_2_20) * t2_1_40;

int rt2_2_21 = (input[i] > (343+offset2));
int t2_2_21 = rt2_2_21 * (8 + t2_1_43);
t2_2_21 += (1 - rt2_2_21) * t2_1_42;

int rt2_2_22 = (input[i] > (359+offset2));
int t2_2_22 = rt2_2_22 * (8 + t2_1_45);
t2_2_22 += (1 - rt2_2_22) * t2_1_44;

int rt2_2_23 = (input[i] > (375+offset2));
int t2_2_23 = rt2_2_23 * (8 + t2_1_47);
t2_2_23 += (1 - rt2_2_23) * t2_1_46;

int rt2_2_24 = (input[i] > (391+offset2));
int t2_2_24 = rt2_2_24 * (8 + t2_1_49);
t2_2_24 += (1 - rt2_2_24) * t2_1_48;

int rt2_2_25 = (input[i] > (407+offset2));
int t2_2_25 = rt2_2_25 * (8 + t2_1_51);
t2_2_25 += (1 - rt2_2_25) * t2_1_50;

int rt2_2_26 = (input[i] > (423+offset2));
int t2_2_26 = rt2_2_26 * (8 + t2_1_53);
t2_2_26 += (1 - rt2_2_26) * t2_1_52;

int rt2_2_27 = (input[i] > (439+offset2));
int t2_2_27 = rt2_2_27 * (8 + t2_1_55);
t2_2_27 += (1 - rt2_2_27) * t2_1_54;

int rt2_2_28 = (input[i] > (455+offset2));
int t2_2_28 = rt2_2_28 * (8 + t2_1_57);
t2_2_28 += (1 - rt2_2_28) * t2_1_56;

int rt2_2_29 = (input[i] > (471+offset2));
int t2_2_29 = rt2_2_29 * (8 + t2_1_59);
t2_2_29 += (1 - rt2_2_29) * t2_1_58;

int rt2_2_30 = (input[i] > (487+offset2));
int t2_2_30 = rt2_2_30 * (8 + t2_1_61);
t2_2_30 += (1 - rt2_2_30) * t2_1_60;

int rt2_2_31 = (input[i] > (503+offset2));
int t2_2_31 = rt2_2_31 * (8 + t2_1_63);
t2_2_31 += (1 - rt2_2_31) * t2_1_62;

int rt2_3_0 = (input[i] > (15+offset2));
int t2_3_0 = rt2_3_0 * (16 + t2_2_1);
t2_3_0 += (1 - rt2_3_0) * t2_2_0;

int rt2_3_1 = (input[i] > (47+offset2));
int t2_3_1 = rt2_3_1 * (16 + t2_2_3);
t2_3_1 += (1 - rt2_3_1) * t2_2_2;

int rt2_3_2 = (input[i] > (79+offset2));
int t2_3_2 = rt2_3_2 * (16 + t2_2_5);
t2_3_2 += (1 - rt2_3_2) * t2_2_4;

int rt2_3_3 = (input[i] > (111+offset2));
int t2_3_3 = rt2_3_3 * (16 + t2_2_7);
t2_3_3 += (1 - rt2_3_3) * t2_2_6;

int rt2_3_4 = (input[i] > (143+offset2));
int t2_3_4 = rt2_3_4 * (16 + t2_2_9);
t2_3_4 += (1 - rt2_3_4) * t2_2_8;

int rt2_3_5 = (input[i] > (175+offset2));
int t2_3_5 = rt2_3_5 * (16 + t2_2_11);
t2_3_5 += (1 - rt2_3_5) * t2_2_10;

int rt2_3_6 = (input[i] > (207+offset2));
int t2_3_6 = rt2_3_6 * (16 + t2_2_13);
t2_3_6 += (1 - rt2_3_6) * t2_2_12;

int rt2_3_7 = (input[i] > (239+offset2));
int t2_3_7 = rt2_3_7 * (16 + t2_2_15);
t2_3_7 += (1 - rt2_3_7) * t2_2_14;

int rt2_3_8 = (input[i] > (271+offset2));
int t2_3_8 = rt2_3_8 * (16 + t2_2_17);
t2_3_8 += (1 - rt2_3_8) * t2_2_16;

int rt2_3_9 = (input[i] > (303+offset2));
int t2_3_9 = rt2_3_9 * (16 + t2_2_19);
t2_3_9 += (1 - rt2_3_9) * t2_2_18;

int rt2_3_10 = (input[i] > (335+offset2));
int t2_3_10 = rt2_3_10 * (16 + t2_2_21);
t2_3_10 += (1 - rt2_3_10) * t2_2_20;

int rt2_3_11 = (input[i] > (367+offset2));
int t2_3_11 = rt2_3_11 * (16 + t2_2_23);
t2_3_11 += (1 - rt2_3_11) * t2_2_22;

int rt2_3_12 = (input[i] > (399+offset2));
int t2_3_12 = rt2_3_12 * (16 + t2_2_25);
t2_3_12 += (1 - rt2_3_12) * t2_2_24;

int rt2_3_13 = (input[i] > (431+offset2));
int t2_3_13 = rt2_3_13 * (16 + t2_2_27);
t2_3_13 += (1 - rt2_3_13) * t2_2_26;

int rt2_3_14 = (input[i] > (463+offset2));
int t2_3_14 = rt2_3_14 * (16 + t2_2_29);
t2_3_14 += (1 - rt2_3_14) * t2_2_28;

int rt2_3_15 = (input[i] > (495+offset2));
int t2_3_15 = rt2_3_15 * (16 + t2_2_31);
t2_3_15 += (1 - rt2_3_15) * t2_2_30;

int rt2_4_0 = (input[i] > (31+offset2));
int t2_4_0 = rt2_4_0 * (32 + t2_3_1);
t2_4_0 += (1 - rt2_4_0) * t2_3_0;

int rt2_4_1 = (input[i] > (95+offset2));
int t2_4_1 = rt2_4_1 * (32 + t2_3_3);
t2_4_1 += (1 - rt2_4_1) * t2_3_2;

int rt2_4_2 = (input[i] > (159+offset2));
int t2_4_2 = rt2_4_2 * (32 + t2_3_5);
t2_4_2 += (1 - rt2_4_2) * t2_3_4;

int rt2_4_3 = (input[i] > (223+offset2));
int t2_4_3 = rt2_4_3 * (32 + t2_3_7);
t2_4_3 += (1 - rt2_4_3) * t2_3_6;

int rt2_4_4 = (input[i] > (287+offset2));
int t2_4_4 = rt2_4_4 * (32 + t2_3_9);
t2_4_4 += (1 - rt2_4_4) * t2_3_8;

int rt2_4_5 = (input[i] > (351+offset2));
int t2_4_5 = rt2_4_5 * (32 + t2_3_11);
t2_4_5 += (1 - rt2_4_5) * t2_3_10;

int rt2_4_6 = (input[i] > (415+offset2));
int t2_4_6 = rt2_4_6 * (32 + t2_3_13);
t2_4_6 += (1 - rt2_4_6) * t2_3_12;

int rt2_4_7 = (input[i] > (479+offset2));
int t2_4_7 = rt2_4_7 * (32 + t2_3_15);
t2_4_7 += (1 - rt2_4_7) * t2_3_14;

int rt2_5_0 = (input[i] > (63+offset2));
int t2_5_0 = rt2_5_0 * (64 + t2_4_1);
t2_5_0 += (1 - rt2_5_0) * t2_4_0;

int rt2_5_1 = (input[i] > (191+offset2));
int t2_5_1 = rt2_5_1 * (64 + t2_4_3);
t2_5_1 += (1 - rt2_5_1) * t2_4_2;

int rt2_5_2 = (input[i] > (319+offset2));
int t2_5_2 = rt2_5_2 * (64 + t2_4_5);
t2_5_2 += (1 - rt2_5_2) * t2_4_4;

int rt2_5_3 = (input[i] > (447+offset2));
int t2_5_3 = rt2_5_3 * (64 + t2_4_7);
t2_5_3 += (1 - rt2_5_3) * t2_4_6;

int rt2_6_0 = (input[i] > (127+offset2));
int t2_6_0 = rt2_6_0 * (128 + t2_5_1);
t2_6_0 += (1 - rt2_6_0) * t2_5_0;

int rt2_6_1 = (input[i] > (383+offset2));
int t2_6_1 = rt2_6_1 * (128 + t2_5_3);
t2_6_1 += (1 - rt2_6_1) * t2_5_2;

int rt2_7_0 = (input[i] > (255+offset2));
int t2_7_0 = rt2_7_0 * (256 + t2_6_1);
t2_7_0 += (1 - rt2_7_0) * t2_6_0;

int rt3_0_0 = (input[i] > (1+offset3));
int t3_0_0 = rt3_0_0 * (2 + (input[i] > (2+offset3)));
t3_0_0 += (1 - rt3_0_0) * (input[i] > (0+offset3));

int rt3_0_1 = (input[i] > (5+offset3));
int t3_0_1 = rt3_0_1 * (2 + (input[i] > (6+offset3)));
t3_0_1 += (1 - rt3_0_1) * (input[i] > (4+offset3));

int rt3_0_2 = (input[i] > (9+offset3));
int t3_0_2 = rt3_0_2 * (2 + (input[i] > (10+offset3)));
t3_0_2 += (1 - rt3_0_2) * (input[i] > (8+offset3));

int rt3_0_3 = (input[i] > (13+offset3));
int t3_0_3 = rt3_0_3 * (2 + (input[i] > (14+offset3)));
t3_0_3 += (1 - rt3_0_3) * (input[i] > (12+offset3));

int rt3_0_4 = (input[i] > (17+offset3));
int t3_0_4 = rt3_0_4 * (2 + (input[i] > (18+offset3)));
t3_0_4 += (1 - rt3_0_4) * (input[i] > (16+offset3));

int rt3_0_5 = (input[i] > (21+offset3));
int t3_0_5 = rt3_0_5 * (2 + (input[i] > (22+offset3)));
t3_0_5 += (1 - rt3_0_5) * (input[i] > (20+offset3));

int rt3_0_6 = (input[i] > (25+offset3));
int t3_0_6 = rt3_0_6 * (2 + (input[i] > (26+offset3)));
t3_0_6 += (1 - rt3_0_6) * (input[i] > (24+offset3));

int rt3_0_7 = (input[i] > (29+offset3));
int t3_0_7 = rt3_0_7 * (2 + (input[i] > (30+offset3)));
t3_0_7 += (1 - rt3_0_7) * (input[i] > (28+offset3));

int rt3_0_8 = (input[i] > (33+offset3));
int t3_0_8 = rt3_0_8 * (2 + (input[i] > (34+offset3)));
t3_0_8 += (1 - rt3_0_8) * (input[i] > (32+offset3));

int rt3_0_9 = (input[i] > (37+offset3));
int t3_0_9 = rt3_0_9 * (2 + (input[i] > (38+offset3)));
t3_0_9 += (1 - rt3_0_9) * (input[i] > (36+offset3));

int rt3_0_10 = (input[i] > (41+offset3));
int t3_0_10 = rt3_0_10 * (2 + (input[i] > (42+offset3)));
t3_0_10 += (1 - rt3_0_10) * (input[i] > (40+offset3));

int rt3_0_11 = (input[i] > (45+offset3));
int t3_0_11 = rt3_0_11 * (2 + (input[i] > (46+offset3)));
t3_0_11 += (1 - rt3_0_11) * (input[i] > (44+offset3));

int rt3_0_12 = (input[i] > (49+offset3));
int t3_0_12 = rt3_0_12 * (2 + (input[i] > (50+offset3)));
t3_0_12 += (1 - rt3_0_12) * (input[i] > (48+offset3));

int rt3_0_13 = (input[i] > (53+offset3));
int t3_0_13 = rt3_0_13 * (2 + (input[i] > (54+offset3)));
t3_0_13 += (1 - rt3_0_13) * (input[i] > (52+offset3));

int rt3_0_14 = (input[i] > (57+offset3));
int t3_0_14 = rt3_0_14 * (2 + (input[i] > (58+offset3)));
t3_0_14 += (1 - rt3_0_14) * (input[i] > (56+offset3));

int rt3_0_15 = (input[i] > (61+offset3));
int t3_0_15 = rt3_0_15 * (2 + (input[i] > (62+offset3)));
t3_0_15 += (1 - rt3_0_15) * (input[i] > (60+offset3));

int rt3_0_16 = (input[i] > (65+offset3));
int t3_0_16 = rt3_0_16 * (2 + (input[i] > (66+offset3)));
t3_0_16 += (1 - rt3_0_16) * (input[i] > (64+offset3));

int rt3_0_17 = (input[i] > (69+offset3));
int t3_0_17 = rt3_0_17 * (2 + (input[i] > (70+offset3)));
t3_0_17 += (1 - rt3_0_17) * (input[i] > (68+offset3));

int rt3_0_18 = (input[i] > (73+offset3));
int t3_0_18 = rt3_0_18 * (2 + (input[i] > (74+offset3)));
t3_0_18 += (1 - rt3_0_18) * (input[i] > (72+offset3));

int rt3_0_19 = (input[i] > (77+offset3));
int t3_0_19 = rt3_0_19 * (2 + (input[i] > (78+offset3)));
t3_0_19 += (1 - rt3_0_19) * (input[i] > (76+offset3));

int rt3_0_20 = (input[i] > (81+offset3));
int t3_0_20 = rt3_0_20 * (2 + (input[i] > (82+offset3)));
t3_0_20 += (1 - rt3_0_20) * (input[i] > (80+offset3));

int rt3_0_21 = (input[i] > (85+offset3));
int t3_0_21 = rt3_0_21 * (2 + (input[i] > (86+offset3)));
t3_0_21 += (1 - rt3_0_21) * (input[i] > (84+offset3));

int rt3_0_22 = (input[i] > (89+offset3));
int t3_0_22 = rt3_0_22 * (2 + (input[i] > (90+offset3)));
t3_0_22 += (1 - rt3_0_22) * (input[i] > (88+offset3));

int rt3_0_23 = (input[i] > (93+offset3));
int t3_0_23 = rt3_0_23 * (2 + (input[i] > (94+offset3)));
t3_0_23 += (1 - rt3_0_23) * (input[i] > (92+offset3));

int rt3_0_24 = (input[i] > (97+offset3));
int t3_0_24 = rt3_0_24 * (2 + (input[i] > (98+offset3)));
t3_0_24 += (1 - rt3_0_24) * (input[i] > (96+offset3));

int rt3_0_25 = (input[i] > (101+offset3));
int t3_0_25 = rt3_0_25 * (2 + (input[i] > (102+offset3)));
t3_0_25 += (1 - rt3_0_25) * (input[i] > (100+offset3));

int rt3_0_26 = (input[i] > (105+offset3));
int t3_0_26 = rt3_0_26 * (2 + (input[i] > (106+offset3)));
t3_0_26 += (1 - rt3_0_26) * (input[i] > (104+offset3));

int rt3_0_27 = (input[i] > (109+offset3));
int t3_0_27 = rt3_0_27 * (2 + (input[i] > (110+offset3)));
t3_0_27 += (1 - rt3_0_27) * (input[i] > (108+offset3));

int rt3_0_28 = (input[i] > (113+offset3));
int t3_0_28 = rt3_0_28 * (2 + (input[i] > (114+offset3)));
t3_0_28 += (1 - rt3_0_28) * (input[i] > (112+offset3));

int rt3_0_29 = (input[i] > (117+offset3));
int t3_0_29 = rt3_0_29 * (2 + (input[i] > (118+offset3)));
t3_0_29 += (1 - rt3_0_29) * (input[i] > (116+offset3));

int rt3_0_30 = (input[i] > (121+offset3));
int t3_0_30 = rt3_0_30 * (2 + (input[i] > (122+offset3)));
t3_0_30 += (1 - rt3_0_30) * (input[i] > (120+offset3));

int rt3_0_31 = (input[i] > (125+offset3));
int t3_0_31 = rt3_0_31 * (2 + (input[i] > (126+offset3)));
t3_0_31 += (1 - rt3_0_31) * (input[i] > (124+offset3));

int rt3_0_32 = (input[i] > (129+offset3));
int t3_0_32 = rt3_0_32 * (2 + (input[i] > (130+offset3)));
t3_0_32 += (1 - rt3_0_32) * (input[i] > (128+offset3));

int rt3_0_33 = (input[i] > (133+offset3));
int t3_0_33 = rt3_0_33 * (2 + (input[i] > (134+offset3)));
t3_0_33 += (1 - rt3_0_33) * (input[i] > (132+offset3));

int rt3_0_34 = (input[i] > (137+offset3));
int t3_0_34 = rt3_0_34 * (2 + (input[i] > (138+offset3)));
t3_0_34 += (1 - rt3_0_34) * (input[i] > (136+offset3));

int rt3_0_35 = (input[i] > (141+offset3));
int t3_0_35 = rt3_0_35 * (2 + (input[i] > (142+offset3)));
t3_0_35 += (1 - rt3_0_35) * (input[i] > (140+offset3));

int rt3_0_36 = (input[i] > (145+offset3));
int t3_0_36 = rt3_0_36 * (2 + (input[i] > (146+offset3)));
t3_0_36 += (1 - rt3_0_36) * (input[i] > (144+offset3));

int rt3_0_37 = (input[i] > (149+offset3));
int t3_0_37 = rt3_0_37 * (2 + (input[i] > (150+offset3)));
t3_0_37 += (1 - rt3_0_37) * (input[i] > (148+offset3));

int rt3_0_38 = (input[i] > (153+offset3));
int t3_0_38 = rt3_0_38 * (2 + (input[i] > (154+offset3)));
t3_0_38 += (1 - rt3_0_38) * (input[i] > (152+offset3));

int rt3_0_39 = (input[i] > (157+offset3));
int t3_0_39 = rt3_0_39 * (2 + (input[i] > (158+offset3)));
t3_0_39 += (1 - rt3_0_39) * (input[i] > (156+offset3));

int rt3_0_40 = (input[i] > (161+offset3));
int t3_0_40 = rt3_0_40 * (2 + (input[i] > (162+offset3)));
t3_0_40 += (1 - rt3_0_40) * (input[i] > (160+offset3));

int rt3_0_41 = (input[i] > (165+offset3));
int t3_0_41 = rt3_0_41 * (2 + (input[i] > (166+offset3)));
t3_0_41 += (1 - rt3_0_41) * (input[i] > (164+offset3));

int rt3_0_42 = (input[i] > (169+offset3));
int t3_0_42 = rt3_0_42 * (2 + (input[i] > (170+offset3)));
t3_0_42 += (1 - rt3_0_42) * (input[i] > (168+offset3));

int rt3_0_43 = (input[i] > (173+offset3));
int t3_0_43 = rt3_0_43 * (2 + (input[i] > (174+offset3)));
t3_0_43 += (1 - rt3_0_43) * (input[i] > (172+offset3));

int rt3_0_44 = (input[i] > (177+offset3));
int t3_0_44 = rt3_0_44 * (2 + (input[i] > (178+offset3)));
t3_0_44 += (1 - rt3_0_44) * (input[i] > (176+offset3));

int rt3_0_45 = (input[i] > (181+offset3));
int t3_0_45 = rt3_0_45 * (2 + (input[i] > (182+offset3)));
t3_0_45 += (1 - rt3_0_45) * (input[i] > (180+offset3));

int rt3_0_46 = (input[i] > (185+offset3));
int t3_0_46 = rt3_0_46 * (2 + (input[i] > (186+offset3)));
t3_0_46 += (1 - rt3_0_46) * (input[i] > (184+offset3));

int rt3_0_47 = (input[i] > (189+offset3));
int t3_0_47 = rt3_0_47 * (2 + (input[i] > (190+offset3)));
t3_0_47 += (1 - rt3_0_47) * (input[i] > (188+offset3));

int rt3_0_48 = (input[i] > (193+offset3));
int t3_0_48 = rt3_0_48 * (2 + (input[i] > (194+offset3)));
t3_0_48 += (1 - rt3_0_48) * (input[i] > (192+offset3));

int rt3_0_49 = (input[i] > (197+offset3));
int t3_0_49 = rt3_0_49 * (2 + (input[i] > (198+offset3)));
t3_0_49 += (1 - rt3_0_49) * (input[i] > (196+offset3));

int rt3_0_50 = (input[i] > (201+offset3));
int t3_0_50 = rt3_0_50 * (2 + (input[i] > (202+offset3)));
t3_0_50 += (1 - rt3_0_50) * (input[i] > (200+offset3));

int rt3_0_51 = (input[i] > (205+offset3));
int t3_0_51 = rt3_0_51 * (2 + (input[i] > (206+offset3)));
t3_0_51 += (1 - rt3_0_51) * (input[i] > (204+offset3));

int rt3_0_52 = (input[i] > (209+offset3));
int t3_0_52 = rt3_0_52 * (2 + (input[i] > (210+offset3)));
t3_0_52 += (1 - rt3_0_52) * (input[i] > (208+offset3));

int rt3_0_53 = (input[i] > (213+offset3));
int t3_0_53 = rt3_0_53 * (2 + (input[i] > (214+offset3)));
t3_0_53 += (1 - rt3_0_53) * (input[i] > (212+offset3));

int rt3_0_54 = (input[i] > (217+offset3));
int t3_0_54 = rt3_0_54 * (2 + (input[i] > (218+offset3)));
t3_0_54 += (1 - rt3_0_54) * (input[i] > (216+offset3));

int rt3_0_55 = (input[i] > (221+offset3));
int t3_0_55 = rt3_0_55 * (2 + (input[i] > (222+offset3)));
t3_0_55 += (1 - rt3_0_55) * (input[i] > (220+offset3));

int rt3_0_56 = (input[i] > (225+offset3));
int t3_0_56 = rt3_0_56 * (2 + (input[i] > (226+offset3)));
t3_0_56 += (1 - rt3_0_56) * (input[i] > (224+offset3));

int rt3_0_57 = (input[i] > (229+offset3));
int t3_0_57 = rt3_0_57 * (2 + (input[i] > (230+offset3)));
t3_0_57 += (1 - rt3_0_57) * (input[i] > (228+offset3));

int rt3_0_58 = (input[i] > (233+offset3));
int t3_0_58 = rt3_0_58 * (2 + (input[i] > (234+offset3)));
t3_0_58 += (1 - rt3_0_58) * (input[i] > (232+offset3));

int rt3_0_59 = (input[i] > (237+offset3));
int t3_0_59 = rt3_0_59 * (2 + (input[i] > (238+offset3)));
t3_0_59 += (1 - rt3_0_59) * (input[i] > (236+offset3));

int rt3_0_60 = (input[i] > (241+offset3));
int t3_0_60 = rt3_0_60 * (2 + (input[i] > (242+offset3)));
t3_0_60 += (1 - rt3_0_60) * (input[i] > (240+offset3));

int rt3_0_61 = (input[i] > (245+offset3));
int t3_0_61 = rt3_0_61 * (2 + (input[i] > (246+offset3)));
t3_0_61 += (1 - rt3_0_61) * (input[i] > (244+offset3));

int rt3_0_62 = (input[i] > (249+offset3));
int t3_0_62 = rt3_0_62 * (2 + (input[i] > (250+offset3)));
t3_0_62 += (1 - rt3_0_62) * (input[i] > (248+offset3));

int rt3_0_63 = (input[i] > (253+offset3));
int t3_0_63 = rt3_0_63 * (2 + (input[i] > (254+offset3)));
t3_0_63 += (1 - rt3_0_63) * (input[i] > (252+offset3));

int rt3_0_64 = (input[i] > (257+offset3));
int t3_0_64 = rt3_0_64 * (2 + (input[i] > (258+offset3)));
t3_0_64 += (1 - rt3_0_64) * (input[i] > (256+offset3));

int rt3_0_65 = (input[i] > (261+offset3));
int t3_0_65 = rt3_0_65 * (2 + (input[i] > (262+offset3)));
t3_0_65 += (1 - rt3_0_65) * (input[i] > (260+offset3));

int rt3_0_66 = (input[i] > (265+offset3));
int t3_0_66 = rt3_0_66 * (2 + (input[i] > (266+offset3)));
t3_0_66 += (1 - rt3_0_66) * (input[i] > (264+offset3));

int rt3_0_67 = (input[i] > (269+offset3));
int t3_0_67 = rt3_0_67 * (2 + (input[i] > (270+offset3)));
t3_0_67 += (1 - rt3_0_67) * (input[i] > (268+offset3));

int rt3_0_68 = (input[i] > (273+offset3));
int t3_0_68 = rt3_0_68 * (2 + (input[i] > (274+offset3)));
t3_0_68 += (1 - rt3_0_68) * (input[i] > (272+offset3));

int rt3_0_69 = (input[i] > (277+offset3));
int t3_0_69 = rt3_0_69 * (2 + (input[i] > (278+offset3)));
t3_0_69 += (1 - rt3_0_69) * (input[i] > (276+offset3));

int rt3_0_70 = (input[i] > (281+offset3));
int t3_0_70 = rt3_0_70 * (2 + (input[i] > (282+offset3)));
t3_0_70 += (1 - rt3_0_70) * (input[i] > (280+offset3));

int rt3_0_71 = (input[i] > (285+offset3));
int t3_0_71 = rt3_0_71 * (2 + (input[i] > (286+offset3)));
t3_0_71 += (1 - rt3_0_71) * (input[i] > (284+offset3));

int rt3_0_72 = (input[i] > (289+offset3));
int t3_0_72 = rt3_0_72 * (2 + (input[i] > (290+offset3)));
t3_0_72 += (1 - rt3_0_72) * (input[i] > (288+offset3));

int rt3_0_73 = (input[i] > (293+offset3));
int t3_0_73 = rt3_0_73 * (2 + (input[i] > (294+offset3)));
t3_0_73 += (1 - rt3_0_73) * (input[i] > (292+offset3));

int rt3_0_74 = (input[i] > (297+offset3));
int t3_0_74 = rt3_0_74 * (2 + (input[i] > (298+offset3)));
t3_0_74 += (1 - rt3_0_74) * (input[i] > (296+offset3));

int rt3_0_75 = (input[i] > (301+offset3));
int t3_0_75 = rt3_0_75 * (2 + (input[i] > (302+offset3)));
t3_0_75 += (1 - rt3_0_75) * (input[i] > (300+offset3));

int rt3_0_76 = (input[i] > (305+offset3));
int t3_0_76 = rt3_0_76 * (2 + (input[i] > (306+offset3)));
t3_0_76 += (1 - rt3_0_76) * (input[i] > (304+offset3));

int rt3_0_77 = (input[i] > (309+offset3));
int t3_0_77 = rt3_0_77 * (2 + (input[i] > (310+offset3)));
t3_0_77 += (1 - rt3_0_77) * (input[i] > (308+offset3));

int rt3_0_78 = (input[i] > (313+offset3));
int t3_0_78 = rt3_0_78 * (2 + (input[i] > (314+offset3)));
t3_0_78 += (1 - rt3_0_78) * (input[i] > (312+offset3));

int rt3_0_79 = (input[i] > (317+offset3));
int t3_0_79 = rt3_0_79 * (2 + (input[i] > (318+offset3)));
t3_0_79 += (1 - rt3_0_79) * (input[i] > (316+offset3));

int rt3_0_80 = (input[i] > (321+offset3));
int t3_0_80 = rt3_0_80 * (2 + (input[i] > (322+offset3)));
t3_0_80 += (1 - rt3_0_80) * (input[i] > (320+offset3));

int rt3_0_81 = (input[i] > (325+offset3));
int t3_0_81 = rt3_0_81 * (2 + (input[i] > (326+offset3)));
t3_0_81 += (1 - rt3_0_81) * (input[i] > (324+offset3));

int rt3_0_82 = (input[i] > (329+offset3));
int t3_0_82 = rt3_0_82 * (2 + (input[i] > (330+offset3)));
t3_0_82 += (1 - rt3_0_82) * (input[i] > (328+offset3));

int rt3_0_83 = (input[i] > (333+offset3));
int t3_0_83 = rt3_0_83 * (2 + (input[i] > (334+offset3)));
t3_0_83 += (1 - rt3_0_83) * (input[i] > (332+offset3));

int rt3_0_84 = (input[i] > (337+offset3));
int t3_0_84 = rt3_0_84 * (2 + (input[i] > (338+offset3)));
t3_0_84 += (1 - rt3_0_84) * (input[i] > (336+offset3));

int rt3_0_85 = (input[i] > (341+offset3));
int t3_0_85 = rt3_0_85 * (2 + (input[i] > (342+offset3)));
t3_0_85 += (1 - rt3_0_85) * (input[i] > (340+offset3));

int rt3_0_86 = (input[i] > (345+offset3));
int t3_0_86 = rt3_0_86 * (2 + (input[i] > (346+offset3)));
t3_0_86 += (1 - rt3_0_86) * (input[i] > (344+offset3));

int rt3_0_87 = (input[i] > (349+offset3));
int t3_0_87 = rt3_0_87 * (2 + (input[i] > (350+offset3)));
t3_0_87 += (1 - rt3_0_87) * (input[i] > (348+offset3));

int rt3_0_88 = (input[i] > (353+offset3));
int t3_0_88 = rt3_0_88 * (2 + (input[i] > (354+offset3)));
t3_0_88 += (1 - rt3_0_88) * (input[i] > (352+offset3));

int rt3_0_89 = (input[i] > (357+offset3));
int t3_0_89 = rt3_0_89 * (2 + (input[i] > (358+offset3)));
t3_0_89 += (1 - rt3_0_89) * (input[i] > (356+offset3));

int rt3_0_90 = (input[i] > (361+offset3));
int t3_0_90 = rt3_0_90 * (2 + (input[i] > (362+offset3)));
t3_0_90 += (1 - rt3_0_90) * (input[i] > (360+offset3));

int rt3_0_91 = (input[i] > (365+offset3));
int t3_0_91 = rt3_0_91 * (2 + (input[i] > (366+offset3)));
t3_0_91 += (1 - rt3_0_91) * (input[i] > (364+offset3));

int rt3_0_92 = (input[i] > (369+offset3));
int t3_0_92 = rt3_0_92 * (2 + (input[i] > (370+offset3)));
t3_0_92 += (1 - rt3_0_92) * (input[i] > (368+offset3));

int rt3_0_93 = (input[i] > (373+offset3));
int t3_0_93 = rt3_0_93 * (2 + (input[i] > (374+offset3)));
t3_0_93 += (1 - rt3_0_93) * (input[i] > (372+offset3));

int rt3_0_94 = (input[i] > (377+offset3));
int t3_0_94 = rt3_0_94 * (2 + (input[i] > (378+offset3)));
t3_0_94 += (1 - rt3_0_94) * (input[i] > (376+offset3));

int rt3_0_95 = (input[i] > (381+offset3));
int t3_0_95 = rt3_0_95 * (2 + (input[i] > (382+offset3)));
t3_0_95 += (1 - rt3_0_95) * (input[i] > (380+offset3));

int rt3_0_96 = (input[i] > (385+offset3));
int t3_0_96 = rt3_0_96 * (2 + (input[i] > (386+offset3)));
t3_0_96 += (1 - rt3_0_96) * (input[i] > (384+offset3));

int rt3_0_97 = (input[i] > (389+offset3));
int t3_0_97 = rt3_0_97 * (2 + (input[i] > (390+offset3)));
t3_0_97 += (1 - rt3_0_97) * (input[i] > (388+offset3));

int rt3_0_98 = (input[i] > (393+offset3));
int t3_0_98 = rt3_0_98 * (2 + (input[i] > (394+offset3)));
t3_0_98 += (1 - rt3_0_98) * (input[i] > (392+offset3));

int rt3_0_99 = (input[i] > (397+offset3));
int t3_0_99 = rt3_0_99 * (2 + (input[i] > (398+offset3)));
t3_0_99 += (1 - rt3_0_99) * (input[i] > (396+offset3));

int rt3_0_100 = (input[i] > (401+offset3));
int t3_0_100 = rt3_0_100 * (2 + (input[i] > (402+offset3)));
t3_0_100 += (1 - rt3_0_100) * (input[i] > (400+offset3));

int rt3_0_101 = (input[i] > (405+offset3));
int t3_0_101 = rt3_0_101 * (2 + (input[i] > (406+offset3)));
t3_0_101 += (1 - rt3_0_101) * (input[i] > (404+offset3));

int rt3_0_102 = (input[i] > (409+offset3));
int t3_0_102 = rt3_0_102 * (2 + (input[i] > (410+offset3)));
t3_0_102 += (1 - rt3_0_102) * (input[i] > (408+offset3));

int rt3_0_103 = (input[i] > (413+offset3));
int t3_0_103 = rt3_0_103 * (2 + (input[i] > (414+offset3)));
t3_0_103 += (1 - rt3_0_103) * (input[i] > (412+offset3));

int rt3_0_104 = (input[i] > (417+offset3));
int t3_0_104 = rt3_0_104 * (2 + (input[i] > (418+offset3)));
t3_0_104 += (1 - rt3_0_104) * (input[i] > (416+offset3));

int rt3_0_105 = (input[i] > (421+offset3));
int t3_0_105 = rt3_0_105 * (2 + (input[i] > (422+offset3)));
t3_0_105 += (1 - rt3_0_105) * (input[i] > (420+offset3));

int rt3_0_106 = (input[i] > (425+offset3));
int t3_0_106 = rt3_0_106 * (2 + (input[i] > (426+offset3)));
t3_0_106 += (1 - rt3_0_106) * (input[i] > (424+offset3));

int rt3_0_107 = (input[i] > (429+offset3));
int t3_0_107 = rt3_0_107 * (2 + (input[i] > (430+offset3)));
t3_0_107 += (1 - rt3_0_107) * (input[i] > (428+offset3));

int rt3_0_108 = (input[i] > (433+offset3));
int t3_0_108 = rt3_0_108 * (2 + (input[i] > (434+offset3)));
t3_0_108 += (1 - rt3_0_108) * (input[i] > (432+offset3));

int rt3_0_109 = (input[i] > (437+offset3));
int t3_0_109 = rt3_0_109 * (2 + (input[i] > (438+offset3)));
t3_0_109 += (1 - rt3_0_109) * (input[i] > (436+offset3));

int rt3_0_110 = (input[i] > (441+offset3));
int t3_0_110 = rt3_0_110 * (2 + (input[i] > (442+offset3)));
t3_0_110 += (1 - rt3_0_110) * (input[i] > (440+offset3));

int rt3_0_111 = (input[i] > (445+offset3));
int t3_0_111 = rt3_0_111 * (2 + (input[i] > (446+offset3)));
t3_0_111 += (1 - rt3_0_111) * (input[i] > (444+offset3));

int rt3_0_112 = (input[i] > (449+offset3));
int t3_0_112 = rt3_0_112 * (2 + (input[i] > (450+offset3)));
t3_0_112 += (1 - rt3_0_112) * (input[i] > (448+offset3));

int rt3_0_113 = (input[i] > (453+offset3));
int t3_0_113 = rt3_0_113 * (2 + (input[i] > (454+offset3)));
t3_0_113 += (1 - rt3_0_113) * (input[i] > (452+offset3));

int rt3_0_114 = (input[i] > (457+offset3));
int t3_0_114 = rt3_0_114 * (2 + (input[i] > (458+offset3)));
t3_0_114 += (1 - rt3_0_114) * (input[i] > (456+offset3));

int rt3_0_115 = (input[i] > (461+offset3));
int t3_0_115 = rt3_0_115 * (2 + (input[i] > (462+offset3)));
t3_0_115 += (1 - rt3_0_115) * (input[i] > (460+offset3));

int rt3_0_116 = (input[i] > (465+offset3));
int t3_0_116 = rt3_0_116 * (2 + (input[i] > (466+offset3)));
t3_0_116 += (1 - rt3_0_116) * (input[i] > (464+offset3));

int rt3_0_117 = (input[i] > (469+offset3));
int t3_0_117 = rt3_0_117 * (2 + (input[i] > (470+offset3)));
t3_0_117 += (1 - rt3_0_117) * (input[i] > (468+offset3));

int rt3_0_118 = (input[i] > (473+offset3));
int t3_0_118 = rt3_0_118 * (2 + (input[i] > (474+offset3)));
t3_0_118 += (1 - rt3_0_118) * (input[i] > (472+offset3));

int rt3_0_119 = (input[i] > (477+offset3));
int t3_0_119 = rt3_0_119 * (2 + (input[i] > (478+offset3)));
t3_0_119 += (1 - rt3_0_119) * (input[i] > (476+offset3));

int rt3_0_120 = (input[i] > (481+offset3));
int t3_0_120 = rt3_0_120 * (2 + (input[i] > (482+offset3)));
t3_0_120 += (1 - rt3_0_120) * (input[i] > (480+offset3));

int rt3_0_121 = (input[i] > (485+offset3));
int t3_0_121 = rt3_0_121 * (2 + (input[i] > (486+offset3)));
t3_0_121 += (1 - rt3_0_121) * (input[i] > (484+offset3));

int rt3_0_122 = (input[i] > (489+offset3));
int t3_0_122 = rt3_0_122 * (2 + (input[i] > (490+offset3)));
t3_0_122 += (1 - rt3_0_122) * (input[i] > (488+offset3));

int rt3_0_123 = (input[i] > (493+offset3));
int t3_0_123 = rt3_0_123 * (2 + (input[i] > (494+offset3)));
t3_0_123 += (1 - rt3_0_123) * (input[i] > (492+offset3));

int rt3_0_124 = (input[i] > (497+offset3));
int t3_0_124 = rt3_0_124 * (2 + (input[i] > (498+offset3)));
t3_0_124 += (1 - rt3_0_124) * (input[i] > (496+offset3));

int rt3_0_125 = (input[i] > (501+offset3));
int t3_0_125 = rt3_0_125 * (2 + (input[i] > (502+offset3)));
t3_0_125 += (1 - rt3_0_125) * (input[i] > (500+offset3));

int rt3_0_126 = (input[i] > (505+offset3));
int t3_0_126 = rt3_0_126 * (2 + (input[i] > (506+offset3)));
t3_0_126 += (1 - rt3_0_126) * (input[i] > (504+offset3));

int rt3_0_127 = (input[i] > (509+offset3));
int t3_0_127 = rt3_0_127 * (2 + (input[i] > (510+offset3)));
t3_0_127 += (1 - rt3_0_127) * (input[i] > (508+offset3));

int rt3_1_0 = (input[i] > (3+offset3));
int t3_1_0 = rt3_1_0 * (4 + t3_0_1);
t3_1_0 += (1 - rt3_1_0) * t3_0_0;

int rt3_1_1 = (input[i] > (11+offset3));
int t3_1_1 = rt3_1_1 * (4 + t3_0_3);
t3_1_1 += (1 - rt3_1_1) * t3_0_2;

int rt3_1_2 = (input[i] > (19+offset3));
int t3_1_2 = rt3_1_2 * (4 + t3_0_5);
t3_1_2 += (1 - rt3_1_2) * t3_0_4;

int rt3_1_3 = (input[i] > (27+offset3));
int t3_1_3 = rt3_1_3 * (4 + t3_0_7);
t3_1_3 += (1 - rt3_1_3) * t3_0_6;

int rt3_1_4 = (input[i] > (35+offset3));
int t3_1_4 = rt3_1_4 * (4 + t3_0_9);
t3_1_4 += (1 - rt3_1_4) * t3_0_8;

int rt3_1_5 = (input[i] > (43+offset3));
int t3_1_5 = rt3_1_5 * (4 + t3_0_11);
t3_1_5 += (1 - rt3_1_5) * t3_0_10;

int rt3_1_6 = (input[i] > (51+offset3));
int t3_1_6 = rt3_1_6 * (4 + t3_0_13);
t3_1_6 += (1 - rt3_1_6) * t3_0_12;

int rt3_1_7 = (input[i] > (59+offset3));
int t3_1_7 = rt3_1_7 * (4 + t3_0_15);
t3_1_7 += (1 - rt3_1_7) * t3_0_14;

int rt3_1_8 = (input[i] > (67+offset3));
int t3_1_8 = rt3_1_8 * (4 + t3_0_17);
t3_1_8 += (1 - rt3_1_8) * t3_0_16;

int rt3_1_9 = (input[i] > (75+offset3));
int t3_1_9 = rt3_1_9 * (4 + t3_0_19);
t3_1_9 += (1 - rt3_1_9) * t3_0_18;

int rt3_1_10 = (input[i] > (83+offset3));
int t3_1_10 = rt3_1_10 * (4 + t3_0_21);
t3_1_10 += (1 - rt3_1_10) * t3_0_20;

int rt3_1_11 = (input[i] > (91+offset3));
int t3_1_11 = rt3_1_11 * (4 + t3_0_23);
t3_1_11 += (1 - rt3_1_11) * t3_0_22;

int rt3_1_12 = (input[i] > (99+offset3));
int t3_1_12 = rt3_1_12 * (4 + t3_0_25);
t3_1_12 += (1 - rt3_1_12) * t3_0_24;

int rt3_1_13 = (input[i] > (107+offset3));
int t3_1_13 = rt3_1_13 * (4 + t3_0_27);
t3_1_13 += (1 - rt3_1_13) * t3_0_26;

int rt3_1_14 = (input[i] > (115+offset3));
int t3_1_14 = rt3_1_14 * (4 + t3_0_29);
t3_1_14 += (1 - rt3_1_14) * t3_0_28;

int rt3_1_15 = (input[i] > (123+offset3));
int t3_1_15 = rt3_1_15 * (4 + t3_0_31);
t3_1_15 += (1 - rt3_1_15) * t3_0_30;

int rt3_1_16 = (input[i] > (131+offset3));
int t3_1_16 = rt3_1_16 * (4 + t3_0_33);
t3_1_16 += (1 - rt3_1_16) * t3_0_32;

int rt3_1_17 = (input[i] > (139+offset3));
int t3_1_17 = rt3_1_17 * (4 + t3_0_35);
t3_1_17 += (1 - rt3_1_17) * t3_0_34;

int rt3_1_18 = (input[i] > (147+offset3));
int t3_1_18 = rt3_1_18 * (4 + t3_0_37);
t3_1_18 += (1 - rt3_1_18) * t3_0_36;

int rt3_1_19 = (input[i] > (155+offset3));
int t3_1_19 = rt3_1_19 * (4 + t3_0_39);
t3_1_19 += (1 - rt3_1_19) * t3_0_38;

int rt3_1_20 = (input[i] > (163+offset3));
int t3_1_20 = rt3_1_20 * (4 + t3_0_41);
t3_1_20 += (1 - rt3_1_20) * t3_0_40;

int rt3_1_21 = (input[i] > (171+offset3));
int t3_1_21 = rt3_1_21 * (4 + t3_0_43);
t3_1_21 += (1 - rt3_1_21) * t3_0_42;

int rt3_1_22 = (input[i] > (179+offset3));
int t3_1_22 = rt3_1_22 * (4 + t3_0_45);
t3_1_22 += (1 - rt3_1_22) * t3_0_44;

int rt3_1_23 = (input[i] > (187+offset3));
int t3_1_23 = rt3_1_23 * (4 + t3_0_47);
t3_1_23 += (1 - rt3_1_23) * t3_0_46;

int rt3_1_24 = (input[i] > (195+offset3));
int t3_1_24 = rt3_1_24 * (4 + t3_0_49);
t3_1_24 += (1 - rt3_1_24) * t3_0_48;

int rt3_1_25 = (input[i] > (203+offset3));
int t3_1_25 = rt3_1_25 * (4 + t3_0_51);
t3_1_25 += (1 - rt3_1_25) * t3_0_50;

int rt3_1_26 = (input[i] > (211+offset3));
int t3_1_26 = rt3_1_26 * (4 + t3_0_53);
t3_1_26 += (1 - rt3_1_26) * t3_0_52;

int rt3_1_27 = (input[i] > (219+offset3));
int t3_1_27 = rt3_1_27 * (4 + t3_0_55);
t3_1_27 += (1 - rt3_1_27) * t3_0_54;

int rt3_1_28 = (input[i] > (227+offset3));
int t3_1_28 = rt3_1_28 * (4 + t3_0_57);
t3_1_28 += (1 - rt3_1_28) * t3_0_56;

int rt3_1_29 = (input[i] > (235+offset3));
int t3_1_29 = rt3_1_29 * (4 + t3_0_59);
t3_1_29 += (1 - rt3_1_29) * t3_0_58;

int rt3_1_30 = (input[i] > (243+offset3));
int t3_1_30 = rt3_1_30 * (4 + t3_0_61);
t3_1_30 += (1 - rt3_1_30) * t3_0_60;

int rt3_1_31 = (input[i] > (251+offset3));
int t3_1_31 = rt3_1_31 * (4 + t3_0_63);
t3_1_31 += (1 - rt3_1_31) * t3_0_62;

int rt3_1_32 = (input[i] > (259+offset3));
int t3_1_32 = rt3_1_32 * (4 + t3_0_65);
t3_1_32 += (1 - rt3_1_32) * t3_0_64;

int rt3_1_33 = (input[i] > (267+offset3));
int t3_1_33 = rt3_1_33 * (4 + t3_0_67);
t3_1_33 += (1 - rt3_1_33) * t3_0_66;

int rt3_1_34 = (input[i] > (275+offset3));
int t3_1_34 = rt3_1_34 * (4 + t3_0_69);
t3_1_34 += (1 - rt3_1_34) * t3_0_68;

int rt3_1_35 = (input[i] > (283+offset3));
int t3_1_35 = rt3_1_35 * (4 + t3_0_71);
t3_1_35 += (1 - rt3_1_35) * t3_0_70;

int rt3_1_36 = (input[i] > (291+offset3));
int t3_1_36 = rt3_1_36 * (4 + t3_0_73);
t3_1_36 += (1 - rt3_1_36) * t3_0_72;

int rt3_1_37 = (input[i] > (299+offset3));
int t3_1_37 = rt3_1_37 * (4 + t3_0_75);
t3_1_37 += (1 - rt3_1_37) * t3_0_74;

int rt3_1_38 = (input[i] > (307+offset3));
int t3_1_38 = rt3_1_38 * (4 + t3_0_77);
t3_1_38 += (1 - rt3_1_38) * t3_0_76;

int rt3_1_39 = (input[i] > (315+offset3));
int t3_1_39 = rt3_1_39 * (4 + t3_0_79);
t3_1_39 += (1 - rt3_1_39) * t3_0_78;

int rt3_1_40 = (input[i] > (323+offset3));
int t3_1_40 = rt3_1_40 * (4 + t3_0_81);
t3_1_40 += (1 - rt3_1_40) * t3_0_80;

int rt3_1_41 = (input[i] > (331+offset3));
int t3_1_41 = rt3_1_41 * (4 + t3_0_83);
t3_1_41 += (1 - rt3_1_41) * t3_0_82;

int rt3_1_42 = (input[i] > (339+offset3));
int t3_1_42 = rt3_1_42 * (4 + t3_0_85);
t3_1_42 += (1 - rt3_1_42) * t3_0_84;

int rt3_1_43 = (input[i] > (347+offset3));
int t3_1_43 = rt3_1_43 * (4 + t3_0_87);
t3_1_43 += (1 - rt3_1_43) * t3_0_86;

int rt3_1_44 = (input[i] > (355+offset3));
int t3_1_44 = rt3_1_44 * (4 + t3_0_89);
t3_1_44 += (1 - rt3_1_44) * t3_0_88;

int rt3_1_45 = (input[i] > (363+offset3));
int t3_1_45 = rt3_1_45 * (4 + t3_0_91);
t3_1_45 += (1 - rt3_1_45) * t3_0_90;

int rt3_1_46 = (input[i] > (371+offset3));
int t3_1_46 = rt3_1_46 * (4 + t3_0_93);
t3_1_46 += (1 - rt3_1_46) * t3_0_92;

int rt3_1_47 = (input[i] > (379+offset3));
int t3_1_47 = rt3_1_47 * (4 + t3_0_95);
t3_1_47 += (1 - rt3_1_47) * t3_0_94;

int rt3_1_48 = (input[i] > (387+offset3));
int t3_1_48 = rt3_1_48 * (4 + t3_0_97);
t3_1_48 += (1 - rt3_1_48) * t3_0_96;

int rt3_1_49 = (input[i] > (395+offset3));
int t3_1_49 = rt3_1_49 * (4 + t3_0_99);
t3_1_49 += (1 - rt3_1_49) * t3_0_98;

int rt3_1_50 = (input[i] > (403+offset3));
int t3_1_50 = rt3_1_50 * (4 + t3_0_101);
t3_1_50 += (1 - rt3_1_50) * t3_0_100;

int rt3_1_51 = (input[i] > (411+offset3));
int t3_1_51 = rt3_1_51 * (4 + t3_0_103);
t3_1_51 += (1 - rt3_1_51) * t3_0_102;

int rt3_1_52 = (input[i] > (419+offset3));
int t3_1_52 = rt3_1_52 * (4 + t3_0_105);
t3_1_52 += (1 - rt3_1_52) * t3_0_104;

int rt3_1_53 = (input[i] > (427+offset3));
int t3_1_53 = rt3_1_53 * (4 + t3_0_107);
t3_1_53 += (1 - rt3_1_53) * t3_0_106;

int rt3_1_54 = (input[i] > (435+offset3));
int t3_1_54 = rt3_1_54 * (4 + t3_0_109);
t3_1_54 += (1 - rt3_1_54) * t3_0_108;

int rt3_1_55 = (input[i] > (443+offset3));
int t3_1_55 = rt3_1_55 * (4 + t3_0_111);
t3_1_55 += (1 - rt3_1_55) * t3_0_110;

int rt3_1_56 = (input[i] > (451+offset3));
int t3_1_56 = rt3_1_56 * (4 + t3_0_113);
t3_1_56 += (1 - rt3_1_56) * t3_0_112;

int rt3_1_57 = (input[i] > (459+offset3));
int t3_1_57 = rt3_1_57 * (4 + t3_0_115);
t3_1_57 += (1 - rt3_1_57) * t3_0_114;

int rt3_1_58 = (input[i] > (467+offset3));
int t3_1_58 = rt3_1_58 * (4 + t3_0_117);
t3_1_58 += (1 - rt3_1_58) * t3_0_116;

int rt3_1_59 = (input[i] > (475+offset3));
int t3_1_59 = rt3_1_59 * (4 + t3_0_119);
t3_1_59 += (1 - rt3_1_59) * t3_0_118;

int rt3_1_60 = (input[i] > (483+offset3));
int t3_1_60 = rt3_1_60 * (4 + t3_0_121);
t3_1_60 += (1 - rt3_1_60) * t3_0_120;

int rt3_1_61 = (input[i] > (491+offset3));
int t3_1_61 = rt3_1_61 * (4 + t3_0_123);
t3_1_61 += (1 - rt3_1_61) * t3_0_122;

int rt3_1_62 = (input[i] > (499+offset3));
int t3_1_62 = rt3_1_62 * (4 + t3_0_125);
t3_1_62 += (1 - rt3_1_62) * t3_0_124;

int rt3_1_63 = (input[i] > (507+offset3));
int t3_1_63 = rt3_1_63 * (4 + t3_0_127);
t3_1_63 += (1 - rt3_1_63) * t3_0_126;

int rt3_2_0 = (input[i] > (7+offset3));
int t3_2_0 = rt3_2_0 * (8 + t3_1_1);
t3_2_0 += (1 - rt3_2_0) * t3_1_0;

int rt3_2_1 = (input[i] > (23+offset3));
int t3_2_1 = rt3_2_1 * (8 + t3_1_3);
t3_2_1 += (1 - rt3_2_1) * t3_1_2;

int rt3_2_2 = (input[i] > (39+offset3));
int t3_2_2 = rt3_2_2 * (8 + t3_1_5);
t3_2_2 += (1 - rt3_2_2) * t3_1_4;

int rt3_2_3 = (input[i] > (55+offset3));
int t3_2_3 = rt3_2_3 * (8 + t3_1_7);
t3_2_3 += (1 - rt3_2_3) * t3_1_6;

int rt3_2_4 = (input[i] > (71+offset3));
int t3_2_4 = rt3_2_4 * (8 + t3_1_9);
t3_2_4 += (1 - rt3_2_4) * t3_1_8;

int rt3_2_5 = (input[i] > (87+offset3));
int t3_2_5 = rt3_2_5 * (8 + t3_1_11);
t3_2_5 += (1 - rt3_2_5) * t3_1_10;

int rt3_2_6 = (input[i] > (103+offset3));
int t3_2_6 = rt3_2_6 * (8 + t3_1_13);
t3_2_6 += (1 - rt3_2_6) * t3_1_12;

int rt3_2_7 = (input[i] > (119+offset3));
int t3_2_7 = rt3_2_7 * (8 + t3_1_15);
t3_2_7 += (1 - rt3_2_7) * t3_1_14;

int rt3_2_8 = (input[i] > (135+offset3));
int t3_2_8 = rt3_2_8 * (8 + t3_1_17);
t3_2_8 += (1 - rt3_2_8) * t3_1_16;

int rt3_2_9 = (input[i] > (151+offset3));
int t3_2_9 = rt3_2_9 * (8 + t3_1_19);
t3_2_9 += (1 - rt3_2_9) * t3_1_18;

int rt3_2_10 = (input[i] > (167+offset3));
int t3_2_10 = rt3_2_10 * (8 + t3_1_21);
t3_2_10 += (1 - rt3_2_10) * t3_1_20;

int rt3_2_11 = (input[i] > (183+offset3));
int t3_2_11 = rt3_2_11 * (8 + t3_1_23);
t3_2_11 += (1 - rt3_2_11) * t3_1_22;

int rt3_2_12 = (input[i] > (199+offset3));
int t3_2_12 = rt3_2_12 * (8 + t3_1_25);
t3_2_12 += (1 - rt3_2_12) * t3_1_24;

int rt3_2_13 = (input[i] > (215+offset3));
int t3_2_13 = rt3_2_13 * (8 + t3_1_27);
t3_2_13 += (1 - rt3_2_13) * t3_1_26;

int rt3_2_14 = (input[i] > (231+offset3));
int t3_2_14 = rt3_2_14 * (8 + t3_1_29);
t3_2_14 += (1 - rt3_2_14) * t3_1_28;

int rt3_2_15 = (input[i] > (247+offset3));
int t3_2_15 = rt3_2_15 * (8 + t3_1_31);
t3_2_15 += (1 - rt3_2_15) * t3_1_30;

int rt3_2_16 = (input[i] > (263+offset3));
int t3_2_16 = rt3_2_16 * (8 + t3_1_33);
t3_2_16 += (1 - rt3_2_16) * t3_1_32;

int rt3_2_17 = (input[i] > (279+offset3));
int t3_2_17 = rt3_2_17 * (8 + t3_1_35);
t3_2_17 += (1 - rt3_2_17) * t3_1_34;

int rt3_2_18 = (input[i] > (295+offset3));
int t3_2_18 = rt3_2_18 * (8 + t3_1_37);
t3_2_18 += (1 - rt3_2_18) * t3_1_36;

int rt3_2_19 = (input[i] > (311+offset3));
int t3_2_19 = rt3_2_19 * (8 + t3_1_39);
t3_2_19 += (1 - rt3_2_19) * t3_1_38;

int rt3_2_20 = (input[i] > (327+offset3));
int t3_2_20 = rt3_2_20 * (8 + t3_1_41);
t3_2_20 += (1 - rt3_2_20) * t3_1_40;

int rt3_2_21 = (input[i] > (343+offset3));
int t3_2_21 = rt3_2_21 * (8 + t3_1_43);
t3_2_21 += (1 - rt3_2_21) * t3_1_42;

int rt3_2_22 = (input[i] > (359+offset3));
int t3_2_22 = rt3_2_22 * (8 + t3_1_45);
t3_2_22 += (1 - rt3_2_22) * t3_1_44;

int rt3_2_23 = (input[i] > (375+offset3));
int t3_2_23 = rt3_2_23 * (8 + t3_1_47);
t3_2_23 += (1 - rt3_2_23) * t3_1_46;

int rt3_2_24 = (input[i] > (391+offset3));
int t3_2_24 = rt3_2_24 * (8 + t3_1_49);
t3_2_24 += (1 - rt3_2_24) * t3_1_48;

int rt3_2_25 = (input[i] > (407+offset3));
int t3_2_25 = rt3_2_25 * (8 + t3_1_51);
t3_2_25 += (1 - rt3_2_25) * t3_1_50;

int rt3_2_26 = (input[i] > (423+offset3));
int t3_2_26 = rt3_2_26 * (8 + t3_1_53);
t3_2_26 += (1 - rt3_2_26) * t3_1_52;

int rt3_2_27 = (input[i] > (439+offset3));
int t3_2_27 = rt3_2_27 * (8 + t3_1_55);
t3_2_27 += (1 - rt3_2_27) * t3_1_54;

int rt3_2_28 = (input[i] > (455+offset3));
int t3_2_28 = rt3_2_28 * (8 + t3_1_57);
t3_2_28 += (1 - rt3_2_28) * t3_1_56;

int rt3_2_29 = (input[i] > (471+offset3));
int t3_2_29 = rt3_2_29 * (8 + t3_1_59);
t3_2_29 += (1 - rt3_2_29) * t3_1_58;

int rt3_2_30 = (input[i] > (487+offset3));
int t3_2_30 = rt3_2_30 * (8 + t3_1_61);
t3_2_30 += (1 - rt3_2_30) * t3_1_60;

int rt3_2_31 = (input[i] > (503+offset3));
int t3_2_31 = rt3_2_31 * (8 + t3_1_63);
t3_2_31 += (1 - rt3_2_31) * t3_1_62;

int rt3_3_0 = (input[i] > (15+offset3));
int t3_3_0 = rt3_3_0 * (16 + t3_2_1);
t3_3_0 += (1 - rt3_3_0) * t3_2_0;

int rt3_3_1 = (input[i] > (47+offset3));
int t3_3_1 = rt3_3_1 * (16 + t3_2_3);
t3_3_1 += (1 - rt3_3_1) * t3_2_2;

int rt3_3_2 = (input[i] > (79+offset3));
int t3_3_2 = rt3_3_2 * (16 + t3_2_5);
t3_3_2 += (1 - rt3_3_2) * t3_2_4;

int rt3_3_3 = (input[i] > (111+offset3));
int t3_3_3 = rt3_3_3 * (16 + t3_2_7);
t3_3_3 += (1 - rt3_3_3) * t3_2_6;

int rt3_3_4 = (input[i] > (143+offset3));
int t3_3_4 = rt3_3_4 * (16 + t3_2_9);
t3_3_4 += (1 - rt3_3_4) * t3_2_8;

int rt3_3_5 = (input[i] > (175+offset3));
int t3_3_5 = rt3_3_5 * (16 + t3_2_11);
t3_3_5 += (1 - rt3_3_5) * t3_2_10;

int rt3_3_6 = (input[i] > (207+offset3));
int t3_3_6 = rt3_3_6 * (16 + t3_2_13);
t3_3_6 += (1 - rt3_3_6) * t3_2_12;

int rt3_3_7 = (input[i] > (239+offset3));
int t3_3_7 = rt3_3_7 * (16 + t3_2_15);
t3_3_7 += (1 - rt3_3_7) * t3_2_14;

int rt3_3_8 = (input[i] > (271+offset3));
int t3_3_8 = rt3_3_8 * (16 + t3_2_17);
t3_3_8 += (1 - rt3_3_8) * t3_2_16;

int rt3_3_9 = (input[i] > (303+offset3));
int t3_3_9 = rt3_3_9 * (16 + t3_2_19);
t3_3_9 += (1 - rt3_3_9) * t3_2_18;

int rt3_3_10 = (input[i] > (335+offset3));
int t3_3_10 = rt3_3_10 * (16 + t3_2_21);
t3_3_10 += (1 - rt3_3_10) * t3_2_20;

int rt3_3_11 = (input[i] > (367+offset3));
int t3_3_11 = rt3_3_11 * (16 + t3_2_23);
t3_3_11 += (1 - rt3_3_11) * t3_2_22;

int rt3_3_12 = (input[i] > (399+offset3));
int t3_3_12 = rt3_3_12 * (16 + t3_2_25);
t3_3_12 += (1 - rt3_3_12) * t3_2_24;

int rt3_3_13 = (input[i] > (431+offset3));
int t3_3_13 = rt3_3_13 * (16 + t3_2_27);
t3_3_13 += (1 - rt3_3_13) * t3_2_26;

int rt3_3_14 = (input[i] > (463+offset3));
int t3_3_14 = rt3_3_14 * (16 + t3_2_29);
t3_3_14 += (1 - rt3_3_14) * t3_2_28;

int rt3_3_15 = (input[i] > (495+offset3));
int t3_3_15 = rt3_3_15 * (16 + t3_2_31);
t3_3_15 += (1 - rt3_3_15) * t3_2_30;

int rt3_4_0 = (input[i] > (31+offset3));
int t3_4_0 = rt3_4_0 * (32 + t3_3_1);
t3_4_0 += (1 - rt3_4_0) * t3_3_0;

int rt3_4_1 = (input[i] > (95+offset3));
int t3_4_1 = rt3_4_1 * (32 + t3_3_3);
t3_4_1 += (1 - rt3_4_1) * t3_3_2;

int rt3_4_2 = (input[i] > (159+offset3));
int t3_4_2 = rt3_4_2 * (32 + t3_3_5);
t3_4_2 += (1 - rt3_4_2) * t3_3_4;

int rt3_4_3 = (input[i] > (223+offset3));
int t3_4_3 = rt3_4_3 * (32 + t3_3_7);
t3_4_3 += (1 - rt3_4_3) * t3_3_6;

int rt3_4_4 = (input[i] > (287+offset3));
int t3_4_4 = rt3_4_4 * (32 + t3_3_9);
t3_4_4 += (1 - rt3_4_4) * t3_3_8;

int rt3_4_5 = (input[i] > (351+offset3));
int t3_4_5 = rt3_4_5 * (32 + t3_3_11);
t3_4_5 += (1 - rt3_4_5) * t3_3_10;

int rt3_4_6 = (input[i] > (415+offset3));
int t3_4_6 = rt3_4_6 * (32 + t3_3_13);
t3_4_6 += (1 - rt3_4_6) * t3_3_12;

int rt3_4_7 = (input[i] > (479+offset3));
int t3_4_7 = rt3_4_7 * (32 + t3_3_15);
t3_4_7 += (1 - rt3_4_7) * t3_3_14;

int rt3_5_0 = (input[i] > (63+offset3));
int t3_5_0 = rt3_5_0 * (64 + t3_4_1);
t3_5_0 += (1 - rt3_5_0) * t3_4_0;

int rt3_5_1 = (input[i] > (191+offset3));
int t3_5_1 = rt3_5_1 * (64 + t3_4_3);
t3_5_1 += (1 - rt3_5_1) * t3_4_2;

int rt3_5_2 = (input[i] > (319+offset3));
int t3_5_2 = rt3_5_2 * (64 + t3_4_5);
t3_5_2 += (1 - rt3_5_2) * t3_4_4;

int rt3_5_3 = (input[i] > (447+offset3));
int t3_5_3 = rt3_5_3 * (64 + t3_4_7);
t3_5_3 += (1 - rt3_5_3) * t3_4_6;

int rt3_6_0 = (input[i] > (127+offset3));
int t3_6_0 = rt3_6_0 * (128 + t3_5_1);
t3_6_0 += (1 - rt3_6_0) * t3_5_0;

int rt3_6_1 = (input[i] > (383+offset3));
int t3_6_1 = rt3_6_1 * (128 + t3_5_3);
t3_6_1 += (1 - rt3_6_1) * t3_5_2;

int rt3_7_0 = (input[i] > (255+offset3));
int t3_7_0 = rt3_7_0 * (256 + t3_6_1);
t3_7_0 += (1 - rt3_7_0) * t3_6_0;

vR[i] = t0_7_0+ (t0_7_0 == t1_7_0)+ (t0_7_0 == t2_7_0)+ (t0_7_0 == t3_7_0);
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
