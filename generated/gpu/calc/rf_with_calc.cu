#include "hip/hip_runtime.h"
#include <sys/time.h>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}
#define N_ELEM 150


#include <hip/hip_runtime.h>
#include <stdio.h>


void initialData(float *ip, int size){

  for (int i = 0; i < size; i++){
      ip[i] = (float)( rand() & 15 );
  }
  return;
}
FILE *inFile;
FILE *outFile;

void readInFile(float **ip) {
    FILE *file = fopen("assets/datasets/Iris.csv","r");
    if (file == NULL) {
        printf("[CUDA]: Failed to open the file.\n");
        exit(1);
    }

    char line[1000];
    
    int row = 0;
    fgets(line, sizeof(line), file);
    while (fgets(line, sizeof(line), file)) {
        char *token = strtok(line, ",");
        
        int column = 0;
        while (token != NULL) {
            ip[column][row] = atof(token);
            token = strtok(NULL, ",");
            column++;
        }

        row++;
        if(row > N_ELEM) break;
    }

    fclose(file);
}
void writeOutFile(int value){
    outFile = fopen("out_rf.csv","a");
    fprintf(outFile, "%d\n", value);
    fclose(outFile);
} 

void registerTime(float value){
    outFile = fopen("results/results.csv","a");
    fprintf(outFile, ",%.2f", value);
    fclose(outFile);
}__global__ void RF_kernel(const int offset0, const int offset1, const int offset2, const int n, const int offset0, const int offset1, const int offset2, float *vR)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Class[0] = 0;
	Class[1] = 0;
	Class[2] = 0;
	if (i < N) {

		__shared__ int classesTree0[] = {0,0,0,0,0,0,0,0,1,1,1,2,2,2,2,2};

		int rt0_0_0 = (F3[i] <= (0.8 + offset0));
		int t0_0_0 = rt0_0_0 * (2 + F3[i] <= (0.8 + offset0));
		t0_0_0 += (1 - rt0_0_0) * ( F3[i] <= (0.8 + offset0));


		int rt0_0_1 = (F3[i] <= (0.8 + offset0));
		int t0_0_1 = rt0_0_1 * (2 + F3[i] <= (0.8 + offset0));
		t0_0_1 += (1 - rt0_0_1) * ( F3[i] <= (0.8 + offset0));


		int rt0_0_2 = (F2[i] <= (4.75 + offset0));
		int t0_0_2 = rt0_0_2 * (2 + F1[i] <= (2.6 + offset0));
		t0_0_2 += (1 - rt0_0_2) * ( F2[i] <= (4.75 + offset0));


		int rt0_0_3 = (F2[i] <= (4.95 + offset0));
		int t0_0_3 = rt0_0_3 * (2 + F2[i] <= (4.95 + offset0));
		t0_0_3 += (1 - rt0_0_3) * ( F2[i] <= (4.95 + offset0));


		int rt0_1_0 = (F3[i] <= (0.8 + offset0));
		int t0_1_0 = rt0_1_0 * (8 + t0_0_1);
		t0_1_0 += (1 - rt0_1_0) * t0_0_0;


		int rt0_1_1 = (F2[i] <= (4.95 + offset0));
		int t0_1_1 = rt0_1_1 * (8 + t0_0_3);
		t0_1_1 += (1 - rt0_1_1) * t0_0_2;


		int rt0_2_0 = (F3[i] <= (0.8 + offset0));
		int t0_2_0 = rt0_2_0 * (12 + t0_1_1);
		t0_2_0 += (1 - rt0_2_0) * t0_1_0;


		Class[classesTree0[t0_2_0]]++;


		__shared__ int classesTree1[] = {0,0,0,0,1,2,2,2};

		int rt1_0_0 = (F3[i] <= (0.8 + offset1));
		int t1_0_0 = rt1_0_0 * (2 + F3[i] <= (0.8 + offset1));
		t1_0_0 += (1 - rt1_0_0) * ( F3[i] <= (0.8 + offset1));


		int rt1_0_1 = (F2[i] <= (4.95 + offset1));
		int t1_0_1 = rt1_0_1 * (2 + F2[i] <= (4.95 + offset1));
		t1_0_1 += (1 - rt1_0_1) * ( F3[i] <= (1.6 + offset1));


		int rt1_1_0 = (F3[i] <= (0.8 + offset1));
		int t1_1_0 = rt1_1_0 * (8 + t1_0_1);
		t1_1_0 += (1 - rt1_1_0) * t1_0_0;


		Class[classesTree1[t1_1_0]]++;


		__shared__ int classesTree2[] = {1,1,2,1,0,0,1,2,2,2,2,2,2,2,2,2};

		int rt2_0_0 = (F0[i] <= (6.05 + offset2));
		int t2_0_0 = rt2_0_0 * (2 + F0[i] <= (6.15 + offset2));
		t2_0_0 += (1 - rt2_0_0) * ( F0[i] <= (6.05 + offset2));


		int rt2_0_1 = (F2[i] <= (3.05 + offset2));
		int t2_0_1 = rt2_0_1 * (2 + F2[i] <= (5.35 + offset2));
		t2_0_1 += (1 - rt2_0_1) * ( F2[i] <= (3.05 + offset2));


		int rt2_0_2 = (F3[i] <= (1.7 + offset2));
		int t2_0_2 = rt2_0_2 * (2 + F3[i] <= (1.7 + offset2));
		t2_0_2 += (1 - rt2_0_2) * ( F3[i] <= (1.7 + offset2));


		int rt2_0_3 = (F3[i] <= (1.7 + offset2));
		int t2_0_3 = rt2_0_3 * (2 + F3[i] <= (1.7 + offset2));
		t2_0_3 += (1 - rt2_0_3) * ( F3[i] <= (1.7 + offset2));


		int rt2_1_0 = (F1[i] <= (2.85 + offset2));
		int t2_1_0 = rt2_1_0 * (8 + t2_0_1);
		t2_1_0 += (1 - rt2_1_0) * t2_0_0;


		int rt2_1_1 = (F3[i] <= (1.7 + offset2));
		int t2_1_1 = rt2_1_1 * (8 + t2_0_3);
		t2_1_1 += (1 - rt2_1_1) * t2_0_2;


		int rt2_2_0 = (F3[i] <= (1.7 + offset2));
		int t2_2_0 = rt2_2_0 * (12 + t2_1_1);
		t2_2_0 += (1 - rt2_2_0) * t2_1_0;


		Class[classesTree2[t2_2_0]]++;

		int p0 = (Class[0] > Class[1])?0:1;
		int Q0 = (Class[0] > Class[1])?Class[0]:Class[1];
		int p1 = (Class[2] > Q0)?2:p0;
		int Q1 = (Class[2] > Q0)?Class[2]:Q0;
		vR[i] = p1;
	}
}int main(int argc, char ** argv) {
     
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 150;
    printf("[CUDA]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float  * h_0,  * h_1,  * h_2,  * h_3,  * h_4;
    int * h_P, * hostRef;
    	h_0 = (float * ) malloc(nBytes);
	h_1 = (float * ) malloc(nBytes);
	h_2 = (float * ) malloc(nBytes);
	h_3 = (float * ) malloc(nBytes);
	h_4 = (float * ) malloc(nBytes);
    hostRef = (int * ) malloc(nBytes);
    h_P = (int * ) malloc(nBytes);

    printf("[CUDA]: Start Reading Dataset\n");
	float* h[5] = { h_0, h_1, h_2, h_3, h_4};
	readInFile(h);
	printf("[CUDA]: Finished Reading Dataset\n");


    memset(hostRef, 0, nBytes);
    memset(h_P, 0, nBytes);

    // malloc device global memory
    float * d_0, * d_1, * d_2, * d_3, * d_4;
    int * d_P;
        CHECK(hipMalloc((float ** ) & d_0, nBytes));
    CHECK(hipMalloc((float ** ) & d_1, nBytes));
    CHECK(hipMalloc((float ** ) & d_2, nBytes));
    CHECK(hipMalloc((float ** ) & d_3, nBytes));
    CHECK(hipMalloc((float ** ) & d_4, nBytes));
    CHECK(hipMalloc((int ** ) & d_P, nBytes));


    // transfer data from host to device
        CHECK(hipMemcpy(d_0, h_0, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_1, h_1, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_2, h_2, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_3, h_3, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_4, h_4, nBytes, hipMemcpyHostToDevice));
    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_kernel << < grid, block >>> (d_0, d_1, d_2, d_3, d_4, nElem, 0,0,0, d_P);
    hipDeviceSynchronize();
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("[CUDA]: RF without IF - execution time = %.6fms\n", elapsed_time);

    registerTime(elapsed_time);
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    printf("\n ");

    // free host memory
        hipFree(d_0);
    hipFree(d_1);
    hipFree(d_2);
    hipFree(d_3);
    hipFree(d_4);
    hipFree(d_P);
        free(h_0);
    free(h_1);
    free(h_2);
    free(h_3);
    free(h_4);
    free(hostRef);

    return (0);
 }