#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define FLAG_NODE_LEAF_MASK 0x80000000
#define COLUMN_MASK 0x7C000000
#define RIGHT_INDEX_MASK 0x03FFE000
#define LEFT_INDEX_MASK 0x00001FFF
#define CLASS_MASK 0x03FFE000
#define NEXT_TREE_INDEX_MASK 0x00001FFF

#define GET_FLAG_NODE_LEAF(x) ((x & FLAG_NODE_LEAF_MASK) >> 31)
#define GET_COLUMN(x) ((x & COLUMN_MASK) >> 26)
#define GET_RIGHT_INDEX(x)  ((x & RIGHT_INDEX_MASK) >> 13)
#define GET_LEFT_INDEX(x)  (x & LEFT_INDEX_MASK)
#define GET_CLASS(x)  ((x & CLASS_MASK) >> 13)
#define GET_NEXT_TREE_INDEX(x)  (x & NEXT_TREE_INDEX_MASK)
#define NUM_THREADS 4
#define N_ELEM 150
#define MAX_LINE_LENGTH 1000

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}
__device__ int counter = 0;


void readInFile(float **ip) {
    FILE *file = fopen("assets/datasets/Iris.csv","r");
    if (file == NULL) {
        printf("[CUDA]: Failed to open the file.\n");
        exit(1);
    }

    char line[MAX_LINE_LENGTH];
    
    int row = 0;
    fgets(line, sizeof(line), file);
    while (fgets(line, sizeof(line), file)) {
        char *token = strtok(line, ",");
        
        int column = 0;
        while (token != NULL) {
            ip[column][row] = atof(token);
            token = strtok(NULL, ",");
            column++;
        }

        row++;
        if(row > N_ELEM) break;
    }

    fclose(file);
}

void writeOutFile(int *h, int size)
{
    FILE *file = fopen("out_rf_table_gpu.csv", "w");
    if (file == NULL) {
        printf("[CUDA]: Failed to open the file.\n");
        return;
    }

    for (int i = 0; i < size; i++) {
        fprintf(file, "%d\n", h[i]);
    }

    fclose(file);
}

void registerTime(int value) {
    FILE *file = fopen("results/results.csv", "a"); // Abre o arquivo no modo de "acrescentar" (append)
    if (file == NULL) {
        printf("[CUDA]: Falha ao abrir o arquivo results.csv.\n");
        return;
    }

    fprintf(file, ",%d", value); // Escreve o valor no arquivo, separado por vírgula
    fclose(file);
}

__global__ void RF_with_table(const int* __restrict__ table, const float* __restrict__ thresholds,float *F0, float *F1, float *F2, float *F3, float *F4, float *F5, int *P, const int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float* h[6] = { F0, F1, F2, F3, F4, F5};




    if(i < N) {
        int Class[3];
        Class[0] = 0;
        Class[1] = 0;
        Class[2] = 0;
        int index = 0;

        while(index < 26){
            //Flag:0 -> Nó Interno, Flag:1 -> Folha
            if(!GET_FLAG_NODE_LEAF(table[index])){
                if(h[GET_COLUMN(table[index])][i] <= thresholds[index]){
                    index = GET_LEFT_INDEX(table[index]);
                    atomicAdd(&counter, 1);
                } else {
                    index = GET_RIGHT_INDEX(table[index]);
                    atomicAdd(&counter, 1);
                }
            } else {
                Class[GET_CLASS(table[index])]++; 
                index = GET_NEXT_TREE_INDEX(table[index]);
            }
        }

		int p0 = (Class[0] > Class[1])?0:1;
		int Q0 = (Class[0] > Class[1])?Class[0]:Class[1];
		int p1 = (Class[2] > Q0)?2:p0;
		int Q1 = (Class[2] > Q0)?Class[2]:Q0;
		P[i] = p1;
    }
}

int main(int argc, char ** argv) {
     
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 150;
    printf("[CUDA]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    size_t nTableBytes = 26 * sizeof(int);    int tableData[] = 
	{ 201342977, -2147483633, 134266883, 201367556, -2147475441, -2147467249, 98311, 201416712, 67190793, -2147475441, -2147467249, -2147467249, 201441293, -2147475441, -2147467249,
	155664, 67256337, -2147475430, -2147483622, 201531412, 134414357, 67297302, -2147475430, -2147483622, -2147467238, -2147467238 };


    float thresholdsData[] = 
	{ 0.8, 0.0, 4.75, 1.65, 0.0, 0.0, 6.6, 1.85, 2.65, 0.0, 0.0, 0.0, 1.75, 0.0, 0.0, 
	5.55, 2.15, 0.0, 0.0, 1.75, 5.3, 3.55, 0.0, 0.0, 0.0, 0.0 };



    float  * h_0,  * h_1,  * h_2,  * h_3,  * h_4,  * h_5;
    int * h_P, * hostRef;
	h_0 = (float * ) malloc(nBytes);
	h_1 = (float * ) malloc(nBytes);
	h_2 = (float * ) malloc(nBytes);
	h_3 = (float * ) malloc(nBytes);
	h_4 = (float * ) malloc(nBytes);
	h_5 = (float * ) malloc(nBytes);
    hostRef = (int * ) malloc(nBytes);
    h_P = (int * ) malloc(nBytes);

	printf("[CUDA]: Start Reading Dataset\n");
	float* h[6] = { h_0, h_1, h_2, h_3, h_4, h_5};
	readInFile(h);
	printf("[CUDA]: Finished Reading Dataset\n");


    memset(hostRef, 0, nBytes);
    memset(h_P, 0, nBytes);

    // malloc device global memory
    float * d_0, * d_1, * d_2, * d_3, * d_4, * d_5,  * thresholds;
    int * d_P, * table;
    CHECK(hipMalloc((float ** ) & d_0, nBytes));
    CHECK(hipMalloc((float ** ) & d_1, nBytes));
    CHECK(hipMalloc((float ** ) & d_2, nBytes));
    CHECK(hipMalloc((float ** ) & d_3, nBytes));
    CHECK(hipMalloc((float ** ) & d_4, nBytes));
    CHECK(hipMalloc((float ** ) & d_5, nBytes));
    CHECK(hipMalloc((int ** ) & d_P, nBytes));

    CHECK(hipMalloc((float ** ) & table, nTableBytes));
    CHECK(hipMalloc((float ** ) & thresholds, nTableBytes));
    // transfer data from host to device
    CHECK(hipMemcpy(table, tableData, nTableBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(thresholds, thresholdsData, nTableBytes, hipMemcpyHostToDevice));    CHECK(hipMemcpy(d_0, h_0, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_1, h_1, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_2, h_2, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_3, h_3, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_4, h_4, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_5, h_5, nBytes, hipMemcpyHostToDevice));
    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_with_table << < grid, block >>> (table, thresholds, d_0, d_1, d_2, d_3, d_4, d_5, d_P, nElem);
    hipDeviceSynchronize();
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("[CUDA]: RF with Table (shared memory) - execution time = %.6fms\n", elapsed_time);

    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
	 int counterValue;
    hipMemcpyFromSymbol(&counterValue, HIP_SYMBOL(counter), sizeof(int), 0, hipMemcpyDeviceToHost);
    registerTime(counterValue);

    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    printf("\n ");

    // free host memory
    hipFree(d_0);
    hipFree(d_1);
    hipFree(d_2);
    hipFree(d_3);
    hipFree(d_4);
    hipFree(d_5);
    hipFree(d_P);
    free(h_0);
    free(h_1);
    free(h_2);
    free(h_3);
    free(h_4);
    free(h_5);
    free(hostRef);

    return (0);
 }